#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <vector>
#include <string>
#include "header.h"
#include <stdlib.h>
#include <streambuf>
#include <sstream>
#include <fstream>
#include <math.h>
#include "CImg.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include "cifar10_reader.hpp"
#include <boost/filesystem.hpp>

#define tau 10
#define exp_coeff 1.442695
#define SIZE 50000  //for ROI, use 30000
#define MAX_TIME 2500000 //in ms
#define TEST_TIME 1000

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      //if (abort) exit(code);
   }
}


#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return;}} while(0)



__global__ void print_log (float *log_v, int i){
	printf("_%f_", log_v[i]);
	//printf("time of %d: _%f_",i,log_v[i]);
}

__global__ void random (float *random_number, int rand_number_size, hiprandState_t *state){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    if (index>=rand_number_size) return;
	random_number[index] = (hiprand(&state[index])%1000)/1000.0;
    //printf("rand_gen_complete\n");
	//if(index==31)printf("The no.%d of random nubmer is %f\n", index, random_number[index]);
	//if(index==31)printf("%f|", random_number[index]);

}

__global__ void rand_init (unsigned int seed, int size, hiprandState_t *states){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    if (index>=size) return;
	hiprand_init(seed, index, 0, &states[index]);
    //printf("rand_init_complete\n");
}

__global__ void read_filter_GPU_one_layer (CNN_struct *settings, float *device_filter_array, int layer_num){
	int counter = 0;
	printf("Printing filter array on GPU\n");

	int filter_size = settings->layer[layer_num].conv_setting.filter_depth * settings->layer[layer_num].conv_setting.filter_width * settings->layer[layer_num].conv_setting.filter_length * settings->layer[layer_num].depth;
	for(int j=0;j<filter_size;j++){
		printf("%f ", device_filter_array[j]);
		counter ++;
	}
	printf("\n");
}

__global__ void read_filter_GPU (CNN_struct *settings, float **device_filter_array){
	int counter = 0;
	printf("Printing filter array on GPU\n");
	for (int i=0;i<CNN_total_layer_num-1;i++){
		int filter_size = settings->layer[i+1].conv_setting.filter_depth * settings->layer[i+1].conv_setting.filter_width * settings->layer[i+1].conv_setting.filter_length * settings->layer[i+1].depth;
		for(int j=0;j<filter_size;j++){
			printf("%f ", device_filter_array[i][j]);
			counter ++;
		}
		printf("\n");
	}
}

__global__ void change_threshold (Neuron *NeuronList, int network_size, float start_depth, float end_depth, float target_threshold){
    //int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    //int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    //printf("StartDepth:%f_End:%f__current:%f||", start_depth, end_depth, NeuronList[index].param[7]);
	for (int index=0; index<=network_size; index ++){
//		printf("%d in %d |", index, network_size);
//		if(index>40000)printf("id %d StartDepth:%f_End:%f__current:%f||", index, start_depth, end_depth, NeuronList[index].param[7]);
		if(index>=network_size){
			return;
		}

		if((NeuronList[index].param[7]<start_depth||NeuronList[index].param[7]>end_depth)){
	//		printf("StartDepth:%f_End:%f__current:%f||", start_depth, end_depth, NeuronList[index].param[7]);
			continue;
		}
		if(NeuronList[index].type==2){
//			if(index>40000)printf("StartDepth:%f_End:%f__current:%f||", start_depth, end_depth, NeuronList[index].param[7]);
			NeuronList[index].param[1] = target_threshold;
		}
		else{
			continue;
		}
	}

}

__global__ void update_param (Neuron *NeuronList, int network_size, float start_depth, float end_depth, int target_param, float target_value){
    //int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    //int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    //printf("StartDepth:%f_End:%f__current:%f||", start_depth, end_depth, NeuronList[index].param[7]);
	for (int index=0; index<=network_size; index ++){
//		printf("%d in %d |", index, network_size);
//		if(index>40000)printf("id %d StartDepth:%f_End:%f__current:%f||", index, start_depth, end_depth, NeuronList[index].param[7]);
		if(index>=network_size){
			return;
		}

		if((NeuronList[index].param[7]<start_depth||NeuronList[index].param[7]>end_depth)){
	//		printf("StartDepth:%f_End:%f__current:%f||", start_depth, end_depth, NeuronList[index].param[7]);
			continue;
		}
		if(NeuronList[index].type==2){
//			if(index>40000)printf("StartDepth:%f_End:%f__current:%f||", start_depth, end_depth, NeuronList[index].param[7]);
			NeuronList[index].param[target_param] = target_value;
		}
		else{
			continue;
		}
	}

}

__global__ void change_state(Neuron *NeuronList, int network_size, float start_depth, float end_depth, int target_param, float target_value){
    //int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    //int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    //printf("StartDepth:%f_End:%f__current:%f||", start_depth, end_depth, NeuronList[index].param[7]);
	for (int index=0; index<=network_size; index ++){
//		printf("%d in %d |", index, network_size);
//		if(index>40000)printf("id %d StartDepth:%f_End:%f__current:%f||", index, start_depth, end_depth, NeuronList[index].param[7]);
		if(index>=network_size){
			return;
		}

		if((NeuronList[index].param[7]<start_depth||NeuronList[index].param[7]>end_depth)){
	//		printf("StartDepth:%f_End:%f__current:%f||", start_depth, end_depth, NeuronList[index].param[7]);
			continue;
		}

		NeuronList[index].state[target_param] = target_value;

	}

}

__global__ void reset_membrane_potential (Neuron *NeuronList, int network_size, float start_depth, float end_depth){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    //printf("StartDepth:%f_End:%f__current:%f||", start_depth, end_depth, NeuronList[index].param[7]);
	for (int index=0; index<network_size; index ++){
//		printf("%d in %d |", index, network_size);
//		if(index>40000)printf("id %d StartDepth:%f_End:%f__current:%f||", index, start_depth, end_depth, NeuronList[index].param[7]);

		if((NeuronList[index].param[7]<start_depth||NeuronList[index].param[7]>end_depth)){
	//		printf("StartDepth:%f_End:%f__current:%f||", start_depth, end_depth, NeuronList[index].param[7]);
			continue;
		}
		if(NeuronList[index].type==2){
//			if(index>40000)printf("StartDepth:%f_End:%f__current:%f||", start_depth, end_depth, NeuronList[index].param[7]);
			NeuronList[index].state[0] = NeuronList[index].param[2];
		}
		else{
			continue;
		}
	}

}

__global__ void reset_all_state (Neuron *NeuronList, int network_size, float start_depth, float end_depth){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    //printf("StartDepth:%f_End:%f__current:%f||", start_depth, end_depth, NeuronList[index].param[7]);
	for (int index=0; index<=network_size; index ++){
//		printf("%d in %d |", index, network_size);
//		if(index>40000)printf("id %d StartDepth:%f_End:%f__current:%f||", index, start_depth, end_depth, NeuronList[index].param[7]);
		if(index>=network_size){
			return;
		}

		if((NeuronList[index].param[7]<start_depth||NeuronList[index].param[7]>end_depth)){
	//		printf("StartDepth:%f_End:%f__current:%f||", start_depth, end_depth, NeuronList[index].param[7]);
			continue;
		}
		if(NeuronList[index].type==2){
			for (int state_id=0; state_id<8; state_id++){
				NeuronList[index].state[state_id] = 0;
				if (state_id==0) NeuronList[index].state[state_id] = NeuronList[index].param[2];
			}
		}
		else{
			continue;
		}
	}

}


__global__ void lateral_inhibition_child (Neuron *NeuronList, int network_size, int inhibit_time, float start_depth, float end_depth, int depth_iter){
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;


	if(index>=network_size){
		return;
	}
	if(depth_iter>0 && fabsf(NeuronList[index].param[7]-depth_iter)>0.01){
		return;
	}
	if(NeuronList[index].type==4){
		return;
	}
	if(NeuronList[index].state[2]>0.1){
		//printf("******************%d*****************\n", index);
		return;
	}

	if((start_depth>0 && end_depth>0) && (NeuronList[index].param[7]<start_depth||NeuronList[index].param[7]>end_depth)){
		//printf("StartDepth:%f_End:%f__current:%f||", start_depth, end_depth, NeuronList[index].param[7]);
		return;
	}
	//printf("%d %d| ", index, network_size);
	//printf("%d | ", index);
	NeuronList[index].state[7] = inhibit_time;	//
	NeuronList[index].state[0] = NeuronList[index].param[2] + 0.5*(NeuronList[index].param[1]-NeuronList[index].param[2]);
	//NeuronList[index].state[0] = NeuronList[index].state[0] - 0.5*(NeuronList[index].param[1]-NeuronList[index].param[2]);//NeuronList[index].param[2];				//change mem potential to reset_value
	//if(NeuronList[index].state[0]<NeuronList[index].param[2]) NeuronList[index].state[0]=NeuronList[index].param[2];
	//float *result = std::find(std::begin(NeuronList[index].state), std::end(NeuronList[index].state), 123);

}

__global__ void lateral_inhibition_mother_thread (Neuron *NeuronList, int network_size, int layer_ind_to_learn, int inhibit_time, CNN_struct *CNN_setttings, int *spike_flag){

	if (threadIdx.x==0){
    	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
    		//if (layer_iter==1) printf("Layer[%d] SpikeFlag: %d\n", layer_iter, spike_flag[layer_iter]);
            if(spike_flag[layer_iter]>0 && layer_iter==layer_ind_to_learn){//use lateral inhibition
            	int SIZE_PER_SIDE = sqrt((float)network_size)+1;
            	float start_depth = CNN_setttings->layer[layer_iter].first_depth_id - 0.1;
            	float end_depth = CNN_setttings->layer[layer_iter].last_depth_id + 0.1;
            	dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
            	dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
            	//printf("Start_depth: %f, end_depth: %f||", start_depth, end_depth);
        		lateral_inhibition_child<<<dimGrid, dimBlock>>>(NeuronList, network_size, inhibit_time, start_depth, end_depth, -1);
            }
    		spike_flag[layer_iter] = 0;
    	}

	}
	__syncthreads();

}

__global__ void lateral_inhibition_depth_wise_mother_thread (Neuron *NeuronList, int network_size, int depth_ind_start, int depth_ind_end, int inhibit_time, CNN_struct *CNN_setttings, float *spike_flag, int total_depth_number){
	if (threadIdx.x==0){
    	for(int depth_iter=depth_ind_start;depth_iter<depth_ind_end;depth_iter++){
    		//if (layer_iter==0) printf("Layer[%d] SpikeFlag: %d\n", layer_iter, spike_flag[layer_iter]);
            if(spike_flag[depth_iter]>0){//use lateral inhibition
            	int SIZE_PER_SIDE = sqrt((float)network_size)+1;

            	dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
            	dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
            	//printf("Start_depth: %f, end_depth: %f||", start_depth, end_depth);
        		lateral_inhibition_child<<<dimGrid, dimBlock>>>(NeuronList, network_size, inhibit_time, -1, -1, depth_iter);
            }
    		spike_flag[depth_iter] = 0;
    	}

	}
	__syncthreads();

}

int find_max_potential(Neuron *NeuronList){
	int max_index = 0;
	float max_v = -100;

	for(int i=0; i<SPIKING_NEURON_NUM; i++){
		printf("v_of_%d_is%f\n", i, NeuronList[i].state[0]);
		if (NeuronList[i].state[0]>max_v){

			max_index = i;
			max_v = NeuronList[i].state[0];
		}
	}
	return max_index;
}



void input_neuron_list_init(Input_neuron *NeuronList, int network_size){

	int i = 0;
	while (i<network_size){
		int j = 0;
		NeuronList[i].index = i+1;
		NeuronList[i].type = 4;
		NeuronList[i].spike_cnt = 0;
		NeuronList[i].spike_frequency = -1;
		while(j<8){
			NeuronList[i].param[j] = -1;
			NeuronList[i].state[j] = 0;
			j++;
		}
	i++;
	}
}

void neuron_list_init(Neuron *NeuronList, int network_size){

	int i = 0;
	while (i<network_size){
		int j = 0;
		NeuronList[i].index = -1;
		NeuronList[i].type = -1;
		NeuronList[i].spike_cnt = 0;
		NeuronList[i].spike_frequency = 0;
		while (j<MAX_CONNECTION){
			NeuronList[i].connected_in[j] = 0;
			NeuronList[i].connected_weight[j] = 0;
			j++;
		}
		j = 0;
		while(j<8){
			NeuronList[i].param[j] = -1;
			NeuronList[i].state[j] = 0;
			j++;
		}
	i++;
	}
}

void neuron_list_init(Neuron *NeuronList){

	int i = 0;
	while (i<SIZE){
		int j = 0;
		NeuronList[i].index = -1;
		NeuronList[i].type = -1;

		while (j<MAX_CONNECTION){
			NeuronList[i].connected_in[j] = 0;
			NeuronList[i].connected_weight[j] = 0;
			j++;
		}
		j = 0;
		while(j<8){
			NeuronList[i].param[j] = -1;
			NeuronList[i].state[j] = 0;
			j++;
		}
	i++;
	}
}

void find_fired(Neuron *NeuronList, int *fire_list, int *fired_no){
	int i = 0;
	int fired_count = 0;
	//int fire_list [SIZE] = { };
	while(i<SIZE){
		if(i<5){
			//printf("The no. %d neuron is timed: %f\n", i, NeuronList[i].state.head->data);
			//NeuronList[2].state.display();
		}
		if (NeuronList[i].state[0] > 0.1){
			//printf("state of %d is %f\n", i, NeuronList[i].state[0]);
			fire_list[fired_count] = i+1;
			fired_count ++;
			//printf("The no. of fired neuron is %d\n", i);
		}
		i++;
	}
	*fired_no = fired_count;
}

void check_neuron(Neuron *NeuronList, int start_index, int end_index){
	cout<<"===check_neuron==="<<endl;
	for(int i=start_index; i<=end_index; i++){
		cout<<NeuronList[i].index<<" "<<NeuronList[i].type<<" ";
		for(int j=0; j<8; j++){
			cout<<NeuronList[i].param[j]<<" ";
		}
		for(int j=0; j<8; j++){
			cout<<NeuronList[i].state[j]<<" ";
		}
	cout<<endl;
	}
}

void izh_parameter_init(float *izh_parameters){
	izh_parameters[0] = SIZE;
}

void init_log_v (float *log_v){
	int i = 0;
	while (i<MAX_TIME){
		log_v[i] = 0;
		i++;
	}
}

void init_data_log (float *log_v_host, float *log_spike_host, float *log_total_spike_host, int inter){
	int i=0;
	while(i<inter){
		log_v_host[i] = 0;
		log_spike_host[i] = 0;
		i++;
	}

	int j=0;
	while(j<SIZE){
		log_total_spike_host[j] = 0;
		j++;
	}


}

void spiking_learning_label(){
	int total_neuron_num = 4300;

    Neuron *NeuronList = new Neuron[total_neuron_num];
	neuron_list_init(NeuronList, total_neuron_num);
	cout<<"1"<<endl;
	read_neuron_list(NeuronList, 1, "device2_output_network.txt");
	float *log_total_spike_host = new float[total_neuron_num];
	for(int i=0; i < total_neuron_num; i++){
		log_total_spike_host[i] = 0;
	}
	float *log_total_spike;
    gpuErrchk( hipMalloc((void **)&log_total_spike, total_neuron_num * sizeof(float)) );
    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,total_neuron_num*sizeof(float),hipMemcpyHostToDevice) );
	cout<<"2"<<endl;

    int mnist_start_index = 0;
    int mnist_end_index = input_image_w*input_image_l*input_image_channel;

    data_check(NeuronList,log_total_spike,total_neuron_num, mnist_start_index, mnist_end_index, 2, "");
}

float spiking_learning_label(string network_data, string flag_file, int input_index, int num_test, int function_select, int data_set_select){

	printf("Extracting weight/n");
	int spiking_neuron_num = 511;
	//spiking_neuron_num = 1000;
	int training_set_number = 10000;
	int input_neuron_num = input_image_w*input_image_l*input_image_channel;
	int *mnist_label = new int[training_set_number];
	Neuron *NeuronList = new Neuron[spiking_neuron_num];


	printf("Reading neuron list: ");
	neuron_list_init(NeuronList, spiking_neuron_num);
	read_neuron_list(NeuronList, 1, network_data);
	printf("Done!\n");
	//print all weight to file;
	string weight_file_name = "all_weight.csv";
    ofstream myfile_weight (weight_file_name);
    if (myfile_weight.is_open()){
    	//myfile << "This is a new test\n";
    	for(int i=0; i < 510; i++){
    		//printf("_%f_", log_v_host[i]);
    		int j = 0;
    		while(NeuronList[i].connected_in[j] > 0.1){
//    			printf("-%f",NeuronList[i].connected_weight[j]);
    			myfile_weight << NeuronList[i].connected_weight[j] << ", ";
    			j++;
    		}
    		myfile_weight<<endl;
    	}
    	myfile_weight.close();
    }
//	float *log_total_spike = new float[SIZE];
    //data_check(NeuronList,log_total_spike,SIZE, 0, input_neuron_num, 2, "");


	delete[] NeuronList;
    return 1.0;
}
//
//void run_test(){
//	//1. Test of single neuron reaction under constant input current
//
//	Neuron *NeuronList = new Neuron[SIZE];
//	hiprandState_t *states;
//	float *random_number_list = new float[SIZE];
//	float *log_v_host = new float[MAX_TIME];
//	float *log_spike_host = new float[MAX_TIME];
//	float *log_total_spike_host = new float[SIZE];
//	init_log_v(log_v_host);
//
//	neuron_list_init(NeuronList);
//	read_neuron_list(NeuronList, 1, "data_test.txt");
//	for(int z=0;z<10;z++){
//		//printf("=%d=",NeuronList[z].type);
//	}
//	Neuron *Neuron_list_device;
//	Neuron *old_device_neurons;
//	float *random_number_list_device;
//	float *log_v;
//	float *log_spike;
//	float *log_total_spike;
//
//	int SIZE_PER_SIDE = sqrt(SIZE)+1;
//    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
//    dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
//	dim3 print_grid(1);
//	dim3 print_block(1);
//
//    hipMalloc((void **)&Neuron_list_device, SIZE*sizeof(Neuron));
//    hipMalloc((void **)&old_device_neurons, SIZE*sizeof(Neuron));
//    hipMalloc((void **)&random_number_list_device,SIZE*sizeof(float));
//    hipMalloc((void **)&states, SIZE * sizeof(hiprandState_t));
//    hipMalloc((void **)&log_v, MAX_TIME * sizeof(float));
//    hipMalloc((void **)&log_spike, MAX_TIME * sizeof(float));
//    hipMalloc((void **)&log_total_spike, SIZE * sizeof(float));
//
//
//    rand_init<<<dimGrid,dimBlock>>>(time(0), SIZE, states);
//
//    hipMemcpy(Neuron_list_device,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
//    hipMemcpy(old_device_neurons,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
//    hipMemcpy(random_number_list_device, random_number_list, SIZE*sizeof(float), hipMemcpyHostToDevice);
//    hipMemcpy(log_v,log_v_host,MAX_TIME*sizeof(float),hipMemcpyHostToDevice);
//    hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice);
//
//    int network_size = SIZE;
//    int time = 0;
//    int max_time = MAX_TIME;
//    while (time<max_time){
//        //random<<<dimGrid,dimBlock>>>(random_number_list_device, states);
//        neuron_test(Neuron_list_device, old_device_neurons, random_number_list_device, network_size, log_v, log_spike, log_total_spike, time);
//
//        hipMemcpy(old_device_neurons,Neuron_list_device,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
//
//    	time ++;
//    }
//
//	int j;
//	for(j=0;j<MAX_TIME;j++){
//		//print_log<<<print_grid,print_block>>>(log_v, j);
//	}
//	//hipDeviceSynchronize();
//    hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
//    hipMemcpy(log_v_host,log_v,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
//    hipMemcpy(log_spike_host,log_spike,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
//    hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost);
//    //printf("cpy Done");
//
//    //====write to file=====
//    ofstream myfile ("out_v.csv");
//    if (myfile.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile << log_v_host[i] << ", ";
//    	}
//    	myfile.close();
//    }
//
//    ofstream myfile_2 ("out_spike.csv");
//    if (myfile_2.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_2 << log_spike_host[i] << ", ";
//    	}
//    	myfile_2.close();
//    }
//
//    ofstream myfile_3 ("out_total_spike.csv");
//    if (myfile_3.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < SIZE; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_3 << log_total_spike_host[i] << ", ";
//    	}
//    	myfile_3.close();
//    }
//
//    //===clean up===
//    delete[] random_number_list;
//    delete[] log_v_host;
//	delete[] NeuronList;
//	delete[] log_spike_host;
//
//
//	hipFree(states);
//	hipFree(log_v);
//	hipFree(log_spike);
//	hipFree(Neuron_list_device);
//	hipFree(old_device_neurons);
//	hipFree(random_number_list_device);
//
//}
//
//void run_Spiking(){
//	Neuron *NeuronList = new Neuron[SIZE];
//
//	neuron_list_init(NeuronList);
//	read_neuron_list(NeuronList, 1, "data_tsp.txt");
//
//	Neuron *Neuron_list_device;
//	Neuron *old_device_neurons;
//
//	int SIZE_PER_SIDE = sqrt(SIZE)+1;
//    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
//    dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
//
//
//    hipMalloc((void **)&Neuron_list_device, SIZE*sizeof(Neuron));
//    hipMalloc((void **)&old_device_neurons, SIZE*sizeof(Neuron));
//    //random<<<dimGrid,dimBlock>>>(random_number_list_device,states,number_of_block_y, number_of_threads_y);
//
//    hipMemcpy(Neuron_list_device,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
//    hipMemcpy(old_device_neurons,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
//
//    //int network_size = SIZE;
//    int time = 0;
//    int max_time = MAX_TIME;
//
//    while (time<max_time){
//    	//printf("|||||time is %d|||||\n", time);
//        //kernel_spiking(Neuron_list_device, network_size);
//        hipMemcpy(old_device_neurons,Neuron_list_device,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
//    	time ++;
//    }
//    hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
//
//	delete[] NeuronList;
//
//	hipFree(Neuron_list_device);
//}
//
//void run_Stoc(){
//
//	Neuron *NeuronList = new Neuron[SIZE];
//	hiprandState_t *states;
//	float *random_number_list = new float[SIZE];
//
//	neuron_list_init(NeuronList);
//	read_neuron_list(NeuronList, 1, "data_tsp.txt");
//
//	Neuron *Neuron_list_device;
//	Neuron *old_device_neurons;
//	float *random_number_list_device;
//
//	int SIZE_PER_SIDE = sqrt(SIZE)+1;
//    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
//    dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
//
//
//    hipMalloc((void **)&Neuron_list_device, SIZE*sizeof(Neuron));
//    hipMalloc((void **)&old_device_neurons, SIZE*sizeof(Neuron));
//    hipMalloc((void **)&random_number_list_device,SIZE*sizeof(float));
//    hipMalloc((void **)&states, SIZE * sizeof(hiprandState_t));
//
//    rand_init<<<dimGrid,dimBlock>>>(time(0), SIZE, states);
//    //random<<<dimGrid,dimBlock>>>(random_number_list_device,states,number_of_block_y, number_of_threads_y);
//
//    hipMemcpy(Neuron_list_device,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
//    hipMemcpy(old_device_neurons,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
//    hipMemcpy(random_number_list_device,random_number_list,SIZE*sizeof(float),hipMemcpyHostToDevice);
//
//
//    int network_size = SIZE;
//    int time = 0;
//    int max_time = MAX_TIME;
//    while (time<max_time){
//    	//printf("|||||time is %d|||||\n", time);
//
//        random<<<dimGrid,dimBlock>>>(random_number_list_device, SIZE,states);
//        kernel_neuron(Neuron_list_device, old_device_neurons, random_number_list_device, network_size);
//        hipMemcpy(old_device_neurons,Neuron_list_device,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
//    	time ++;
//    }
//    hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
//    //printf("cpy Done");
//    int *fired_list = new int[SIZE];
//    int fired_no;
//    find_fired(NeuronList, fired_list, &fired_no);
//	printf("fired neuron no. is %d\n", fired_no);
//
//    for (int i = 0; i<fired_no; i++){
//    	//printf("fired neuron is %d\n", fired_list[i]);
//    }
//
//
//    delete[] random_number_list;
//    delete[] fired_list;
//	delete[] NeuronList;
//
//	hipFree(Neuron_list_device);
//	hipFree(old_device_neurons);
//	hipFree(random_number_list_device);
//}
//
//void run_ROI(){
//	//ROI
//	//First read image
//	cimg_library::CImg<unsigned char> image("color_small.jpg");
//	float signal_max = 0.6;
//	float signal_min = 1.8;
//	float img_signal [img_width][img_len][3];
//
//	//unsigned char* ptr = image.data(10,10, 0, 1); // get pointer to pixel @ 10,10
//	//unsigned char pixel = *ptr;
//
//	int img_i;
//	int img_j;
//	int img_k;
//	for (img_i=0;img_i<img_width;img_i++){
//		for (img_j=0;img_j<img_len;img_j++){
//			for(img_k=0;img_k<3;img_k++){
//				float img_temp = (float)image(img_i, img_j, 0, img_k)/255;
//				img_temp = img_temp*(signal_max-signal_min)+signal_min; //
//				img_signal[img_i][img_j][img_k] = img_temp;
//				//printf("pixel%d, %d, signal is: %f \n",img_i, img_j, img_temp);
//			}
//		}
//	}
//	//finish reading image
//	int signal_start_1 = img_width*img_len*3;
//	int signal_end_1 = img_width*img_len*6;
//
//	Neuron *NeuronList = new Neuron[SIZE];
//	hiprandState_t *states;
//	float *random_number_list = new float[SIZE];
//	float *log_v_host = new float[MAX_TIME];
//	float *log_spike_host = new float[MAX_TIME];
//	float *log_total_spike_host = new float[SIZE];
//
//	init_log_v(log_v_host);
//	neuron_list_init(NeuronList);
//	//printf("=0=\n");
//	read_neuron_list(NeuronList, 1, "visual_IZH.txt");
//
//	Neuron *Neuron_list_device;
//	Neuron *old_device_neurons;
//	float *random_number_list_device;
//	float *log_v;
//	float *log_spike;
//	float *log_total_spike;
//	//printf("=1=\n");
//	int SIZE_PER_SIDE = sqrt(SIZE)+1;
//    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
//    dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
//	dim3 print_grid(1);
//	dim3 print_block(1);
//
//    hipMalloc((void **)&Neuron_list_device, SIZE*sizeof(Neuron));
//    hipMalloc((void **)&old_device_neurons, SIZE*sizeof(Neuron));
//    hipMalloc((void **)&random_number_list_device,SIZE*sizeof(float));
//    hipMalloc((void **)&states, SIZE * sizeof(hiprandState_t));
//    hipMalloc((void **)&log_v, MAX_TIME * sizeof(float));
//    hipMalloc((void **)&log_spike, MAX_TIME * sizeof(float));
//    hipMalloc((void **)&log_total_spike, SIZE * sizeof(float));
//
//    rand_init<<<dimGrid,dimBlock>>>(time(0), SIZE, states);
//
//    hipMemcpy(Neuron_list_device,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
//    hipMemcpy(old_device_neurons,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
//    hipMemcpy(random_number_list_device, random_number_list, SIZE*sizeof(float), hipMemcpyHostToDevice);
//    hipMemcpy(log_v,log_v_host,MAX_TIME*sizeof(float),hipMemcpyHostToDevice);
//    hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice);
//
//    int network_size = SIZE;
//    int time = 0;
//    int max_time = MAX_TIME;
//
//    ROI_drive(old_device_neurons, (float *)img_signal, network_size, signal_start_1, signal_end_1, 1);
//    ROI_drive(old_device_neurons, (float *)img_signal, network_size, signal_start_1, signal_end_1, 0);
//    hipMemcpy(Neuron_list_device,old_device_neurons,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
//
//    hipDeviceSynchronize();
//
//    while (time<max_time){
//        //random<<<dimGrid,dimBlock>>>(random_number_list_device, states);
//        neuron_test(Neuron_list_device, old_device_neurons, random_number_list_device, network_size, log_v, log_spike, log_total_spike, time);
//        hipDeviceSynchronize();
//        hipMemcpy(old_device_neurons,Neuron_list_device,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
//        //hipDeviceSynchronize();
//        //print_log<<<print_grid,print_block>>>(log_v, 34);//correct this point
//    	time ++;
//    }
//
//	int j;
//	for(j=0;j<MAX_TIME;j++){
//		//print_log<<<print_grid,print_block>>>(log_v, j);
//	}
//	//hipDeviceSynchronize();
//    hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
//    hipMemcpy(log_v_host,log_v,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
//    hipMemcpy(log_spike_host,log_spike,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
//    hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost);
//    //printf("cpy Done");
//
//    ofstream myfile ("ROI_out.csv");
//    if (myfile.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < SIZE; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile << log_total_spike_host[i] << ", ";
//    	}
//    	myfile.close();
//    }
//
//    ofstream myfile_0 ("out_v.csv");
//    if (myfile_0.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_0 << log_v_host[i] << ", ";
//    	}
//    	myfile.close();
//    }
//
//    ofstream myfile_2 ("out_spike.csv");
//    if (myfile_2.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_2 << log_spike_host[i] << ", ";
//    	}
//    	myfile_2.close();
//    }
//
//    //===clean up===
//    delete[] random_number_list;
//    delete[] log_v_host;
//	delete[] NeuronList;
//	delete[] log_spike_host;
//
//
//	hipFree(states);
//	hipFree(log_v);
//	hipFree(log_spike);
//	hipFree(log_total_spike);
//	hipFree(Neuron_list_device);
//	hipFree(old_device_neurons);
//	hipFree(random_number_list_device);
//
//}
//
//void run_Spiking_learn(string index_prefix, float input_float, float input_float_2, int input_int, int input_int_2, string input_img){
//		hipSetDevice(0);
//		//set parameters
//
//		int training_time_each_img = input_int;
//		int calculated_total_time = training_time_each_img*50;
//		#undef MAX_TIME
//		#define MAX_TIME calculated_total_time
//		printf("==Training Total Iter: %d==", MAX_TIME);
//
//		float max_frequency = 22; //in Hz default 22
//		float min_frequency = 1;
//		int training_set_number = 60000;
//		int input_neuron_num = input_image_w * input_image_l*input_image_channel;
//		int spiking_neuron_num = SPIKING_NEURON_NUM;
//		int output_layer_neuron_num = OUTPUT_LAYER_NEURON_NUM;
//		int tenpercent_iter = MAX_TIME/10;
//
//		int connection_size = 900;
//		int syn_timer_max = 25;
//		int input_signal_width = 25;	//default 25
//		int inhibition_time = input_int_2;	//default 10
//
//		float target_frequency_param = 0.5;
//		float target_frequency = target_frequency_param*(1/(SPIKING_NEURON_NUM*inhibition_time));
//
//		float *mnist_img = new float[input_neuron_num*training_set_number];
//		string image_file = input_img; //"train-images-idx3-ubyte";
//		MNIST_read_image(image_file, mnist_img, training_set_number);
//		int *mnist_label = new int[training_set_number];
//		string image_label_file = "train-labels-idx1-ubyte";
//		MNIST_read_label(image_label_file, mnist_label, training_set_number);
//		//special_function: learn one category
//		int learn_one_digit = 0;
//		int *num_one_digit_img = new int[1];
//		if(learn_one_digit){
//			//
//			MNIST_labeling("abc", 60000, mnist_img, mnist_label, mnist_img, num_one_digit_img, spiking_neuron_num, 1, 5);
//		}
//
//		//int synapse_size = SIZE*SIZE;
//		Neuron *NeuronList = new Neuron[SIZE];
//		//unsigned char *synapse_timer = new unsigned char[synapse_size];  //this is the array that stores timer used in STPD. e.g Neuron x --->  Neuron y Spike! In the array index [(x-1)*SIZE+(y-1)]  => 1
//		//hiprandState_t *states;
//		//float *random_number_list = new float[SIZE];
//		float *log_v_host = new float[MAX_TIME];
//		float *log_spike_host = new float[MAX_TIME];
//		float *log_total_spike_host = new float[SIZE];
//		int *spike_flag = new int[1];
//		spike_flag[0] = 0;
//
//		//init_log_v(log_v_host);
//		init_data_log(log_v_host,log_spike_host,log_total_spike_host, MAX_TIME);
//		neuron_list_init(NeuronList);
//		//printf("=0=\n");
//		read_neuron_list(NeuronList, 1, "spike_learning_1000_v1.txt");
//	    //write_neuron_list(NeuronList, "learning_output_confirm.txt", SIZE);
//
//		Neuron *Neuron_list_device;
//		Neuron *old_device_neurons;
//		//unsigned char *snapse_timer_device;
//		float *log_v;
//		float *log_spike;
//		float *log_total_spike;
//		int *spike_flag_device;
//
//		//printf("=1=\n");
//		hiprandState_t *states;
//		hipMalloc((void **)&states, SIZE * sizeof(hiprandState_t));
//		int SIZE_PER_SIDE = sqrt(SIZE)+1;
//	    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
//	    dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
//		dim3 print_grid(1);
//		dim3 print_block(1);
//		rand_init<<<dimGrid,dimBlock>>>(time(0), SIZE, states);
//
//		int rand_numb_size = SPIKING_NEURON_NUM*MAX_CONNECTION;
//
//		float *random_number_list = new float[rand_numb_size];
//		float *random_number_list_device;
//		SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
//		dim3 dimBlock_synapse( ThreadsPerBlock, ThreadsPerBlock );
//		dim3 dimGrid_synapse( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
//		hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
//		hipMemcpy(random_number_list_device,random_number_list,rand_numb_size*sizeof(float),hipMemcpyHostToDevice);
//        random<<<dimGrid_synapse,dimBlock_synapse>>>(random_number_list_device, rand_numb_size, states);
//
//	    hipMalloc((void **)&Neuron_list_device, SIZE*sizeof(Neuron));
//	    hipMalloc((void **)&old_device_neurons, SIZE*sizeof(Neuron));
//
//	    //hipMalloc((void **)&states, SIZE * sizeof(hiprandState_t));
//	    hipMalloc((void **)&log_v, MAX_TIME * sizeof(float));
//	    hipMalloc((void **)&log_spike, MAX_TIME * sizeof(float));
//	    //hipMalloc((void **)&log_total_spike, SIZE * sizeof(float));
//	    hipMalloc((void **)&log_total_spike, SIZE * sizeof(float));
//	    hipMalloc((void **)&spike_flag_device, sizeof(int));
//	    //rand_init<<<dimGrid,dimBlock>>>(time(0), states);
//
//	    hipMemcpy(Neuron_list_device,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
//	    hipMemcpy(old_device_neurons,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
//	    //hipMemcpy(random_number_list_device, random_number_list, SIZE*sizeof(float), hipMemcpyHostToDevice);
//	    hipMemcpy(log_v,log_v_host,MAX_TIME*sizeof(float),hipMemcpyHostToDevice);
//	    hipMemcpy(log_spike,log_spike_host,MAX_TIME*sizeof(float),hipMemcpyHostToDevice);
//	    hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice);
//	    hipMemcpy(spike_flag_device,spike_flag,sizeof(int),hipMemcpyHostToDevice);
//
//	    int network_size = SIZE;
//	    int time = 0;
//	    int max_time = MAX_TIME;
//
//	    hipMemcpy(Neuron_list_device,old_device_neurons,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
//	    //first change raw img data into frequency
//	    int mnist_start_index = spiking_neuron_num;
//	    int mnist_end_index = spiking_neuron_num + input_neuron_num;
//	    MNIST_drive(NeuronList, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 1);
//
//
//	    hipDeviceSynchronize();
//
//	    //data_check(Neuron_list_device,log_total_spike,SIZE,1);
//	    float *one_mnist_img = new float[input_neuron_num];
//	    int training_img_index = 0;
//	    clock_t iter_start, iter_log;
//	    iter_start = clock();
//	    int log_interval = MAX_TIME/25;
//	    while (time<max_time){
//	        //random<<<dimGrid,dimBlock>>>(random_number_list_device, states);
//	    	//first create an array of 1 MNIST image
//	    	printf("\n iter_%d\n",time);
//	    	if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING){
//	            random<<<dimGrid,dimBlock>>>(random_number_list_device, rand_numb_size, states);
//	    	}
//	    	if(time%log_interval == 0){
//	    		hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
//	    		string interval_file_name = "device2_output_at_iter_" + to_string(time) + ".txt";
//	    		write_neuron_list(NeuronList, interval_file_name, network_size);
//	    		//printf("%");
//	    	}
//
//	    	if(time%tenpercent_iter == 0){
//	    		iter_log = clock();
//	    		cout<<to_string(10*(time/tenpercent_iter))<<"% done, time used is: " << (iter_log - iter_start)/1000 << " (ms)" << endl;
//	    	}
//
//	    	if(time%training_time_each_img==0){//at the beginning of each img's training, load into
//	    		for(int i=0;i<input_neuron_num;i++){
//	    			one_mnist_img[i] = mnist_img[training_img_index*input_neuron_num+i];
//	    		}
//	    	    for (int y=0; y<28; ++y) {
//	    	    	    for (int x=0; x<28; ++x) {
//	    	    	      //std::cout << ((one_mnist_img[y*28+x] == 0.0)? ' ' : '*');
//	    	    	      std::cout << std::to_string(int(one_mnist_img[y*28+x])) << ' ';
//	    	    	    }
//	    	    	    std::cout << std::endl;
//	    	    }
//	    		MNIST_drive(Neuron_list_device, one_mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
//	    		MNIST_drive(old_device_neurons, one_mnist_img, network_size,training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
//	    		training_img_index ++;
//	    		//confirm the data in signal neuron
//	    		//hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
//	    		//data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 3);
//	    		//printf("\n\n\n************************\n\n\n\n");
//	    	}
//	    	spiking_learning_main(Neuron_list_device, old_device_neurons, random_number_list_device, network_size, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, time);
//
//	    	synapse_drive_v1(Neuron_list_device, network_size, syn_timer_max, connection_size, random_number_list_device, input_float, input_float_2);
//	    	if(HOMEOSTASIS_ENABLE){
//				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
//					spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
//				}
//	    	}
//	        hipDeviceSynchronize();
//
//	        //if any neuron spikes, run inhibition
//		    hipMemcpy(spike_flag, spike_flag_device, sizeof(int),hipMemcpyDeviceToHost);
//		    //printf("AtTime:%d_spike_flag_is:%d\n",time,spike_flag[0]);
//	        if(spike_flag[0]>0){//use lateral inhibition
//	        	//printf("inInhibit\n");
//	        	spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 0);
//	        	spike_flag[0] = 0;
//
//	    	    hipMemcpy(spike_flag_device,spike_flag,sizeof(int),hipMemcpyHostToDevice);
//	        }
//
//	        hipMemcpy(old_device_neurons,Neuron_list_device,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
//	        //hipDeviceSynchronize();
//	    	time ++;
//	    }
//	    //spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, 2);
//		//hipDeviceSynchronize();
//	    hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
//	    hipMemcpy(log_v_host,log_v,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
//	    hipMemcpy(log_spike_host,log_spike,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
//	    hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost);
//
//	    //print out the synapse conductance data
//	    data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, "");
//
//	    ofstream myfile ((index_prefix+"device2_spike_of_neuron_out.csv"));
//	    if (myfile.is_open()){
//	    	//myfile << "This is a new test\n";
//	    	for(int i=0; i < SIZE; i++){
//	    		//printf("_%f_", log_v_host[i]);
//	    		myfile << log_total_spike_host[i] << ", ";
//	    	}
//	    	myfile.close();
//	    }
//
//	    ofstream myfile_0 ((index_prefix+"device2_out_v.csv"));
//	    if (myfile_0.is_open()){
//	    	//myfile << "This is a new test\n";
//	    	for(int i=0; i < MAX_TIME; i++){
//	    		//printf("_%f_", log_v_host[i]);
//	    		myfile_0 << log_v_host[i] << ", ";
//	    	}
//	    	myfile.close();
//	    }
//
//	    ofstream myfile_2 ((index_prefix+"device2_spike_of_one.csv"));
//	    if (myfile_2.is_open()){
//	    	//myfile << "This is a new test\n";
//	    	for(int i=0; i < MAX_TIME; i++){
//	    		//printf("_%f_", log_v_host[i]);
//	    		myfile_2 << log_spike_host[i] << ", ";
//	    	}
//	    	myfile_2.close();
//	    }
//
//	    write_neuron_list(NeuronList, (index_prefix+"device2_output_network.txt"), network_size);
//
//	    //===clean up===
//	    //delete[] random_number_list;
//	    delete[] log_v_host;
//		delete[] NeuronList;
//		delete[] log_spike_host;
//
//
//		//hipFree(states);
//		hipFree(log_v);
//		hipFree(log_spike);
//		hipFree(log_total_spike);
//		hipFree(Neuron_list_device);
//		hipFree(old_device_neurons);
//		hipFree(random_number_list_device);
//}

//void last_layer_learn(string index_prefix, float input_float, float input_float_2, int input_int, int input_int_2, string input_img){
//	/*
//	int training_set_number = 1;
//	int size_per_img = input_image_w * input_image_l*input_image_channel;
//	float *mnist_img = new float[size_per_img*training_set_number];
//	string image_file = "train-images-idx3-ubyte"; //"train-images-idx3-ubyte";
//	MNIST_read_image(image_file, mnist_img, training_set_number);
//	int *mnist_label = new int[training_set_number];
//	string image_label_file = "train-labels-idx1-ubyte";
//	MNIST_read_label(image_label_file, mnist_label, training_set_number);
//
//	float *filter;
//	float *output = new float[size_per_img*training_set_number];
//
//	convolution_kernel(mnist_img, filter, output);
//	img_util(output, "test_output.jpg", 0);
//	*/
//
//
//	CNN_struct *network_config = new CNN_struct;
//	network_config_generator(3, network_config);
//	Neuron *NeuronList_temp = new Neuron[1];
//	CNN_struct *d_network_config;
//	hipMalloc((void **)&d_network_config,sizeof(CNN_struct));
//	hipMemcpy(d_network_config,network_config,sizeof(CNN_struct),hipMemcpyHostToDevice);
//	int total_depth_number = 0;
//	for(int i=0;i<CNN_total_layer_num; i++){
//		total_depth_number = total_depth_number + network_config->layer[i].depth;
//		cout<<"depth number: "<<network_config->layer[i].depth<<endl;
//	}
//
//	cout<<endl<<"Total depth number: "<<total_depth_number<<endl;
//	float **h_filter_array;
//	float **d_filter_array;
//	int filter_array_size = CNN_total_layer_num-1;
//	hipMalloc(&d_filter_array, filter_array_size*sizeof(float *));
//	h_filter_array = (float**)malloc(filter_array_size * sizeof(float*));
//	filter_util(network_config, NeuronList_temp, SIZE, h_filter_array, d_filter_array, 0);
//
//	/*
//	img_util(mnist_img, "tensorflow_small.png", 1);
//	img_util(mnist_img, "test_output_-1.png", 0);
//
//	float *output = new float[size_per_img*training_set_number];
//
//	int image_bytes = input_image_channel * input_image_l * input_image_w * sizeof(float);
//
//	float* convolution_device_input{nullptr};
//	hipMalloc(&convolution_device_input, image_bytes);
//	hipMemcpy(convolution_device_input, mnist_img, image_bytes, hipMemcpyHostToDevice);
//
//	int filter_in_channel = input_image_channel;
//	int filter_out_channel = input_image_channel;
//	int filter_height = 3;
//	int filter_width = 3;
//	const float kernel_template[3][3] = {
//	{1, 1, 1},
//	{1, -8, 1},
//	{1, 1, 1}
//	};
//	float h_kernel[filter_in_channel][filter_out_channel][filter_height][filter_width];
//	for (int kernel = 0; kernel < filter_in_channel; ++kernel) {
//		for (int channel = 0; channel < filter_out_channel; ++channel) {
//		  for (int row = 0; row < filter_height; ++row) {
//			for (int column = 0; column < filter_width; ++column) {
//			  h_kernel[kernel][channel][row][column] = kernel_template[row][column];
//			}
//		  }
//		}
//	}
//	float* filter{nullptr};
//	hipMalloc(&filter, sizeof(h_kernel));
//	hipMemcpy(filter, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);
//
//
//	convolution_kernel(convolution_device_input, filter, output);
//	img_util(output, "test_output_1.png", 0);
//
//	hipFree(filter);
//	hipFree(convolution_device_input);
//
//	*/
//
////===========END of CNN special setting-up phase============
//
//	hipSetDevice(0);
//	//set parameters
//
//	int training_time_each_img = input_int;
//	int calculated_total_time = training_time_each_img*50000;
//	#undef MAX_TIME
//	#define MAX_TIME calculated_total_time
//	printf("==Training Total Iter: %d==", MAX_TIME);
//	int total_neuron_num = 0;
//	int total_spiking_num = 0;
//	for(int i=0;i<CNN_total_layer_num;i++){
//		total_neuron_num += network_config->layer[i].neuron_num;
//		if(i!=0)
//		total_spiking_num += network_config->layer[i].neuron_num;
//	}
//	total_neuron_num += 100;
//	//total_neuron_num = 20000;
//	cout<<endl<<"total neuron num: "<<total_neuron_num<<endl;
//	cout<<"total spiking neuron num: "<<total_spiking_num<<endl;
//	#undef SIZE
//	#define SIZE total_neuron_num
//	#undef SPIKING_NEURON_NUM
//	#define SPIKING_NEURON_NUM total_spiking_num
//
//
//	float max_frequency = 22; //in Hz default 22
//	float min_frequency = 1;
//	int training_set_number = 55000;
//	int input_neuron_num = input_image_w*input_image_l*input_image_channel;
//	int spiking_neuron_num = SPIKING_NEURON_NUM;
//	int output_layer_neuron_num = OUTPUT_LAYER_NEURON_NUM;
//	int tenpercent_iter = MAX_TIME/10;
//	int connection_size = MAX_CONNECTION;
//	int syn_timer_max = 25;
//	int input_signal_width = 25;	//default 25
//	int inhibition_time = input_int_2;	//default 10
//
//	float target_frequency_param = 0.5;
//	float target_frequency = 100;
//	float *mnist_img = new float[input_neuron_num*training_set_number];
//	for(int i=0;i<input_neuron_num*training_set_number;i++) mnist_img[i] = 0;
//	string image_file = "train-images-idx3-ubyte";//"fashion-train-images-idx3-ubyte";//"train_dataset_noisy";//"train_dataset_noisy"; //"train-images-idx3-ubyte";
//	read_filter_data(image_file, mnist_img, training_set_number, input_neuron_num);
//	int *mnist_label = new int[training_set_number];
//	string image_label_file = "train-labels-idx1-ubyte";
//	MNIST_read_label(image_label_file, mnist_label, training_set_number);
//	//special_function: learn one category
//	int learn_one_digit = 0;
//	int *num_one_digit_img = new int[1];
//	if(learn_one_digit){
//		MNIST_labeling("abc", 60000, mnist_img, mnist_label, mnist_img, num_one_digit_img, spiking_neuron_num, 1, 5);
//		printf("Learning only one digit, number of img: %d\n", num_one_digit_img);
//	}
//
//	//int synapse_size = SIZE*SIZE;
//	//cout<<SIZE<<endl;
//    Neuron *NeuronList = new Neuron[SIZE];
//	//unsigned char *synapse_timer = new unsigned char[synapse_size];  //this is the array that stores timer used in STPD. e.g Neuron x --->  Neuron y Spike! In the array index [(x-1)*SIZE+(y-1)]  => 1
//	//hiprandState_t *states;
//	//float *random_number_list = new float[SIZE];
//	float *log_v_host = new float[MAX_TIME];
//	float *log_spike_host = new float[total_depth_number];
//
//	float *log_total_spike_host = new float[SIZE];
//	for(int i=0; i < SIZE; i++){
//		log_total_spike_host[i] = 0;
//	}
//	int *spike_flag = new int[CNN_total_layer_num];
//	for(int i=0; i < CNN_total_layer_num; i++){
//		spike_flag[i] = 0;
//	}
//	for(int i=0; i<total_depth_number; i++) log_spike_host[i] = 0;
//	//init_log_v(log_v_host);
//	//init_data_log(log_v_host,log_spike_host,log_total_spike_host, MAX_TIME);
//	neuron_list_init(NeuronList, total_neuron_num);
//	//printf("=0=\n");
//	read_neuron_list(NeuronList, 1, "spike_cnn.txt");
//    //write_neuron_list(NeuronList, "learning_output_confirm.txt", SIZE);
//	//check_neuron(NeuronList, 800, 820);
//
//	Neuron *Neuron_list_device;
//	//Neuron *old_device_neurons;
//	//unsigned char *snapse_timer_device;
//	float *log_v;
//	float *log_spike;
//	float *log_spike_default;
//	float *log_total_spike;
//	int *spike_flag_device;
//
//
//    printf("2\n");
//	//printf("=1=\n");
//	//random number function:
//    float rand_list_size_to_total_connection_ratio = 1;
//	int rand_numb_size = SPIKING_NEURON_NUM*MAX_CONNECTION;
//	hiprandState_t *states;
//	hipMalloc((void **)&states, rand_numb_size * sizeof(hiprandState_t));
//	int SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
//    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
//    dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
//	dim3 print_grid(1);
//	dim3 print_block(1);
//    printf("2.1\n");
//	rand_init<<<dimGrid,dimBlock>>>(time(0), rand_numb_size, states);
//
//	float *random_number_list = new float[rand_numb_size];
//	float *random_number_list_device;
//	SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
//	dim3 dimBlock_synapse( ThreadsPerBlock, ThreadsPerBlock );
//	dim3 dimGrid_synapse( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
//	hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
//	hipMemcpy(random_number_list_device,random_number_list,rand_numb_size*sizeof(float),hipMemcpyHostToDevice);
//
//    random<<<dimGrid_synapse,dimBlock_synapse>>>(random_number_list_device, rand_numb_size, states);
//    printf("2.11\n");
//    //Setting up input instance matrix:
//    float **d_input_instance;
//    float **d_convolution_result;
//    float **h_input_instance;
//    float **h_convolution_result;
//    float *probe = new float[1000];
//	int instance_array_size = CNN_total_layer_num;
//	hipMalloc(&d_input_instance, instance_array_size*sizeof(float *));
//	int convolution_result_size = CNN_total_layer_num - 1;
//	hipMalloc(&d_convolution_result, convolution_result_size*sizeof(float *));
//    h_input_instance = (float**)malloc(instance_array_size * sizeof(float*));
//    h_convolution_result = (float**)malloc(convolution_result_size * sizeof(float*));
//    CNN_util(network_config, d_input_instance, d_convolution_result, h_input_instance, h_convolution_result, 0);
//
////	float **add = &h_convolution_result[0];
////	printf("Address On GPU: %p\n", add);
//
//    //Setting up others
//    hipMalloc((void **)&Neuron_list_device, SIZE*sizeof(Neuron));
//    //hipMalloc((void **)&old_device_neurons, SIZE*sizeof(Neuron));
//    printf("2.2\n");
//    //hipMalloc((void **)&states, SIZE * sizeof(hiprandState_t));
//    hipMalloc((void **)&log_v, MAX_TIME * sizeof(float));
//    hipMalloc((void **)&log_spike, total_depth_number * sizeof(float));
//    hipMalloc((void **)&log_spike_default, total_depth_number * sizeof(float));
//    //hipMalloc((void **)&log_total_spike, SIZE * sizeof(float));
//    gpuErrchk( hipMalloc((void **)&log_total_spike, SIZE * sizeof(float)) );
//    hipMalloc((void **)&spike_flag_device, instance_array_size*sizeof(int));
//    //rand_init<<<dimGrid,dimBlock>>>(time(0), states);
//
//    hipMemcpy(Neuron_list_device,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
//    //hipMemcpy(old_device_neurons,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
//    //hipMemcpy(random_number_list_device, random_number_list, SIZE*sizeof(float), hipMemcpyHostToDevice);
//    hipMemcpy(log_v,log_v_host,MAX_TIME*sizeof(float),hipMemcpyHostToDevice);
//    hipMemcpy(log_spike,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
//    hipMemcpy(log_spike_default,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
//    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
//    hipMemcpy(spike_flag_device,spike_flag,instance_array_size*sizeof(int),hipMemcpyHostToDevice);
//    printf("3\n");
//    //cout<<"network size: "<<SIZE<<endl;
//    int network_size = SIZE;
//
//    int max_time = MAX_TIME;
//
//
//    //hipMemcpy(Neuron_list_device,old_device_neurons,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
//    //first change raw img data into frequency
//    int mnist_start_index = 0;
//    int mnist_end_index = input_neuron_num;
//    //change pixel signal to frequency
//
//    MNIST_drive(NeuronList, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 1);
//
//
//    hipDeviceSynchronize();
//
//    //data_check(Neuron_list_device,log_total_spike,SIZE,1);
//    float *one_mnist_img = new float[input_neuron_num];
//
//    clock_t iter_start, iter_log;
//    iter_start = clock();
//    int log_interval = MAX_TIME/10;
//    //read_filter_GPU_one_layer<<<1, 1>>>(d_network_config, h_filter_array[0], 1);
//    //read_filter_GPU<<<1, 1>>>(d_network_config, d_filter_array);
//
//    int reiter_run = 1;
//
//    int time = 0;
//    int training_img_index = 0;
//    while (time<max_time){
//    	//cout<<endl<<" It: "<<time<<endl;
//        //random<<<dimGrid,dimBlock>>>(random_number_list_device, states);
//    	//first create an array of 1 MNIST image
////    	if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING){
////            random<<<dimGrid_synapse,dimBlock_synapse>>>(random_number_list_device, rand_numb_size, states);
////    	}
//    	if(time%log_interval == 0){
//    		hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
//    		string interval_file_name = "device2_output_at_iter_" + to_string(time) + ".txt";
//    		//write_neuron_list(NeuronList, interval_file_name, network_size);
//    		//printf("%");
//    	}
//
//    	if(time%tenpercent_iter == 0){
//    		iter_log = clock();
//    		cout<<to_string(10*(time/tenpercent_iter))<<"% done, time used is: " << (iter_log - iter_start)/1000 << " (ms)" << endl;
//    	}
//    	//fault below here:
//
//    	if(time%training_time_each_img==0){//at the beginning of each img's training, load into
//    		//cout<<"Image Load Iter: "<<time<<endl;
//    		for(int i=0;i<input_neuron_num;i++){
//    			one_mnist_img[i] = mnist_img[training_img_index*input_neuron_num+i];
//    		}
////    	    for (int y=0; y<28; ++y) {
////    	    	    for (int x=0; x<28; ++x) {
////    	    	      std::cout << ((one_mnist_img[y*28+x] <= 1.1)? ' ' : '*');
////    	    	      //std::cout << int(one_mnist_img[y*28+x]) << ' ';
////    	    	    }
////    	    	    std::cout << std::endl;
////    	    }
//    		MNIST_drive(Neuron_list_device, one_mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
//    		//MNIST_drive(old_device_neurons, one_mnist_img, network_size,training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
//    		training_img_index ++;
//    		//confirm the data in signal neuron
//    		//hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
//    		//data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 3);danshe.2010@gamil.com
//    		//printf("\n\n\n************************\n\n\n\n");
//    	}
//    	//cout<<"One IMG loaded"<<endl;
//    	//enter spiking neuron simulation:
//
//
//    	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
//    		int convolution_result_index = layer_iter - 1;
//    		if (layer_iter==0) {//fault at convolution kernel and spiking cnn
//    			convolution_result_index = 0;
//    	    	//CNN_struct *settings; int layer_index; float **d_input_2d; float **filter_2d; float **output_2d;
//    	    	//convolution_kernel(settings, layer_index, d_input_2d, filter_2d, output_2d);
//    	    	//problem is in spiking_cnn_main
//    			spiking_cnn_main(Neuron_list_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, layer_iter, network_size, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time);
//    			//spiking_cnn_main(Neuron_list_device, old_device_neurons, d_network_config, random_number_list_device, d_convolution_result[convolution_result_index], d_input_instance[layer_iter], layer_iter, network_size, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, time);
//    			//spiking_cnn_main(Neuron_list_device, old_device_neurons, d_network_config, random_number_list_device, d_convolution_result[0], d_input_instance[0], layer_iter, network_size, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, time);
//    			convolution_kernel(network_config, layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
//    		}else{
//    			//printf("In layer: %d\n", layer_iter);
//    			spiking_cnn_main(Neuron_list_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, layer_iter, network_size, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time);
//    			//network_config->layer[layer_iter].depth;
//    			if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(network_config, layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
//				synapse_drive_cnn_v2(Neuron_list_device, network_config, d_network_config, d_filter_array, layer_iter, network_size, syn_timer_max, connection_size, random_number_list_device, states, -1.0, -1.0);//STDP
//    		}
//
//    	}
//    	//=================TRY WITH LAYER wise inhibition=====================
//    	hipMemcpy(spike_flag, spike_flag_device, CNN_total_layer_num*sizeof(int),hipMemcpyDeviceToHost);
//    	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
//    		//if (layer_iter==0) printf("Layer[%d] SpikeFlag: %d\n", layer_iter, spike_flag[layer_iter]);
//            if(spike_flag[layer_iter]>0){//use lateral inhibition
//            	spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, layer_iter, network_config, 4);
//            }
//    		spike_flag[layer_iter] = 0;
//    	}
//    	hipMemcpy(spike_flag_device,spike_flag,CNN_total_layer_num*sizeof(int),hipMemcpyHostToDevice);
//	//=================TRY WITH NO LAYERAL INHIBITION, MAY BE WRONG=====================
//    	//printf("network_size: %d", network_size);
//    	//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 3); //lateral inhibition
//	//==================================================================================
//		if(HOMEOSTASIS_ENABLE){
//			if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
//				//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
//			}
//		}
//        hipDeviceSynchronize();
//	hipMemcpy(log_spike,log_spike_default,total_depth_number*sizeof(float),hipMemcpyDeviceToDevice);	//set the log_spike to default value
//
//        //if any neuron spikes, run inhibition
//	//hipMemcpy(spike_flag, spike_flag_device, CNN_total_layer_num*sizeof(int),hipMemcpyDeviceToHost);
//	//printf("AtTime:%d_spike_flag_is:%d\n",time,spike_flag[0]);
//        //if(spike_flag[0]>0){//use lateral inhibition
//        	//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0);
//        	//spike_flag[0] = 0;
//    	    	//hipMemcpy(spike_flag_device,spike_flag,sizeof(int),hipMemcpyHostToDevice);
//        //}
//
//        //hipMemcpy(old_device_neurons,Neuron_list_device,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
//        //hipDeviceSynchronize();
//    	time ++;
//    }
//    //spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, 2);
//	//hipDeviceSynchronize();
//
//	filter_util(network_config, Neuron_list_device, network_size, h_filter_array, d_filter_array, 2);
//    hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
//    hipMemcpy(log_v_host,log_v,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
//    hipMemcpy(log_spike_host,log_spike,total_depth_number*sizeof(float),hipMemcpyDeviceToHost);
//    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );
//
//
//    //print out the synapse conductance data
//    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2);
//
//    ofstream myfile ((index_prefix+"device2_spike_of_neuron_out.csv"));
//    if (myfile.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < SIZE; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile << log_total_spike_host[i] << ", ";
//    	}
//    	myfile.close();
//    }
//
//    ofstream myfile_p ((index_prefix+"probe.csv"));
//    if (myfile_p.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < 1000; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_p << probe[i] << ", ";
//    	}
//    	myfile_p.close();
//    }
//
////
////    ofstream myfile_0 ((index_prefix+"device2_out_v.csv"));
////    if (myfile_0.is_open()){
////    	//myfile << "This is a new test\n";
////    	for(int i=0; i < MAX_TIME; i++){
////    		//printf("_%f_", log_v_host[i]);
////    		myfile_0 << log_v_host[i] << ", ";
////    	}
////    	myfile.close();
////    }
////
////    ofstream myfile_2 ((index_prefix+"device2_spike_of_one.csv"));
////    if (myfile_2.is_open()){
////    	//myfile << "This is a new test\n";
////    	for(int i=0; i < MAX_TIME; i++){
////    		//printf("_%f_", log_v_host[i]);
////    		myfile_2 << log_spike_host[i] << ", ";
////    	}
////    	myfile_2.close();
////    }
//
//    data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, "");
//
//    hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
//	filter_util(network_config, NeuronList, network_size, h_filter_array, d_filter_array, 1);	//write filter to file
//    write_neuron_list(NeuronList, (index_prefix+"device2_output_network.txt"), network_size);
//
//    //===clean up===
//    //delete[] random_number_list;
//    delete[] log_v_host;
//	delete[] NeuronList;
//	delete[] log_spike_host;
//	delete[] log_total_spike_host;
//	delete[] mnist_img;
//	delete[] NeuronList_temp;
//	delete[] one_mnist_img;
//	delete[] probe;
//	delete[] random_number_list;
//	delete[] mnist_label;
//	delete[] spike_flag;
//	delete[] num_one_digit_img;
//	//hipFree(states);
//	hipFree(log_v);
//	hipFree(log_spike);
//	hipFree(log_total_spike);
//	hipFree(Neuron_list_device);
//	//hipFree(old_device_neurons);
//	hipFree(random_number_list_device);
//	hipFree(d_network_config);
//	hipFree(states);
//	hipFree(spike_flag_device);
//	hipFree(log_spike_default);
//
//}
//
//
//void space_transfer(string index_prefix, float input_float, float input_float_2, int input_int, int input_int_2, string input_img){
//	/*
//	int training_set_number = 1;
//	int size_per_img = input_image_w * input_image_l*input_image_channel;
//	float *mnist_img = new float[size_per_img*training_set_number];
//	string image_file = "train-images-idx3-ubyte"; //"train-images-idx3-ubyte";
//	MNIST_read_image(image_file, mnist_img, training_set_number);
//	int *mnist_label = new int[training_set_number];
//	string image_label_file = "train-labels-idx1-ubyte";
//	MNIST_read_label(image_label_file, mnist_label, training_set_number);
//
//	float *filter;
//	float *output = new float[size_per_img*training_set_number];
//
//	convolution_kernel(mnist_img, filter, output);
//	img_util(output, "test_output.jpg", 0);
//	*/
//	int resume_learning = 0;
//	CNN_struct *network_config = new CNN_struct;
//	network_config_generator(3, network_config);
//	Neuron *NeuronList_temp = new Neuron[1];
//	CNN_struct *d_network_config;
//	hipMalloc((void **)&d_network_config,sizeof(CNN_struct));
//	hipMemcpy(d_network_config,network_config,sizeof(CNN_struct),hipMemcpyHostToDevice);
//	int total_depth_number = 0;
//	for(int i=0;i<CNN_total_layer_num; i++){
//		total_depth_number = total_depth_number + network_config->layer[i].depth;
//		cout<<"depth number: "<<network_config->layer[i].depth<<endl;
//	}
//
//	cout<<endl<<"Total depth number: "<<total_depth_number<<endl;
//	float **h_filter_array;
//	float **d_filter_array;
//	int filter_array_size = CNN_total_layer_num-1;
//	hipMalloc(&d_filter_array, filter_array_size*sizeof(float *));
//	h_filter_array = (float**)malloc(filter_array_size * sizeof(float*));
//	filter_util(network_config, NeuronList_temp, 0, h_filter_array, d_filter_array, 0);
//
//	/*
//	img_util(mnist_img, "tensorflow_small.png", 1);
//	img_util(mnist_img, "test_output_-1.png", 0);
//
//	float *output = new float[size_per_img*training_set_number];
//
//	int image_bytes = input_image_channel * input_image_l * input_image_w * sizeof(float);
//
//	float* convolution_device_input{nullptr};
//	hipMalloc(&convolution_device_input, image_bytes);
//	hipMemcpy(convolution_device_input, mnist_img, image_bytes, hipMemcpyHostToDevice);
//
//	int filter_in_channel = input_image_channel;
//	int filter_out_channel = input_image_channel;
//	int filter_height = 3;
//	int filter_width = 3;
//	const float kernel_template[3][3] = {
//	{1, 1, 1},
//	{1, -8, 1},
//	{1, 1, 1}
//	};
//	float h_kernel[filter_in_channel][filter_out_channel][filter_height][filter_width];
//	for (int kernel = 0; kernel < filter_in_channel; ++kernel) {
//		for (int channel = 0; channel < filter_out_channel; ++channel) {
//		  for (int row = 0; row < filter_height; ++row) {
//			for (int column = 0; column < filter_width; ++column) {
//			  h_kernel[kernel][channel][row][column] = kernel_template[row][column];
//			}
//		  }
//		}
//	}
//	float* filter{nullptr};
//	hipMalloc(&filter, sizeof(h_kernel));
//	hipMemcpy(filter, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);
//
//
//	convolution_kernel(convolution_device_input, filter, output);
//	img_util(output, "test_output_1.png", 0);
//
//	hipFree(filter);
//	hipFree(convolution_device_input);
//
//	*/
//
////===========END of CNN special setting-up phase============
//
//
//	//set parameters
//
//	int training_time_each_img = input_int;
//	int calculated_total_time = training_time_each_img*1;
//	#undef MAX_TIME
//	#define MAX_TIME calculated_total_time
//	printf("==Training Total Iter: %d==", MAX_TIME);
//	int total_neuron_num = 0;
//	int total_spiking_num = 0;
//	for(int i=0;i<CNN_total_layer_num;i++){
//		total_neuron_num += network_config->layer[i].neuron_num;
//		if(i!=0)
//		total_spiking_num += network_config->layer[i].neuron_num;
//	}
//	total_neuron_num += 100;
//	//total_neuron_num = 20000;
//	cout<<endl<<"total neuron num: "<<total_neuron_num<<endl;
//	cout<<"total spiking neuron num: "<<total_spiking_num<<endl;
//	#undef SIZE
//	#define SIZE total_neuron_num
//	#undef SPIKING_NEURON_NUM
//	#define SPIKING_NEURON_NUM total_spiking_num
//
//
//	float max_frequency = 22; //in Hz default 22
//	float min_frequency = 1;
//	int training_set_number = 50000;
//	int input_neuron_num = input_image_w*input_image_l*input_image_channel;
//	int input_image_signal_channel_size  = input_image_w*input_image_l;
//	int spiking_neuron_num = SPIKING_NEURON_NUM;
//	int output_layer_neuron_num = OUTPUT_LAYER_NEURON_NUM;
//	int tenpercent_iter = MAX_TIME/10;
//	int connection_size = MAX_CONNECTION;
//	int syn_timer_max = 25;
//	int input_signal_width = 15;	//default 25
//	int inhibition_time = 3;	//default 10
//
//	float target_frequency_param = 0.5;
//	float target_frequency = 100;
//	float *mnist_img = new float[input_neuron_num*training_set_number];
//	for(int i=0;i<input_neuron_num*training_set_number;i++) mnist_img[i] = 0;
//	string image_file = "train_dataset_noisy_cifar";//"fashion-train-images-idx3-ubyte";//"train_dataset_noisy";//"train_dataset_noisy"; //"train-images-idx3-ubyte";
//	if(input_image_channel==1){
//		CIFAR_read_image(mnist_img, input_neuron_num, 0, 1);
//	}else{
//		CIFAR_read_image(mnist_img, input_neuron_num, 0, 0);
//	}
//	//CIFAR_read_image_one_channel(mnist_img, input_image_signal_channel_size, input_int_2, 0);
//	//MNIST_read_image(image_file, mnist_img, training_set_number);
//	int *mnist_label = new int[training_set_number];
//	string image_label_file = "train-labels-idx1-ubyte";
//	CIFAR_read_label(mnist_label, 0);
//	//MNIST_read_label(image_label_file, mnist_label, training_set_number);
//	//special_function: learn one category
//	int learn_one_digit = 0;
//	int *num_one_digit_img = new int[1];
//	if(learn_one_digit){
//		MNIST_labeling("abc", 60000, mnist_img, mnist_label, mnist_img, num_one_digit_img, spiking_neuron_num, 1, 5);
//		printf("Learning only one digit, number of img: %d\n", num_one_digit_img);
//	}
//
//	//int synapse_size = SIZE*SIZE;
//	//cout<<SIZE<<endl;
//    Neuron *NeuronList = new Neuron[SIZE];
//	//unsigned char *synapse_timer = new unsigned char[synapse_size];  //this is the array that stores timer used in STPD. e.g Neuron x --->  Neuron y Spike! In the array index [(x-1)*SIZE+(y-1)]  => 1
//	//hiprandState_t *states;
//	//float *random_number_list = new float[SIZE];
//	float *log_v_host = new float[MAX_TIME];
//	float *log_spike_host = new float[total_depth_number];
//
//	float *log_total_spike_host = new float[SIZE];
//	for(int i=0; i < SIZE; i++){
//		log_total_spike_host[i] = 0;
//	}
//	int *spike_flag = new int[CNN_total_layer_num];
//	for(int i=0; i < CNN_total_layer_num; i++){
//		spike_flag[i] = 0;
//	}
//	for(int i=0; i<total_depth_number; i++) log_spike_host[i] = 0;
//	//init_log_v(log_v_host);
//	//init_data_log(log_v_host,log_spike_host,log_total_spike_host, MAX_TIME);
//	neuron_list_init(NeuronList, total_neuron_num);
//	//printf("=0=\n");
//	//
//	if(resume_learning){
//		printf("RESUME LEARNING\n");
//		read_neuron_list(NeuronList, 1, "device2_output_network.txt");
//	}else{
//		read_neuron_list(NeuronList, 1, "spike_cnn.txt");
//	}
//    //write_neuron_list(NeuronList, "learning_output_confirm.txt", SIZE);
//	//check_neuron(NeuronList, 800, 820);
//
//	Neuron *Neuron_list_device;
//	//Neuron *old_device_neurons;
//	//unsigned char *snapse_timer_device;
//	float *log_v;
//	float *log_spike;
//	float *log_spike_default;
//	float *log_total_spike;
//	int *spike_flag_device;
//
//
////    printf("2\n");
//	//printf("=1=\n");
//	//random number function:
//    float rand_list_size_to_total_connection_ratio = 1;
//	int rand_numb_size = SPIKING_NEURON_NUM*MAX_CONNECTION;
//	hiprandState_t *states;
//	hipMalloc((void **)&states, rand_numb_size * sizeof(hiprandState_t));
//	int SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
//    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
//    dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
//	dim3 print_grid(1);
//	dim3 print_block(1);
////    printf("2.1\n");
//	rand_init<<<dimGrid,dimBlock>>>(time(0), rand_numb_size, states);
//
//	float *random_number_list = new float[rand_numb_size];
//	float *random_number_list_device;
//	SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
//	dim3 dimBlock_synapse( ThreadsPerBlock, ThreadsPerBlock );
//	dim3 dimGrid_synapse( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
//	hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
//	hipMemcpy(random_number_list_device,random_number_list,rand_numb_size*sizeof(float),hipMemcpyHostToDevice);
//
//    random<<<dimGrid_synapse,dimBlock_synapse>>>(random_number_list_device, rand_numb_size, states);
////    printf("2.11\n");
//    //Setting up input instance matrix:
//    float **d_input_instance;
//    float **d_convolution_result;
//    float **h_input_instance;
//    float **h_convolution_result;
//    float *probe = new float[1000];
//	int instance_array_size = CNN_total_layer_num;
//	hipMalloc(&d_input_instance, instance_array_size*sizeof(float *));
//	int convolution_result_size = CNN_total_layer_num - 1;
//	hipMalloc(&d_convolution_result, convolution_result_size*sizeof(float *));
//    h_input_instance = (float**)malloc(instance_array_size * sizeof(float*));
//    h_convolution_result = (float**)malloc(convolution_result_size * sizeof(float*));
//    CNN_util(network_config, d_input_instance, d_convolution_result, h_input_instance, h_convolution_result, 0);
//
////	float **add = &h_convolution_result[0];
////	printf("Address On GPU: %p\n", add);
//
//    //Setting up others
//    hipMalloc((void **)&Neuron_list_device, SIZE*sizeof(Neuron));
//    //hipMalloc((void **)&old_device_neurons, SIZE*sizeof(Neuron));
////    printf("2.2\n");
//    //hipMalloc((void **)&states, SIZE * sizeof(hiprandState_t));
//    hipMalloc((void **)&log_v, MAX_TIME * sizeof(float));
//    hipMalloc((void **)&log_spike, total_depth_number * sizeof(float));
//    hipMalloc((void **)&log_spike_default, total_depth_number * sizeof(float));
//    //hipMalloc((void **)&log_total_spike, SIZE * sizeof(float));
//    gpuErrchk( hipMalloc((void **)&log_total_spike, SIZE * sizeof(float)) );
//    hipMalloc((void **)&spike_flag_device, instance_array_size*sizeof(int));
//    //rand_init<<<dimGrid,dimBlock>>>(time(0), states);
//
//    hipMemcpy(Neuron_list_device,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
//    //hipMemcpy(old_device_neurons,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
//    //hipMemcpy(random_number_list_device, random_number_list, SIZE*sizeof(float), hipMemcpyHostToDevice);
//    hipMemcpy(log_v,log_v_host,MAX_TIME*sizeof(float),hipMemcpyHostToDevice);
//    hipMemcpy(log_spike,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
//    hipMemcpy(log_spike_default,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
//    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
//    hipMemcpy(spike_flag_device,spike_flag,instance_array_size*sizeof(int),hipMemcpyHostToDevice);
//    printf("3\n");
//    //cout<<"network size: "<<SIZE<<endl;
//    int network_size = SIZE;
//
//    int max_time = MAX_TIME;
//
//
//    //hipMemcpy(Neuron_list_device,old_device_neurons,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
//    //first change raw img data into frequency
//    int mnist_start_index = 0;
//    int mnist_end_index = input_neuron_num;
//    //change pixel signal to frequency
//
//    MNIST_drive(NeuronList, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 1);
//
//
//    hipDeviceSynchronize();
//
//    //data_check(Neuron_list_device,log_total_spike,SIZE,1);
//    float *one_mnist_img = new float[input_neuron_num];
//
//    clock_t iter_start, iter_log;
//    iter_start = clock();
//    int log_interval = MAX_TIME/10;
//    //read_filter_GPU_one_layer<<<1, 1>>>(d_network_config, h_filter_array[0], 1);
//    //read_filter_GPU<<<1, 1>>>(d_network_config, d_filter_array);
//
//    int reiter_run = 1;
//
//    int time = 0;
//    int training_img_index = 0;
//    while (time<max_time){
//    	//cout<<endl<<" It: "<<time<<endl;
//        //random<<<dimGrid,dimBlock>>>(random_number_list_device, states);
//    	//first create an array of 1 MNIST image
////    	if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING){
////            random<<<dimGrid_synapse,dimBlock_synapse>>>(random_number_list_device, rand_numb_size, states);
////    	}
//    	if(time%log_interval == 0){
//    		hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
//    		string interval_file_name = "device2_output_at_iter_" + to_string(time) + ".txt";
//    		write_neuron_list(NeuronList, interval_file_name, network_size);
//
//    		//printf("%");
//    	}
//
//    	if(time%tenpercent_iter == 0){
//    		iter_log = clock();
//    		cout<<to_string(10*(time/tenpercent_iter))<<"% done, time used is: " << (iter_log - iter_start)/1000 << " (ms)" << endl;
//    	}
//    	//fault below here:
//
//    	if(time%training_time_each_img==0){//at the beginning of each img's training, load into
//    		//cout<<"Image Load Iter: "<<time<<endl;
//    		for(int i=0;i<input_neuron_num;i++){
//    			one_mnist_img[i] = mnist_img[training_img_index*input_neuron_num+i];
//    		}
////    	    for (int y=0; y<28; ++y) {
////    	    	    for (int x=0; x<28; ++x) {
////    	    	      std::cout << ((one_mnist_img[y*28+x] <= 1.1)? ' ' : '*');
////    	    	      //std::cout << int(one_mnist_img[y*28+x]) << ' ';
////    	    	    }
////    	    	    std::cout << std::endl;
////    	    }
//    		MNIST_drive(Neuron_list_device, one_mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
//    		//MNIST_drive(old_device_neurons, one_mnist_img, network_size,training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
//    		training_img_index ++;
//    		if(training_img_index>=49999) training_img_index = 0;
//    		//confirm the data in signal neuron
//    		//hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
//    		//data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 3);
//    		//printf("\n\n\n************************\n\n\n\n");
//    	}
//    	//cout<<"One IMG loaded"<<endl;
//    	//enter spiking neuron simulation:
//
//
//    	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
//    		int convolution_result_index = layer_iter - 1;
//    		if (layer_iter==0) {//fault at convolution kernel and spiking cnn
//    			convolution_result_index = 0;
//    	    	//CNN_struct *settings; int layer_index; float **d_input_2d; float **filter_2d; float **output_2d;
//    	    	//convolution_kernel(settings, layer_index, d_input_2d, filter_2d, output_2d);
//    	    	//problem is in spiking_cnn_main
//    			spiking_cnn_main(Neuron_list_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, layer_iter, network_size, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time);
//    			//spiking_cnn_main(Neuron_list_device, old_device_neurons, d_network_config, random_number_list_device, d_convolution_result[convolution_result_index], d_input_instance[layer_iter], layer_iter, network_size, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, time);
//    			//spiking_cnn_main(Neuron_list_device, old_device_neurons, d_network_config, random_number_list_device, d_convolution_result[0], d_input_instance[0], layer_iter, network_size, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, time);
//    			convolution_kernel(network_config, layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
//    		}else{
//    			//printf("In layer: %d\n", layer_iter);
//    			spiking_cnn_main(Neuron_list_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, layer_iter, network_size, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time);
//    			//network_config->layer[layer_iter].depth;
//    			if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(network_config, layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
//				synapse_drive_cnn_v2(Neuron_list_device, network_config, d_network_config, d_filter_array, layer_iter, network_size, syn_timer_max, connection_size, random_number_list_device, states, -1.0, -1.0);//STDP
//    		}
//
//    	}
//    	//=================TRY WITH LAYER wise inhibition=====================
//
//    	hipMemcpy(spike_flag, spike_flag_device, CNN_total_layer_num*sizeof(int),hipMemcpyDeviceToHost);
//    	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
//    		//if (layer_iter==0) printf("Layer[%d] SpikeFlag: %d\n", layer_iter, spike_flag[layer_iter]);
//            if(spike_flag[layer_iter]>0){//use lateral inhibition
//            	//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, layer_iter, network_config, 4);
//            }
//    		spike_flag[layer_iter] = 0;
//    	}
//    	hipMemcpy(spike_flag_device,spike_flag,CNN_total_layer_num*sizeof(int),hipMemcpyHostToDevice);
//    	hipMemcpy(log_spike,log_spike_default,total_depth_number*sizeof(float),hipMemcpyDeviceToDevice);	//set the log_spike to default value
//
//	//=================TRY WITH NO LAYERAL INHIBITION, MAY BE WRONG=====================
//    	//printf("network_size: %d", network_size);
//    	//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 3); //lateral inhibition
//	//==================================================================================
//		if(HOMEOSTASIS_ENABLE){
//			if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
//				//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
//			}
//		}
//        hipDeviceSynchronize();
//
//
//        //if any neuron spikes, run inhibition
//	//hipMemcpy(spike_flag, spike_flag_device, CNN_total_layer_num*sizeof(int),hipMemcpyDeviceToHost);
//	//printf("AtTime:%d_spike_flag_is:%d\n",time,spike_flag[0]);
//        //if(spike_flag[0]>0){//use lateral inhibition
//        	//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0);
//        	//spike_flag[0] = 0;
//    	    	//hipMemcpy(spike_flag_device,spike_flag,sizeof(int),hipMemcpyHostToDevice);
//        //}
//
//        //hipMemcpy(old_device_neurons,Neuron_list_device,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
//        //hipDeviceSynchronize();
//    	time ++;
//    }
//    //spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, 2);
//	//hipDeviceSynchronize();
//
//	filter_util(network_config, Neuron_list_device, network_size, h_filter_array, d_filter_array, 2);
//    hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
//    hipMemcpy(log_v_host,log_v,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
//    hipMemcpy(log_spike_host,log_spike,total_depth_number*sizeof(float),hipMemcpyDeviceToHost);
//    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );
//
//
//    //print out the synapse conductance data
//    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2);
//
//    ofstream myfile ((index_prefix+"device2_spike_of_neuron_out.csv"));
//    if (myfile.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < SIZE; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile << log_total_spike_host[i] << ", ";
//    	}
//    	myfile.close();
//    }
//
//    ofstream myfile_p ((index_prefix+"probe.csv"));
//    if (myfile_p.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < 1000; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_p << probe[i] << ", ";
//    	}
//    	myfile_p.close();
//    }
//
////
////    ofstream myfile_0 ((index_prefix+"device2_out_v.csv"));
////    if (myfile_0.is_open()){
////    	//myfile << "This is a new test\n";
////    	for(int i=0; i < MAX_TIME; i++){
////    		//printf("_%f_", log_v_host[i]);
////    		myfile_0 << log_v_host[i] << ", ";
////    	}
////    	myfile.close();
////    }
////
////    ofstream myfile_2 ((index_prefix+"device2_spike_of_one.csv"));
////    if (myfile_2.is_open()){
////    	//myfile << "This is a new test\n";
////    	for(int i=0; i < MAX_TIME; i++){
////    		//printf("_%f_", log_v_host[i]);
////    		myfile_2 << log_spike_host[i] << ", ";
////    	}
////    	myfile_2.close();
////    }
//
//
//
//    hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
//	filter_util(network_config, NeuronList, network_size, h_filter_array, d_filter_array, 1);	//write filter to file
//    write_neuron_list(NeuronList, (index_prefix+"device2_output_network.txt"), network_size);
//    data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, "");
//    //===clean up===
//    //delete[] random_number_list;
//    delete[] log_v_host;
//	delete[] NeuronList;
//	delete[] log_spike_host;
//	delete[] log_total_spike_host;
//	delete[] mnist_img;
//	delete[] NeuronList_temp;
//	delete[] one_mnist_img;
//	delete[] probe;
//	delete[] random_number_list;
//	delete[] mnist_label;
//	delete[] spike_flag;
//	delete[] num_one_digit_img;
//	//hipFree(states);
//	hipFree(log_v);
//	hipFree(log_spike);
//	hipFree(log_total_spike);
//	hipFree(Neuron_list_device);
//	//hipFree(old_device_neurons);
//	hipFree(random_number_list_device);
//	hipFree(d_network_config);
//	hipFree(states);
//	hipFree(spike_flag_device);
//	hipFree(log_spike_default);
//
//}

void run_cnn(string index_prefix, float input_float, float input_float_2, int input_int, int input_int_2, string input_img){
	/*
	int training_set_number = 1;
	int size_per_img = input_image_w * input_image_l*input_image_channel;
	float *mnist_img = new float[size_per_img*training_set_number];
	string image_file = "train-images-idx3-ubyte"; //"train-images-idx3-ubyte";
	MNIST_read_image(image_file, mnist_img, training_set_number);
	int *mnist_label = new int[training_set_number];
	string image_label_file = "train-labels-idx1-ubyte";
	MNIST_read_label(image_label_file, mnist_label, training_set_number);

	float *filter;
	float *output = new float[size_per_img*training_set_number];

	convolution_kernel(mnist_img, filter, output);
	img_util(output, "test_output.jpg", 0);
	*/
	int resume_learning = 0;
	CNN_struct *network_config = new CNN_struct;
	network_config_generator(3, network_config);
	Neuron *NeuronList_temp = new Neuron[1];
	CNN_struct *d_network_config;
	hipMalloc((void **)&d_network_config,sizeof(CNN_struct));
	hipMemcpy(d_network_config,network_config,sizeof(CNN_struct),hipMemcpyHostToDevice);
	int total_depth_number = 0;
	for(int i=0;i<CNN_total_layer_num; i++){
		total_depth_number = total_depth_number + network_config->layer[i].depth;
		cout<<"depth number: "<<network_config->layer[i].depth<<endl;
	}

	cout<<endl<<"Total depth number: "<<total_depth_number<<endl;
	float **h_filter_array;
	float **d_filter_array;
	int filter_array_size = CNN_total_layer_num-1;
	hipMalloc(&d_filter_array, filter_array_size*sizeof(float *));
	h_filter_array = (float**)malloc(filter_array_size * sizeof(float*));
	filter_util(network_config, NeuronList_temp, 0,0,  h_filter_array, d_filter_array, index_prefix, 0);

	/*
	img_util(mnist_img, "tensorflow_small.png", 1);
	img_util(mnist_img, "test_output_-1.png", 0);

	float *output = new float[size_per_img*training_set_number];

	int image_bytes = input_image_channel * input_image_l * input_image_w * sizeof(float);

	float* convolution_device_input{nullptr};
	hipMalloc(&convolution_device_input, image_bytes);
	hipMemcpy(convolution_device_input, mnist_img, image_bytes, hipMemcpyHostToDevice);

	int filter_in_channel = input_image_channel;
	int filter_out_channel = input_image_channel;
	int filter_height = 3;
	int filter_width = 3;
	const float kernel_template[3][3] = {
	{1, 1, 1},
	{1, -8, 1},
	{1, 1, 1}
	};
	float h_kernel[filter_in_channel][filter_out_channel][filter_height][filter_width];
	for (int kernel = 0; kernel < filter_in_channel; ++kernel) {
		for (int channel = 0; channel < filter_out_channel; ++channel) {
		  for (int row = 0; row < filter_height; ++row) {
			for (int column = 0; column < filter_width; ++column) {
			  h_kernel[kernel][channel][row][column] = kernel_template[row][column];
			}
		  }
		}
	}
	float* filter{nullptr};
	hipMalloc(&filter, sizeof(h_kernel));
	hipMemcpy(filter, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);


	convolution_kernel(convolution_device_input, filter, output);
	img_util(output, "test_output_1.png", 0);

	hipFree(filter);
	hipFree(convolution_device_input);

	*/

//===========END of CNN special setting-up phase============

	Convolution_setting_struct *convolution_settings = new Convolution_setting_struct[CNN_total_layer_num];

	//set parameters

	int training_time_each_img = input_int;
	int calculated_total_time = training_time_each_img*1000;
	#undef MAX_TIME
	#define MAX_TIME calculated_total_time
	printf("==Training Total Iter: %d==", MAX_TIME);
	int total_neuron_num = 0;
	int total_spiking_num = 0;
	for(int i=0;i<CNN_total_layer_num;i++){
		total_neuron_num += network_config->layer[i].neuron_num;
		if(i!=0)
		total_spiking_num += network_config->layer[i].neuron_num;
	}
	total_spiking_num += 10;
	total_neuron_num += 10;
	//total_neuron_num = 20000;
	cout<<endl<<"total neuron num: "<<total_neuron_num<<endl;
	cout<<"total spiking neuron num: "<<total_spiking_num<<endl;
	#undef SIZE
	#define SIZE total_neuron_num
	#undef SPIKING_NEURON_NUM
	#define SPIKING_NEURON_NUM total_spiking_num


	float max_frequency = 100; //in Hz default 22
	float min_frequency = 1;
	int training_set_number = 6;
	int training_folder_number = 800;
	int input_neuron_num = input_image_w*input_image_l*input_image_channel;
	int input_image_signal_channel_size  = input_image_w*input_image_l;
	int spiking_neuron_num = SPIKING_NEURON_NUM;
	int output_layer_neuron_num = OUTPUT_LAYER_NEURON_NUM;
	int tenpercent_iter = MAX_TIME/10;
	int connection_size = MAX_CONNECTION;
	int syn_timer_max = 25;
	int input_signal_width = 15;	//default 25
	int inhibition_time = 10;	//default 10

	float target_frequency_param = 0.5;
	float target_frequency = 100;
	float *mnist_img = new float[input_neuron_num*training_set_number];
//	for(int mi=0; mi<input_neuron_num*training_set_number; mi++) mnist_img[mi] = (rand()%100);
	//for(int i=0;i<input_neuron_num*trainingx_set_number;i++) mnist_img[i] = 0;
	string image_file = "train-images-idx3-ubyte";//"MNIST_train_dataset_8x8_3bit_simple_resize";//"train_dataset_noisy_cifar";//"fashion-train-images-idx3-ubyte";//"train_dataset_noisy";//"train_dataset_noisy"; //"train-images-idx3-ubyte";
	cout<<endl<<"Image loading"<<endl;
	clock_t load_start = clock();
    std::vector<std::string> folder_list;
    int input_folder_cnt = 0;
	bool learn_imageNet = false;
	if(input_image_channel==1){
		//CIFAR_read_image(mnist_img, input_neuron_num, 0, 1);
		//GTVIR_read_image(mnist_img, input_neuron_num, training_set_number);
		//MNIST_read_image(image_file, mnist_img, training_set_number);
		read_polygon("/inverse_polygon/tri", mnist_img, training_set_number);
	}else{

		if(learn_imageNet){

			//ifstream file ("imageNet_selected_folder_list.csv");
			ifstream file ("imageNet_folder_list.csv");
			string val;

		    while(file.good()) {
				getline(file, val, ',');
		    	folder_list.push_back(val);
		    }

			imageNET_read_image(folder_list[input_folder_cnt], mnist_img, training_set_number);
		}else{
			CIFAR_read_image(mnist_img, input_neuron_num, training_set_number, 0, 0);
			//KAIST_PED_read_image("", mnist_img , training_set_number);
		}
	}
	clock_t load_end = clock();

	cout<<endl<<"Image loading done"<<", time used is " << (load_end - load_start)/1000 << " (ms)"<<endl;

	//CIFAR_read_image_one_channel(mnist_img, input_image_signal_channel_size, input_int_2, 0);
	//MNIST_read_image(image_file, mnist_img, training_set_number);
	int *mnist_label = new int[training_set_number];
	string image_label_file = "train-labels-idx1-ubyte";
	//CIFAR_read_label(mnist_label, 0);
	//MNIST_read_label(image_label_file, mnist_label, training_set_number);
	//special_function: learn one category
	printf("=0=\n");
	int learn_one_digit = 0;
	int *num_one_digit_img = new int[1];
	if(learn_one_digit){
		MNIST_labeling("abc", 60000, mnist_img, mnist_label, mnist_img, num_one_digit_img, spiking_neuron_num, 1, 5);
		//printf("Learning only one digit, number of img: %d\n", num_one_digit_img[0]);
	}

	//int synapse_size = SIZE*SIZE;
	//cout<<SIZE<<endl;
    Neuron *NeuronList = new Neuron[spiking_neuron_num];
    Input_neuron *Input_neuronlist = new Input_neuron[input_neuron_num];
	//unsigned char *synapse_timer = new unsigned char[synapse_size];  //this is the array that stores timer used in STPD. e.g Neuron x --->  Neuron y Spike! In the array index [(x-1)*SIZE+(y-1)]  => 1
	//hiprandState_t *states;
	//float *random_number_list = new float[SIZE];
	float *log_v_host = new float[MAX_TIME];
	float *log_spike_host = new float[total_depth_number];

	float *log_total_spike_host = new float[SIZE];
	for(int i=0; i < SIZE; i++){
		log_total_spike_host[i] = 0;
	}
	int *spike_flag = new int[CNN_total_layer_num];
	for(int i=0; i < CNN_total_layer_num; i++){
		spike_flag[i] = 0;
	}
	for(int i=0; i<total_depth_number; i++) log_spike_host[i] = 0;
	//init_log_v(log_v_host);
	//init_data_log(log_v_host,log_spike_host,log_total_spike_host, MAX_TIME);
	neuron_list_init(NeuronList, spiking_neuron_num);
	input_neuron_list_init(Input_neuronlist, input_neuron_num);
	printf("=1=\n");
	//
	if(resume_learning){
		printf("RESUME LEARNING\n");
		read_neuron_list(NeuronList, 1, "device2_output_network.txt");
	}else{
		read_neuron_list(NeuronList, 1, "spike_cnn.txt");
	}
    //write_neuron_list(NeuronList, "learning_output_confirm.txt", SIZE);
	//check_neuron(NeuronList, 800, 820);


	//Neuron *old_device_neurons;
	//unsigned char *snapse_timer_device;
	float *log_v;
	float *log_spike;
	float *log_spike_default;
	float *log_total_spike;
	int *spike_flag_device;


    printf("2\n");
	//random number function:

    float rand_list_size_to_total_connection_ratio = 1;
	int rand_numb_size = SPIKING_NEURON_NUM*MAX_CONNECTION;

	int SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
    dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
	dim3 print_grid(1);
	dim3 print_block(1);
	dim3 dimBlock_unit( 1, 1 );
	dim3 dimGrid_unit(1, 1);
    printf("2.1\n");

	hiprandState_t *states;
	hipMalloc((void **)&states, rand_numb_size * sizeof(hiprandState_t));
//	if (STOCHASTIC_STDP) rand_init<<<dimGrid,dimBlock>>>(time(0), rand_numb_size, states);
//	float *random_number_list = new float[rand_numb_size];
//	float *random_number_list_device;
//	SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
//	dim3 dimBlock_synapse( ThreadsPerBlock, ThreadsPerBlock );
//	dim3 dimGrid_synapse( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));

//	hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
//	hipMemcpy(random_number_list_device,random_number_list,rand_numb_size*sizeof(float),hipMemcpyHostToDevice);
//	if (STOCHASTIC_STDP) random<<<dimGrid_synapse,dimBlock_synapse>>>(random_number_list_device, rand_numb_size, states);

    hiprandGenerator_t gen_uniform;
    float *random_number_list_device;
    hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
    hiprandCreateGenerator(&gen_uniform, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen_uniform, time(0));
    hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);


    hiprandGenerator_t gen_normal;
    float *random_number_normal_device;
    float normal_mean = 0;
    float normal_sd = 5.0;
    hipMalloc((void **)&random_number_normal_device,rand_numb_size*sizeof(float));
    hiprandCreateGenerator(&gen_normal, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen_normal, time(0));
    hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);


//    printf("2.11\n");
    //Setting up input instance matrix:
    float **d_input_instance;
    float **d_convolution_result;
    float **h_input_instance;
    float **h_convolution_result;

    float *probe = new float[1000];
	int instance_array_size = CNN_total_layer_num;
	hipMalloc(&d_input_instance, instance_array_size*sizeof(float *));
	int convolution_result_size = CNN_total_layer_num - 1;
	hipMalloc(&d_convolution_result, convolution_result_size*sizeof(float *));
    h_input_instance = (float**)malloc(instance_array_size * sizeof(float*));
    h_convolution_result = (float**)malloc(convolution_result_size * sizeof(float*));
    CNN_util(network_config, d_input_instance, d_convolution_result, h_input_instance, h_convolution_result, 0);
    printf("2.2\n");
//	float **add = &h_convolution_result[0];
//	printf("Address On GPU: %p\n", add);

    //========Setting up device neuron list============

	Neuron *Neuron_list_device;
    Input_neuron *Input_neuronlist_device;
    hipMalloc((void **)&Neuron_list_device, spiking_neuron_num*sizeof(Neuron));
    hipMalloc((void **)&Input_neuronlist_device, input_neuron_num*sizeof(Input_neuron));
    //hipMalloc((void **)&old_device_neurons, SIZE*sizeof(Neuron));

    //hipMalloc((void **)&states, SIZE * sizeof(hiprandState_t));
    hipMalloc((void **)&log_v, MAX_TIME * sizeof(float));
    hipMalloc((void **)&log_spike, total_depth_number * sizeof(float));
    hipMalloc((void **)&log_spike_default, total_depth_number * sizeof(float));
    //hipMalloc((void **)&log_total_spike, SIZE * sizeof(float));
    gpuErrchk( hipMalloc((void **)&log_total_spike, SIZE * sizeof(float)) );
    hipMalloc((void **)&spike_flag_device, instance_array_size*sizeof(int));
    //rand_init<<<dimGrid,dimBlock>>>(time(0), states);
    printf("2.3\n");
    hipMemcpy(Neuron_list_device,NeuronList,spiking_neuron_num*sizeof(Neuron),hipMemcpyHostToDevice);
    hipMemcpy(Input_neuronlist_device,Input_neuronlist,input_neuron_num*sizeof(Input_neuron),hipMemcpyHostToDevice);
    //hipMemcpy(old_device_neurons,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
    //hipMemcpy(random_number_list_device, random_number_list, SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(log_v,log_v_host,MAX_TIME*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike_default,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
    hipMemcpy(spike_flag_device,spike_flag,instance_array_size*sizeof(int),hipMemcpyHostToDevice);
    printf("3\n");
    //cout<<"network size: "<<SIZE<<endl;
    int network_size = SIZE;

    int max_time = MAX_TIME;


    //hipMemcpy(Neuron_list_device,old_device_neurons,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
    //first change raw img data into frequency
    int mnist_start_index = 0;
    int mnist_end_index = input_neuron_num;
    //change pixel signal to frequency

    MNIST_drive(NeuronList, Input_neuronlist, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 1);
    std::srand ( unsigned ( std::time(0) ) );
    std::vector<int> myvector;
    for (int i=0; i<training_set_number; ++i) myvector.push_back(i); // 1 2 3 4 5 6 7 8 9

    if(shuffle_image){
    	  std::random_shuffle ( myvector.begin(), myvector.end() );
    }

    hipDeviceSynchronize();

    //data_check(Neuron_list_device,log_total_spike,SIZE,1);
    float *one_mnist_img = new float[input_neuron_num];

    clock_t iter_start, iter_log;
    iter_start = clock();
    int log_interval = MAX_TIME/10;
    //read_filter_GPU_one_layer<<<1, 1>>>(d_network_config, h_filter_array[0], 1);
    //read_filter_GPU<<<1, 1>>>(d_network_config, d_filter_array);

    int reiter_run = 1;

    int time = 0;
    int training_img_index = 0;

    //============now load all convolution settings===========
	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
		if (layer_iter==0) {
			convolution_kernel_setup(convolution_settings, network_config, layer_iter);
		}else{
			if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel_setup(convolution_settings, network_config, layer_iter);
		}
	}
	bool enable_log_interval = false;
	while (time<max_time){
    	//cout<<endl<<" It: "<<time<<endl;
        //random<<<dimGrid,dimBlock>>>(random_number_list_device, states);
    	//first create an array of 1 MNIST image
//    	if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING){
//            random<<<dimGrid_synapse,dimBlock_synapse>>>(random_number_list_device, rand_numb_size, states);
//    	}

    	if(DEVICE_VARIATION){
            hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);
    	}
    	if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING){
    	    hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);
    	}

    	if(time%log_interval == 0 && enable_log_interval){
    		hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
    		printf("NN data copy complete\n");
    		string interval_file_name = "device2_output_at_iter_" + to_string(time) + ".txt";
    		//data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, (to_string(time)+"_"));
    		//if (time>0) write_neuron_list(NeuronList, interval_file_name, network_size);
    	}

    	if(time%tenpercent_iter == 0){
    		iter_log = clock();
    		cout<<to_string(10*(time/tenpercent_iter))<<"% done, time used is: " << (iter_log - iter_start)/1000 << " (ms)" << endl;
    	}

    	/****************************************/
    	/**********at the beginning of each img's training, load into input variable
    	/****************************************/
    	if(time%training_time_each_img==0){
    		//cout<<"Image Load Iter: "<<time<<endl;
			int locate_index = myvector[training_img_index];
			//cout<<"loading index: "<<locate_index<<endl;
    		for(int i=0;i<input_neuron_num;i++){
    			one_mnist_img[i] = mnist_img[locate_index*input_neuron_num+i];
    		}
    		MNIST_drive(Neuron_list_device, Input_neuronlist_device, one_mnist_img, input_neuron_num, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
    		//MNIST_drive(old_device_neurons, one_mnist_img, network_size,training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
    		training_img_index ++;
    		if(training_img_index>=training_set_number-1){
    			if(learn_imageNet&&(input_folder_cnt<training_folder_number-1)){
    				input_folder_cnt++;
    				imageNET_read_image(folder_list[input_folder_cnt], mnist_img, training_set_number);
    				if(input_folder_cnt==training_folder_number-1) input_folder_cnt = 0;
    			}
    	    	if (shuffle_image) std::random_shuffle ( myvector.begin(), myvector.end() );
    			training_img_index = 0;
    		}
    		//confirm the data in signal neuron
    		//hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
    		//data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 3);
    		//printf("\n\n\n************************\n\n\n\n");
    	}
    	//cout<<"One IMG loaded"<<endl;

    	/****************************************/
    	/**********simulate all layers
    	/****************************************/

    	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
    		int convolution_result_index = layer_iter - 1;
    		if (layer_iter==0) {
    			convolution_result_index = 0;
    			spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
    					layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
    			convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
    		}else{
    			spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
    					layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, through_depth_inhibition);
    			if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				synapse_drive_cnn_v2(Neuron_list_device, Input_neuronlist_device, network_config, d_network_config, d_filter_array, layer_iter, spiking_neuron_num, input_neuron_num, syn_timer_max, connection_size, random_number_list_device, random_number_normal_device, states, -1.0, -1.0, log_total_spike);//STDP
    		}

    	}
    	//=================TRY WITH LAYER wise inhibition=====================
    	//hipDeviceSynchronize();
    	if(depth_wise_inhibition) {
    		//lateral_inhibition_depth_wise_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, input_image_channel, inhibition_time, d_network_config, log_spike, total_depth_number);
    	}else if(through_depth_inhibition){

    	}else if(apply_local_inhibition){

    	}else{
    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 1, inhibition_time, d_network_config, spike_flag_device);
    	}
//    	hipMemcpy(spike_flag, spike_flag_device, CNN_total_layer_num*sizeof(int),hipMemcpyDeviceToHost);
//    	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
//    		//if (layer_iter==0) printf("Layer[%d] SpikeFlag: %d\n", layer_iter, spike_flag[layer_iter]);
//            if(spike_flag[layer_iter]>0){//use lateral inhibition
//            	spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, layer_iter, network_config, 4);
//            }
//    		spike_flag[layer_iter] = 0;
//    	}
//    	hipMemcpy(spike_flag_device,spike_flag,CNN_total_layer_num*sizeof(int),hipMemcpyHostToDevice);
//    	hipMemcpy(log_spike,log_spike_default,total_depth_number*sizeof(float),hipMemcpyDeviceToDevice);	//set the log_spike to default value

	//=================TRY WITH NO LAYERAL INHIBITION, MAY BE WRONG=====================
    	//printf("network_size: %d", network_size);
    	//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 3); //lateral inhibition
	//==================================================================================
		if(HOMEOSTASIS_ENABLE){
			if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
				//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
			}
		}
        //hipDeviceSynchronize();


        //if any neuron spikes, run inhibition
	//hipMemcpy(spike_flag, spike_flag_device, CNN_total_layer_num*sizeof(int),hipMemcpyDeviceToHost);
	//printf("AtTime:%d_spike_flag_is:%d\n",time,spike_flag[0]);
        //if(spike_flag[0]>0){//use lateral inhibition
        	//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0);
        	//spike_flag[0] = 0;
    	    	//hipMemcpy(spike_flag_device,spike_flag,sizeof(int),hipMemcpyHostToDevice);
        //}

        //hipMemcpy(old_device_neurons,Neuron_list_device,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
        //hipDeviceSynchronize();
    	time ++;
    }
    //spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, 2);
	//hipDeviceSynchronize();

	filter_util(network_config, Neuron_list_device, SIZE, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 2);
    hipMemcpy(NeuronList,Neuron_list_device,spiking_neuron_num*sizeof(Neuron),hipMemcpyDeviceToHost);
    hipMemcpy(log_v_host,log_v,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(log_spike_host,log_spike,total_depth_number*sizeof(float),hipMemcpyDeviceToHost);
    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );


    //print out the synapse conductance data
    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2);

    ofstream myfile ((index_prefix+"device2_spike_of_neuron_out.csv"));
    if (myfile.is_open()){
    	//myfile << "This is a new test\n";
    	for(int i=0; i < SIZE; i++){
    		//printf("_%f_", log_v_host[i]);
    		myfile << log_total_spike_host[i] << ", ";
    	}
    	myfile.close();
    }

    ofstream myfile_p ((index_prefix+"probe.csv"));
    if (myfile_p.is_open()){
    	//myfile << "This is a new test\n";
    	for(int i=0; i < 1000; i++){
    		//printf("_%f_", log_v_host[i]);
    		myfile_p << probe[i] << ", ";
    	}
    	myfile_p.close();
    }

//
//    ofstream myfile_0 ((index_prefix+"device2_out_v.csv"));
//    if (myfile_0.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_0 << log_v_host[i] << ", ";
//    	}
//    	myfile.close();
//    }
//
//    ofstream myfile_2 ((index_prefix+"device2_spike_of_one.csv"));
//    if (myfile_2.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_2 << log_spike_host[i] << ", ";
//    	}
//    	myfile_2.close();
//    }



    //hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
	filter_util(network_config, NeuronList, network_size, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 1);	//write filter to file
    write_neuron_list(NeuronList, (index_prefix+"device2_output_network.txt"), spiking_neuron_num);
    data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, "");
    //===clean up===
    //delete[] random_number_list;
    delete[] log_v_host;
	delete[] NeuronList;
	delete[] log_spike_host;
	delete[] log_total_spike_host;
	delete[] mnist_img;
	delete[] NeuronList_temp;
	delete[] one_mnist_img;
	delete[] probe;
//	delete[] random_number_list;
	delete[] mnist_label;
	delete[] spike_flag;
	delete[] num_one_digit_img;
	//hipFree(states);
	hipFree(log_v);
	hipFree(log_spike);
	hipFree(log_total_spike);
	hipFree(Neuron_list_device);
	//hipFree(old_device_neurons);
	hipFree(random_number_list_device);
	hipFree(d_network_config);
	hipFree(states);
	hipFree(spike_flag_device);
	hipFree(log_spike_default);

}

void run_cnn_multilayer(string index_prefix, float input_float, float input_float_2, int input_int, int input_int_2, string input_img){
	//#undef shuffle_image
	//#define shuffle_image 1
	/*
	int training_set_number = 1;
	int size_per_img = input_image_w * input_image_l*input_image_channel;
	float *mnist_img = new float[size_per_img*training_set_number];
	string image_file = "train-images-idx3-ubyte"; //"train-images-idx3-ubyte";
	MNIST_read_image(image_file, mnist_img, training_set_number);
	int *mnist_label = new int[training_set_number];
	string image_label_file = "train-labels-idx1-ubyte";
	MNIST_read_label(image_label_file, mnist_label, training_set_number);

	float *filter;
	float *output = new float[size_per_img*training_set_number];

	convolution_kernel(mnist_img, filter, output);
	img_util(output, "test_output.jpg", 0);
	*/
	int resume_learning = 0;
	CNN_struct *network_config = new CNN_struct;
	network_config_generator(3, network_config);
	Neuron *NeuronList_temp = new Neuron[1];
	CNN_struct *d_network_config;
	hipMalloc((void **)&d_network_config,sizeof(CNN_struct));
	hipMemcpy(d_network_config,network_config,sizeof(CNN_struct),hipMemcpyHostToDevice);
	int total_depth_number = 0;
	for(int i=0;i<CNN_total_layer_num; i++){
		total_depth_number = total_depth_number + network_config->layer[i].depth;
		cout<<"depth number: "<<network_config->layer[i].depth<<endl;
	}

	cout<<endl<<"Total depth number: "<<total_depth_number<<endl;
	float **h_filter_array;
	float **d_filter_array;
	int filter_array_size = CNN_total_layer_num-1;
	hipMalloc(&d_filter_array, filter_array_size*sizeof(float *));
	h_filter_array = (float**)malloc(filter_array_size * sizeof(float*));
	filter_util(network_config, NeuronList_temp, 0,0,  h_filter_array, d_filter_array, index_prefix, 0);

	/*
	img_util(mnist_img, "tensorflow_small.png", 1);
	img_util(mnist_img, "test_output_-1.png", 0);

	float *output = new float[size_per_img*training_set_number];

	int image_bytes = input_image_channel * input_image_l * input_image_w * sizeof(float);

	float* convolution_device_input{nullptr};
	hipMalloc(&convolution_device_input, image_bytes);
	hipMemcpy(convolution_device_input, mnist_img, image_bytes, hipMemcpyHostToDevice);

	int filter_in_channel = input_image_channel;
	int filter_out_channel = input_image_channel;
	int filter_height = 3;
	int filter_width = 3;
	const float kernel_template[3][3] = {
	{1, 1, 1},
	{1, -8, 1},
	{1, 1, 1}
	};
	float h_kernel[filter_in_channel][filter_out_channel][filter_height][filter_width];
	for (int kernel = 0; kernel < filter_in_channel; ++kernel) {
		for (int channel = 0; channel < filter_out_channel; ++channel) {
		  for (int row = 0; row < filter_height; ++row) {
			for (int column = 0; column < filter_width; ++column) {
			  h_kernel[kernel][channel][row][column] = kernel_template[row][column];
			}
		  }
		}
	}
	float* filter{nullptr};
	hipMalloc(&filter, sizeof(h_kernel));
	hipMemcpy(filter, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);


	convolution_kernel(convolution_device_input, filter, output);
	img_util(output, "test_output_1.png", 0);

	hipFree(filter);
	hipFree(convolution_device_input);

	*/

//===========END of CNN special setting-up phase============

	Convolution_setting_struct *convolution_settings = new Convolution_setting_struct[CNN_total_layer_num];

	//set parameters

	int training_time_each_img = input_int;
	int calculated_total_time = training_time_each_img*1;
    if(resume_learning) calculated_total_time = calculated_total_time/3;

	#undef MAX_TIME
	#define MAX_TIME calculated_total_time
	printf("==Training Total Iter: %d==", MAX_TIME);
	int total_neuron_num = 0;
	int total_spiking_num = 0;
	for(int i=0;i<CNN_total_layer_num;i++){
		total_neuron_num += network_config->layer[i].neuron_num;
		if(i!=0) total_spiking_num += network_config->layer[i].neuron_num;
	}
	total_spiking_num += 5;
	total_neuron_num += 5;
	//total_neuron_num = 20000;
	cout<<endl<<"total neuron num: "<<total_neuron_num<<endl;
	cout<<"total spiking neuron num: "<<total_spiking_num<<endl;
	#undef SIZE
	#define SIZE total_neuron_num
	#undef SPIKING_NEURON_NUM
	#define SPIKING_NEURON_NUM total_spiking_num


	float max_frequency = 30; //in Hz default 22
	float min_frequency = 3;
	int training_set_number = 50000;
	bool batch_load = false;
	int batched_load_remain = 0;
	int batch_load_grand_total = 0;
	int img_load_offset = 0;
	int img_load_max = 50000;

	if (training_set_number>img_load_max){ //manually set the maximum number of images to be loaded once is 60000
		cout<<"Using batch loading"<<endl;
		batch_load_grand_total = training_set_number;
		batch_load = true;
		batched_load_remain = training_set_number - img_load_max;
		training_set_number = img_load_max;
	}
	int input_neuron_num = input_image_w*input_image_l*input_image_channel;
	int input_image_signal_channel_size  = input_image_w*input_image_l;
	int spiking_neuron_num = SPIKING_NEURON_NUM;
	int output_layer_neuron_num = OUTPUT_LAYER_NEURON_NUM;
	int tenpercent_iter = MAX_TIME/10;
	int connection_size = MAX_CONNECTION;
	int syn_timer_max = 25;
	int input_signal_width = 10;	//default 25
	int inhibition_time = 10;	//default 10

	float target_frequency_param = 0.5;
	float target_frequency = 100;
	float *mnist_img = new float[input_neuron_num*training_set_number];
	for(int i=0;i<input_neuron_num*training_set_number;i++) mnist_img[i] = 0;
	string image_file = "train-images-idx3-ubyte";//"dvs_gesture_1bit";"./NTU_Skeleton/binary_file_2000";//"train_dataset_noisy_cifar";//"fashion-train-images-idx3-ubyte";//"train_dataset_noisy";//"train_dataset_noisy"; //"train-images-idx3-ubyte";
	cout<<endl<<"Image loading"<<endl;
	clock_t load_start = clock();
    std::vector<std::string> folder_list;
    int input_folder_cnt = 0;
	if(input_image_channel==1 || input_image_channel==2){
		//CIFAR_read_image(mnist_img, input_neuron_num, 0, 1);
		//GTVIR_read_image(mnist_img, input_neuron_num, training_set_number);
		//NTU_skeleton_read_image(image_file, mnist_img, training_set_number, img_load_offset);
		//DVS_read_image(image_file, mnist_img, training_set_number);
		MNIST_read_image(image_file, mnist_img, training_set_number);
		//read_polygon("/inverse_polygon/drawings", mnist_img, training_set_number);
		//read_polygon("/rotating_mnist/readMNIST/readMNIST/training/", mnist_img, training_set_number);
		//MNIST_read_image("./rotating_f_mnist/rotating_mnist", mnist_img, training_set_number);
	}else{
		bool learn_imageNet = false;
		if(learn_imageNet){

			ifstream file ("imageNet_folder_list.csv");
			string val;

		    while(file.good()) {
				getline(file, val, ',');
		    	folder_list.push_back(val);
		    }

			imageNET_read_image(folder_list[input_folder_cnt], mnist_img, training_set_number);
		}else{
			CIFAR_read_image(mnist_img, input_neuron_num, training_set_number, 0, 0);
			//KAIST_PED_read_image("", mnist_img , training_set_number);
		}

	}
	clock_t load_end = clock();

	cout<<endl<<"Image loading done"<<", time used is " << (load_end - load_start)/1000 << " (ms)"<<endl;
	//CIFAR_read_image_one_channel(mnist_img, input_image_signal_channel_size, input_int_2, 0);
	//MNIST_read_image(image_file, mnist_img, training_set_number);
	int *mnist_label = new int[training_set_number];
	string image_label_file = "train-labels-idx1-ubyte";
	//CIFAR_read_label(mnist_label, 0);
	//MNIST_read_label(image_label_file, mnist_label, training_set_number);
	//special_function: learn one category
	printf("=0=\n");
	int learn_one_digit = 0;
	int *num_one_digit_img = new int[1];
	if(learn_one_digit){
		MNIST_labeling("abc", 60000, mnist_img, mnist_label, mnist_img, num_one_digit_img, spiking_neuron_num, 1, 5);
		printf("Learning only one digit, number of img: %d\n", num_one_digit_img);
	}

	//int synapse_size = SIZE*SIZE;
	//cout<<SIZE<<endl;
    Neuron *NeuronList = new Neuron[spiking_neuron_num];
    Input_neuron *Input_neuronlist = new Input_neuron[input_neuron_num];
	//unsigned char *synapse_timer = new unsigned char[synapse_size];  //this is the array that stores timer used in STPD. e.g Neuron x --->  Neuron y Spike! In the array index [(x-1)*SIZE+(y-1)]  => 1
	//hiprandState_t *states;
	//float *random_number_list = new float[SIZE];
	float *log_v_host = new float[MAX_TIME];
	float *log_spike_host = new float[total_depth_number];

	float *log_total_spike_host = new float[SIZE];
	for(int i=0; i < SIZE; i++){
		log_total_spike_host[i] = 0;
	}
	int *spike_flag = new int[CNN_total_layer_num];
	for(int i=0; i < CNN_total_layer_num; i++){
		spike_flag[i] = 0;
	}
	for(int i=0; i<total_depth_number; i++) log_spike_host[i] = 0;
	//init_log_v(log_v_host);
	//init_data_log(log_v_host,log_spike_host,log_total_spike_host, MAX_TIME);
	neuron_list_init(NeuronList, spiking_neuron_num);
	input_neuron_list_init(Input_neuronlist, input_neuron_num);
	printf("=1=\n");
	//
	if(resume_learning){
		printf("------RESUME LEARNING-------\n");
		read_neuron_list(NeuronList, 1, "spike_cnn.txt");
		//read_neuron_list(NeuronList, 1, "device2_output_network.txt");
		int start_layer = 3;//the layer that starts to learn
		read_neuron_list_special(NeuronList, (start_layer-1), network_config, "device2_output_network_org.txt"); //duplicate the previous layer
		bool do_reset_weight = true;
		if(do_reset_weight){

			float start_depth = network_config->layer[start_layer].first_depth_id - 0.1;
			float end_depth = network_config->layer[start_layer].last_depth_id + 0.1;

			reset_weight(NeuronList, start_depth, end_depth, 1, spiking_neuron_num);

		}
		//read_neuron_list(NeuronList, 1, "spike_cnn.txt");
	}else{
		read_neuron_list(NeuronList, 1, "spike_cnn.txt");
		//read_neuron_list(NeuronList, 1, "device2_output_network.txt");

	}
	//printf("read out one neuron depth: %f", NeuronList[116000].param[7]);
    //write_neuron_list(NeuronList, "learning_output_confirm.txt", SIZE);
	//check_neuron(NeuronList, 800, 820);


	//Neuron *old_device_neurons;
	//unsigned char *snapse_timer_device;
	float *log_v;
	float *log_spike;
	float *log_spike_default;
	float *log_total_spike;
	int *spike_flag_device;


    printf("2\n");
	//random number function:

    float rand_list_size_to_total_connection_ratio = 1;
	int rand_numb_size = SPIKING_NEURON_NUM*MAX_CONNECTION;

	int SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
    dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
	dim3 print_grid(1);
	dim3 print_block(1);
	dim3 dimBlock_unit( 1, 1 );
	dim3 dimGrid_unit(1, 1);

	int SIZE_PER_SIDE_whole_network = sqrt(spiking_neuron_num)+1;
    dim3 dimBlock_whole_network( ThreadsPerBlock*2, ThreadsPerBlock );
    dim3 dimGrid_whole_network( (SIZE_PER_SIDE_whole_network/dimBlock.x+1), (SIZE_PER_SIDE_whole_network/dimBlock.y+1));
    printf("2.1\n");

	hiprandState_t *states;

//	if (STOCHASTIC_STDP) rand_init<<<dimGrid,dimBlock>>>(time(0), rand_numb_size, states);
//	float *random_number_list = new float[rand_numb_size];
//	float *random_number_list_device;
//	SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
//	dim3 dimBlock_synapse( ThreadsPerBlock, ThreadsPerBlock );
//	dim3 dimGrid_synapse( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));

//	hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
//	hipMemcpy(random_number_list_device,random_number_list,rand_numb_size*sizeof(float),hipMemcpyHostToDevice);
//	if (STOCHASTIC_STDP) random<<<dimGrid_synapse,dimBlock_synapse>>>(random_number_list_device, rand_numb_size, states);

    hiprandGenerator_t gen_uniform;
    float *random_number_list_device;
    if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING || DEVICE_VARIATION){
    	hipMalloc((void **)&states, rand_numb_size * sizeof(hiprandState_t));
		hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
		hiprandCreateGenerator(&gen_uniform, HIPRAND_RNG_PSEUDO_DEFAULT);
		hiprandSetPseudoRandomGeneratorSeed(gen_uniform, time(0));
		hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);
    }


    hiprandGenerator_t gen_normal;
    float *random_number_normal_device;
    float normal_mean = 0;
    float normal_sd = 5.0;
    if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING || DEVICE_VARIATION){
		hipMalloc((void **)&random_number_normal_device,rand_numb_size*sizeof(float));
		hiprandCreateGenerator(&gen_normal, HIPRAND_RNG_PSEUDO_DEFAULT);
		hiprandSetPseudoRandomGeneratorSeed(gen_normal, time(0));
		hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);
    }

//    printf("2.11\n");
    //Setting up input instance matrix:
    float **d_input_instance;
    float **d_convolution_result;
    float **h_input_instance;
    float **h_convolution_result;
    float *probe = new float[1000];
	int instance_array_size = CNN_total_layer_num;
	hipMalloc(&d_input_instance, instance_array_size*sizeof(float *));
	int convolution_result_size = CNN_total_layer_num - 1;
	hipMalloc(&d_convolution_result, convolution_result_size*sizeof(float *));
    h_input_instance = (float**)malloc(instance_array_size * sizeof(float*));
    h_convolution_result = (float**)malloc(convolution_result_size * sizeof(float*));
    CNN_util(network_config, d_input_instance, d_convolution_result, h_input_instance, h_convolution_result, 0);
    printf("2.2\n");
//	float **add = &h_convolution_result[0];
//	printf("Address On GPU: %p\n", add);

    //========Setting up device neuron list============

	Neuron *Neuron_list_device;
    Input_neuron *Input_neuronlist_device;
    hipMalloc((void **)&Neuron_list_device, spiking_neuron_num*sizeof(Neuron));
    hipMalloc((void **)&Input_neuronlist_device, input_neuron_num*sizeof(Input_neuron));
    //hipMalloc((void **)&old_device_neurons, SIZE*sizeof(Neuron));

    //hipMalloc((void **)&states, SIZE * sizeof(hiprandState_t));
    hipMalloc((void **)&log_v, MAX_TIME * sizeof(float));
    hipMalloc((void **)&log_spike, total_depth_number * sizeof(float));
    hipMalloc((void **)&log_spike_default, total_depth_number * sizeof(float));
    //hipMalloc((void **)&log_total_spike, SIZE * sizeof(float));
    gpuErrchk( hipMalloc((void **)&log_total_spike, SIZE * sizeof(float)) );
    hipMalloc((void **)&spike_flag_device, instance_array_size*sizeof(int));
    //rand_init<<<dimGrid,dimBlock>>>(time(0), states);
    printf("2.3\n");
    hipMemcpy(Neuron_list_device,NeuronList,spiking_neuron_num*sizeof(Neuron),hipMemcpyHostToDevice);
    hipMemcpy(Input_neuronlist_device,Input_neuronlist,input_neuron_num*sizeof(Input_neuron),hipMemcpyHostToDevice);
    //hipMemcpy(old_device_neurons,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
    //hipMemcpy(random_number_list_device, random_number_list, SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(log_v,log_v_host,MAX_TIME*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike_default,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
    hipMemcpy(spike_flag_device,spike_flag,instance_array_size*sizeof(int),hipMemcpyHostToDevice);
    printf("3.0\n");
    //cout<<"network size: "<<SIZE<<endl;
    int network_size = SIZE;

    int max_time = MAX_TIME;
    int first_layer_time = max_time*1/6;
    int second_layer_time = max_time*1/3;
    int third_layer_time = max_time*2/3;
    if(CNN_total_layer_num==3){
    	first_layer_time = max_time*1/3;
    	second_layer_time = max_time + 1;
    	third_layer_time = max_time + 1;
    }
    else if (CNN_total_layer_num==2){
    	first_layer_time = max_time;
    	second_layer_time = max_time + 1;
    	third_layer_time = max_time + 1;
    }else if (CNN_total_layer_num==4){
    	first_layer_time = max_time*1/5;
    	second_layer_time = max_time*3/5;
    	third_layer_time = max_time + 1;
    }else if (CNN_total_layer_num==5){
    	first_layer_time = max_time*1/7;
    	second_layer_time = max_time*3/7;
    	third_layer_time = max_time*5/7;
    }
    if(resume_learning){
    	first_layer_time = 1;
        if(CNN_total_layer_num==3) second_layer_time = max_time;
        if(CNN_total_layer_num==4) second_layer_time = 2;
        if(CNN_total_layer_num==4){
        	second_layer_time = 2;
        	third_layer_time = 3;
        }
    }

    //hipMemcpy(Neuron_list_device,old_device_neurons,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
    //first change raw img data into frequency
    int mnist_start_index = 0;
    int mnist_end_index = input_neuron_num;
    //change pixel signal to frequency

    MNIST_drive(NeuronList, Input_neuronlist, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, \
    		max_frequency, min_frequency, 1); //change to spike frequency
    std::srand ( unsigned ( std::time(0) ) );
    std::vector<int> myvector;
    for (int i=0; i<training_set_number; ++i) myvector.push_back(i); // 1 2 3 4 5 6 7 8 9

    if(shuffle_image){
    	  std::random_shuffle ( myvector.begin(), myvector.end() );
    }

    hipDeviceSynchronize();

    //data_check(Neuron_list_device,log_total_spike,SIZE,1);
    float *one_mnist_img = new float[input_neuron_num];

    clock_t iter_start, iter_log;
    iter_start = clock();
    int log_interval = MAX_TIME/10;
	bool enable_log_interval = false;
	bool last_layer_teach = false;

	int total_class = 10;
	int num_per_class = 500;
	int frame_per_seq = 50;

	int rotation_num = 2;
	int translation_num = 1;
	int frame_per_class = frame_per_seq*rotation_num*num_per_class;
	int frame_per_rotation = frame_per_seq;
	int frame_per_translation = frame_per_seq*rotation_num*num_per_class*total_class;

    std::vector<int> seq_vector_head;
    std::vector<int> seq_vector;
    for (int i=0; i<training_set_number/frame_per_seq; ++i) seq_vector_head.push_back(i); // 1 2 3 4 5 6 7 8 9
	std::random_shuffle ( seq_vector_head.begin(), seq_vector_head.end() );

	for (int i=0 ; i<training_set_number/frame_per_seq; ++i){
		int begin_index = seq_vector_head[i];
		for (int j=0; j<frame_per_seq; ++j) seq_vector.push_back(10*begin_index+j);

	}

    //read_filter_GPU_one_layer<<<1, 1>>>(d_network_config, h_filter_array[0], 1);
    //read_filter_GPU<<<1, 1>>>(d_network_config, d_filter_array);

//    int reiter_run = 1;

    int time = 0;
    int training_img_index = 0;

    //============now load all convolution settings===========
	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
		if (layer_iter==0) {
			cout<<endl<<"Setting up the first layer"<<endl;
			convolution_kernel_setup(convolution_settings, network_config, layer_iter);
		}else{

			if (layer_iter!=(CNN_total_layer_num-1))
			{
				convolution_kernel_setup(convolution_settings, network_config, layer_iter);
				cout<<"Setting up the"<<" layer "<< layer_iter <<endl;
			}
		}
	}
    if(resume_learning){
		copy_filter_to_cuDNN(Neuron_list_device, d_network_config, d_filter_array, spiking_neuron_num);
		hipDeviceSynchronize();
    }
	float start_depth = network_config->layer[1].first_depth_id - 0.1;
	float end_depth = network_config->layer[1].last_depth_id + 0.1;
//	cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//	change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -62.2);
	//return;
	while (time<max_time){

		//if(time==first_layer_time)MNIST_drive(NeuronList, Input_neuronlist, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, max_frequency*2, min_frequency, 1);
    	if(DEVICE_VARIATION){
            hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);
    	}
    	if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING){
    	    hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);
    	}

    	if(time%log_interval == 0 && enable_log_interval){
    	    hipMemcpy(NeuronList,Neuron_list_device,spiking_neuron_num*sizeof(Neuron),hipMemcpyDeviceToHost);
    		printf("NN data copy complete\n");
    		string interval_file_name = "device2_output_at_iter_" + to_string(time) + ".txt";
    		//string interval_weight_file_name = to_string(time);
    	    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, (to_string(time)+"_"));
    		if (time>0) {
    			write_neuron_list(NeuronList, interval_file_name, spiking_neuron_num);
    		    hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
    			filter_util(network_config, NeuronList, network_size, input_neuron_num, h_filter_array, d_filter_array, to_string(time), 1);	//write filter to file
    		}
    	}

    	if(time==first_layer_time){

    	    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );
    	    ofstream myfile ((index_prefix+"first_stage_device2_spike_of_neuron_out.csv"));
    	    if (myfile.is_open()){
    	    	//myfile << "This is a new test\n";
    	    	for(int i=0; i < SIZE; i++){
    	    		//printf("_%f_", log_v_host[i]);
    	    		myfile << log_total_spike_host[i] << ", ";
    	    	}
    	    	myfile.close();
    	    }

        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
//    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -58.2);
//    		cout<<"Changing param of long-term neuron, start: "<< start_depth+32<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+32, end_depth, 5, -1.6);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+32, end_depth, 4, 0.16);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+32, end_depth, 0, -0.001);
    		//change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+32, end_depth, -56.2);

        	start_depth = network_config->layer[2].first_depth_id - 0.1;
        	end_depth = network_config->layer[2].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -62.0);


    		//training_time_each_img = training_time_each_img*1.3;
    		hipDeviceSynchronize();
    	}else if(time==second_layer_time){

    	    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );
    	    ofstream myfile ((index_prefix+"second_stage_device2_spike_of_neuron_out.csv"));
    	    if (myfile.is_open()){
    	    	//myfile << "This is a new test\n";
    	    	for(int i=0; i < SIZE; i++){
    	    		//printf("_%f_", log_v_host[i]);
    	    		myfile << log_total_spike_host[i] << ", ";
    	    	}
    	    	myfile.close();
    	    }

        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -60);


        	start_depth = network_config->layer[2].first_depth_id - 0.1;
        	end_depth = network_config->layer[2].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
//    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63);
//    		cout<<"Changing param, start: "<< start_depth+32<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+64, end_depth, 5, -1.6);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+64, end_depth, 4, 0.16);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+64, end_depth, 0, -0.001);

        	start_depth = network_config->layer[3].first_depth_id - 0.1;
        	end_depth = network_config->layer[3].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.0);
//    		if(last_layer_teach)change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -64.0);

    		//training_time_each_img = training_time_each_img*1.3;
    		if(last_layer_teach){
    			myvector = seq_vector;
        		training_img_index = 0;
    			training_time_each_img = input_int;
    		}
    		hipDeviceSynchronize();
    	}else if(time==third_layer_time){

    	    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );
    	    ofstream myfile ((index_prefix+"third_stage_device2_spike_of_neuron_out.csv"));
    	    if (myfile.is_open()){
    	    	//myfile << "This is a new test\n";
    	    	for(int i=0; i < SIZE; i++){
    	    		//printf("_%f_", log_v_host[i]);
    	    		myfile << log_total_spike_host[i] << ", ";
    	    	}
    	    	myfile.close();
    	    }

        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		//change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -68.2);

//
        	start_depth = network_config->layer[3].first_depth_id - 0.1;
        	end_depth = network_config->layer[3].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
//    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63);
//    		cout<<"Changing param, start: "<< start_depth+32<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 5, -1.6);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 4, 0.16);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 0, -0.001);

        	start_depth = network_config->layer[4].first_depth_id - 0.1;
        	end_depth = network_config->layer[4].last_depth_id + 0.1;
//    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -60.0);

    		cout<<"Parameter Changing complete.\n";
    		hipDeviceSynchronize();
    		training_time_each_img = training_time_each_img*1.3;
    	}

    	if(time%tenpercent_iter == 0){
    		iter_log = clock();
    		cout<<to_string(10*(time/tenpercent_iter))<<"% done, time used is: " << (iter_log - iter_start)/1000 << " (ms)" << endl;
    	}
    	//fault below here:

    	if(time%training_time_each_img==0){//at the beginning of each img's training, load into
    		//cout<<"Image Load Iter: "<<time<<endl;
			int locate_index = myvector[training_img_index];
//			cout<<"loading index: "<<locate_index<<endl;
    		for(int i=0;i<input_neuron_num;i++){
    			one_mnist_img[i] = mnist_img[locate_index*input_neuron_num+i];
    		}
//    	    for (int y=0; y<28; ++y) {
//    	    	    for (int x=0; x<28; ++x) {
//    	    	      std::cout << ((one_mnist_img[y*28+x] <= 1.1)? ' ' : '*');
//    	    	      //std::cout << int(one_mnist_img[y*28+x]) << ' ';
//    	    	    }
//    	    	    std::cout << std::endl;
//    	    }
    		MNIST_drive(Neuron_list_device, Input_neuronlist_device, one_mnist_img, input_neuron_num, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
    		//MNIST_drive(old_device_neurons, one_mnist_img, network_size,training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
    		training_img_index ++;
    		bool one_iter=false;
    		if(training_img_index>training_set_number-1){


        		if(batch_load && batched_load_remain>0){
        			if (batched_load_remain>img_load_max){
        				img_load_offset += training_set_number;
        				training_set_number = img_load_max;
        			}else{
        				img_load_offset += training_set_number;
        				training_set_number = batched_load_remain;
        			}

    				batched_load_remain -= training_set_number;

    				if(batched_load_remain<=0){
    					training_set_number = img_load_max;
    					batched_load_remain = batch_load_grand_total - training_set_number;
    					img_load_offset = 0;
    				}
    				myvector.clear();
    			    for (int i=0; i<training_set_number; ++i)myvector.push_back(i); // 1 2 3 4 5 6 7 8 9
    				NTU_skeleton_read_image(image_file, mnist_img, training_set_number, img_load_offset);
    			    //DVS_read_image_8bit(image_file, mnist_img, training_set_number);
    			    MNIST_drive(NeuronList, Input_neuronlist, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, \
    			    		max_frequency, min_frequency, 1); //change to spike frequency
    			    cout<<"Next batch loaded, total number: "<<training_set_number<<", remaining data: "<<batched_load_remain<<endl;
        		}

    			training_img_index = 0;

    			if(shuffle_image) std::random_shuffle ( myvector.begin(), myvector.end() );
//    			one_iter = true;
    		}

    		if(last_layer_teach&&time>=second_layer_time){
	    		if (one_iter) {
	            	start_depth = network_config->layer[3].first_depth_id - 0.1;
	            	end_depth = network_config->layer[3].last_depth_id + 0.1;
	        		cout<<"One Iter ended. Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//	        		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -64.0);

	    			last_layer_teach = false;
	    		}
    			if(locate_index%frame_per_seq==0){
    				int class_index = locate_index/frame_per_class;

    	        	float start_depth = network_config->layer[3].first_depth_id - 0.1;
    	        	float end_depth = network_config->layer[3].last_depth_id + 0.1;
		    		reset_all_state<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);
//    	    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -61.0);
	        		change_state<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, 1);//change index[5] to 1, which means no potentiation



    	        	start_depth = network_config->layer[3].first_depth_id - 0.1+class_index;
    	        	end_depth = network_config->layer[3].first_depth_id + 0.1+class_index;

//    	    		cout<<"Changing threshold at training index: "<<locate_index<<" class index: "<<class_index<<", start: "<< start_depth<<" end: "<<end_depth<<endl;
//    	    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.0);

    	        	cout<<"Changing state at training index: "<<locate_index<<" class index: "<<class_index<<", start: "<< start_depth<<" end: "<<end_depth<<endl;
	        		change_state<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, 2);//change index[5] to 2, which means no depression


    			}

    		}
    		hipDeviceSynchronize();

    	}
    	//cout<<"One IMG loaded"<<endl;
    	//enter spiking neuron simulation:
    	int one_layer_neuron_num = 0;
    	if(time<first_layer_time){
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, true);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, \
							h_convolution_result, probe);
					synapse_drive_cnn_v2(Neuron_list_device, Input_neuronlist_device, network_config, d_network_config, d_filter_array, layer_iter, \
							spiking_neuron_num, input_neuron_num, syn_timer_max, connection_size, random_number_list_device, random_number_normal_device, states, -1.0, -1.0, log_total_spike);//STDP
				}
			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {
	    		lateral_inhibition_depth_wise_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, input_image_channel, input_image_channel+network_config->layer[1].depth, inhibition_time, d_network_config, log_spike, total_depth_number);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition){

	    	}
	    	else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 1, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}
    	}else if(time<second_layer_time){
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==2){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 4*input_float, time, true);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, \
							h_convolution_result, probe);
					synapse_drive_cnn_v2(Neuron_list_device, Input_neuronlist_device, network_config, d_network_config, d_filter_array, layer_iter, spiking_neuron_num, input_neuron_num, \
							syn_timer_max, connection_size, random_number_list_device, random_number_normal_device, states, -1.0, -1.0, log_total_spike);//STDP
				}

			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {
	    		lateral_inhibition_depth_wise_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, input_image_channel+network_config->layer[1].depth, input_image_channel+network_config->layer[1].depth+network_config->layer[2].depth, inhibition_time, d_network_config, log_spike, total_depth_number);
	    	}else if(forced_lateral_inhibition_at_last_layer && CNN_total_layer_num==3){//if this is the last layer, use lateral_inhibition
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition && CNN_total_layer_num!=3){

	    	}
	    	else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}

    	}else if(time<third_layer_time){
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==2){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, \
							h_convolution_result, probe);
				}else if(layer_iter==3){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 30*input_float, time, true);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, \
							h_convolution_result, probe);
					synapse_drive_cnn_v2(Neuron_list_device, Input_neuronlist_device, network_config, d_network_config, d_filter_array, layer_iter, spiking_neuron_num, input_neuron_num, \
							syn_timer_max, connection_size, random_number_list_device, random_number_normal_device, states, -1.0, -1.0, log_total_spike);//STDP
				}

			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {
	    		lateral_inhibition_depth_wise_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, input_image_channel+network_config->layer[1].depth+network_config->layer[2].depth, input_image_channel+network_config->layer[1].depth+network_config->layer[2].depth+network_config->layer[3].depth, inhibition_time, d_network_config, log_spike, total_depth_number);
	    	}else if(forced_lateral_inhibition_at_last_layer && CNN_total_layer_num==3){//if this is the last layer, use lateral_inhibition
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition && CNN_total_layer_num!=3){

	    	}
	    	else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}

    	}else{
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1 ){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 3*input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==2){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.5*input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==3){
					bool last_layer_inhib = !last_layer_teach;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 4*input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
					//synapse_drive_cnn_v2(Neuron_list_device, Input_neuronlist_device, network_config, d_network_config, d_filter_array, layer_iter, spiking_neuron_num, input_neuron_num, syn_timer_max, connection_size, random_number_list_device, random_number_normal_device, states, -1.0, -1.0, log_total_spike);//STDP
				}else if(layer_iter==4){
					bool last_layer_inhib = !last_layer_teach;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 15*input_float, time, true);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
					synapse_drive_cnn_v2(Neuron_list_device, Input_neuronlist_device, network_config, d_network_config, d_filter_array, layer_iter, spiking_neuron_num, input_neuron_num, syn_timer_max, connection_size, random_number_list_device, random_number_normal_device, states, -1.0, -1.0, log_total_spike);//STDP
				}

			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {
	    		lateral_inhibition_depth_wise_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, input_image_channel+network_config->layer[1].depth+network_config->layer[2].depth+network_config->layer[3].depth, input_image_channel+network_config->layer[1].depth+network_config->layer[2].depth+network_config->layer[3].depth+network_config->layer[4].depth, inhibition_time, d_network_config, log_spike, total_depth_number);
	    	}else if(forced_lateral_inhibition_at_last_layer && CNN_total_layer_num==4){//if this is the last layer, use lateral_inhibition
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition && CNN_total_layer_num!=3){

	    	}
	    	else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}

    	}
//    	printf("T: %d_",time);
//    	if(time==100) 		break;
    	time ++;
    }
    //spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, 2);
	//hipDeviceSynchronize();

	filter_util(network_config, Neuron_list_device, spiking_neuron_num, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 2);
    hipMemcpy(NeuronList,Neuron_list_device,spiking_neuron_num*sizeof(Neuron),hipMemcpyDeviceToHost);
    hipMemcpy(log_v_host,log_v,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(log_spike_host,log_spike,total_depth_number*sizeof(float),hipMemcpyDeviceToHost);
    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );


    //print out the synapse conductance data
    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2);

    ofstream myfile ((index_prefix+"device2_spike_of_neuron_out.csv"));
    if (myfile.is_open()){
    	//myfile << "This is a new test\n";
    	for(int i=0; i < SIZE; i++){
    		//printf("_%f_", log_v_host[i]);
    		myfile << log_total_spike_host[i] << ", ";
    	}
    	myfile.close();
    }

    ofstream myfile_p ((index_prefix+"probe.csv"));
    if (myfile_p.is_open()){
    	//myfile << "This is a new test\n";
    	for(int i=0; i < 1000; i++){
    		//printf("_%f_", log_v_host[i]);
    		myfile_p << probe[i] << ", ";
    	}
    	myfile_p.close();
    }

//
//    ofstream myfile_0 ((index_prefix+"device2_out_v.csv"));
//    if (myfile_0.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_0 << log_v_host[i] << ", ";
//    	}
//    	myfile.close();
//    }
//
//    ofstream myfile_2 ((index_prefix+"device2_spike_of_one.csv"));
//    if (myfile_2.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_2 << log_spike_host[i] << ", ";
//    	}
//    	myfile_2.close();
//    }



    hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
	filter_util(network_config, NeuronList, network_size, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 1);	//write filter to file
    write_neuron_list(NeuronList, (index_prefix+"device2_output_network.txt"), spiking_neuron_num);
    data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, "");
    //===clean up===
    //delete[] random_number_list;
    delete[] log_v_host;
	delete[] NeuronList;
	delete[] log_spike_host;
	delete[] log_total_spike_host;
	delete[] mnist_img;
	delete[] NeuronList_temp;
	delete[] one_mnist_img;
	delete[] probe;
//	delete[] random_number_list;
	delete[] mnist_label;
	delete[] spike_flag;
	delete[] num_one_digit_img;
	//hipFree(states);
	hipFree(log_v);
	hipFree(log_spike);
	hipFree(log_total_spike);
	hipFree(Neuron_list_device);
	//hipFree(old_device_neurons);
	hipFree(random_number_list_device);
	hipFree(d_network_config);
	hipFree(states);
	hipFree(spike_flag_device);
	hipFree(log_spike_default);

}



void run_autotune(string index_prefix, float input_float, float input_float_2, int input_int, int input_int_2, string input_img){
	//#undef shuffle_image
	//#define shuffle_image 1
	/*
	int training_set_number = 1;
	int size_per_img = input_image_w * input_image_l*input_image_channel;
	float *mnist_img = new float[size_per_img*training_set_number];
	string image_file = "train-images-idx3-ubyte"; //"train-images-idx3-ubyte";
	MNIST_read_image(image_file, mnist_img, training_set_number);
	int *mnist_label = new int[training_set_number];
	string image_label_file = "train-labels-idx1-ubyte";
	MNIST_read_label(image_label_file, mnist_label, training_set_number);

	float *filter;
	float *output = new float[size_per_img*training_set_number];

	convolution_kernel(mnist_img, filter, output);
	img_util(output, "test_output.jpg", 0);
	*/

	float learning_1st_layer_1st_layer_threshold;

	float learning_2nd_layer_1st_layer_threshold;
	float learning_2nd_layer_2nd_layer_threshold;

	float learning_3rd_layer_1st_layer_threshold;
	float learning_3rd_layer_2nd_layer_threshold;
	float learning_3rd_layer_3rd_layer_threshold;


	cout<<"Load input"<<endl;

	cout<<"learning_1st_layer_1st_layer_threshold";
	cin >> learning_1st_layer_1st_layer_threshold;
	cout<<"learning_2nd_layer_1st_layer_threshold";
	cin >> learning_2nd_layer_1st_layer_threshold;
	cout<<"learning_2nd_layer_2nd_layer_threshold";
	cin >> learning_2nd_layer_2nd_layer_threshold;

	cout<<"learning_3rd_layer_1st_layer_threshold";
	cin >> learning_3rd_layer_1st_layer_threshold;
	cout<<"learning_3rd_layer_2nd_layer_threshold";
	cin >> learning_3rd_layer_2nd_layer_threshold;
	cout<<"learning_3rd_layer_3rd_layer_threshold";
	cin >> learning_3rd_layer_3rd_layer_threshold;

	cout<<"loaded input: "<<learning_1st_layer_1st_layer_threshold<<' '<< learning_2nd_layer_1st_layer_threshold<<' '<<learning_2nd_layer_2nd_layer_threshold<<' ';
	cout<<learning_3rd_layer_1st_layer_threshold<<' '<<learning_3rd_layer_2nd_layer_threshold<<' '<<learning_3rd_layer_3rd_layer_threshold<<endl;
	cout<<"Input Loading Done\n"<<endl;

	int resume_learning = 0;
	CNN_struct *network_config = new CNN_struct;
	network_config_generator(3, network_config);
	Neuron *NeuronList_temp = new Neuron[1];
	CNN_struct *d_network_config;
	hipMalloc((void **)&d_network_config,sizeof(CNN_struct));
	hipMemcpy(d_network_config,network_config,sizeof(CNN_struct),hipMemcpyHostToDevice);
	int total_depth_number = 0;
	for(int i=0;i<CNN_total_layer_num; i++){
		total_depth_number = total_depth_number + network_config->layer[i].depth;
		cout<<"depth number: "<<network_config->layer[i].depth<<endl;
	}

	cout<<endl<<"Total depth number: "<<total_depth_number<<endl;
	float **h_filter_array;
	float **d_filter_array;
	int filter_array_size = CNN_total_layer_num-1;
	hipMalloc(&d_filter_array, filter_array_size*sizeof(float *));
	h_filter_array = (float**)malloc(filter_array_size * sizeof(float*));
	filter_util(network_config, NeuronList_temp, 0,0,  h_filter_array, d_filter_array, index_prefix, 0);

	/*
	img_util(mnist_img, "tensorflow_small.png", 1);
	img_util(mnist_img, "test_output_-1.png", 0);

	float *output = new float[size_per_img*training_set_number];

	int image_bytes = input_image_channel * input_image_l * input_image_w * sizeof(float);

	float* convolution_device_input{nullptr};
	hipMalloc(&convolution_device_input, image_bytes);
	hipMemcpy(convolution_device_input, mnist_img, image_bytes, hipMemcpyHostToDevice);

	int filter_in_channel = input_image_channel;
	int filter_out_channel = input_image_channel;
	int filter_height = 3;
	int filter_width = 3;
	const float kernel_template[3][3] = {
	{1, 1, 1},
	{1, -8, 1},
	{1, 1, 1}
	};
	float h_kernel[filter_in_channel][filter_out_channel][filter_height][filter_width];
	for (int kernel = 0; kernel < filter_in_channel; ++kernel) {
		for (int channel = 0; channel < filter_out_channel; ++channel) {
		  for (int row = 0; row < filter_height; ++row) {
			for (int column = 0; column < filter_width; ++column) {
			  h_kernel[kernel][channel][row][column] = kernel_template[row][column];
			}
		  }
		}
	}
	float* filter{nullptr};
	hipMalloc(&filter, sizeof(h_kernel));
	hipMemcpy(filter, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);


	convolution_kernel(convolution_device_input, filter, output);
	img_util(output, "test_output_1.png", 0);

	hipFree(filter);
	hipFree(convolution_device_input);

	*/

//===========END of CNN special setting-up phase============

	Convolution_setting_struct *convolution_settings = new Convolution_setting_struct[CNN_total_layer_num];

	//set parameters
	int training_set_number = 50000;
	int training_repeat_time = 2;

	int training_time_each_img = input_int;
	int calculated_total_time = training_time_each_img*training_set_number*training_repeat_time;
    if(resume_learning) calculated_total_time = calculated_total_time/3;

	#undef MAX_TIME
	#define MAX_TIME calculated_total_time
	printf("==Training Total Iter: %d==", MAX_TIME);
	int total_neuron_num = 0;
	int total_spiking_num = 0;
	for(int i=0;i<CNN_total_layer_num;i++){
		total_neuron_num += network_config->layer[i].neuron_num;
		if(i!=0) total_spiking_num += network_config->layer[i].neuron_num;
	}
	total_spiking_num += 5;
	total_neuron_num += 5;
	//total_neuron_num = 20000;
	cout<<endl<<"total neuron num: "<<total_neuron_num<<endl;
	cout<<"total spiking neuron num: "<<total_spiking_num<<endl;
	#undef SIZE
	#define SIZE total_neuron_num
	#undef SPIKING_NEURON_NUM
	#define SPIKING_NEURON_NUM total_spiking_num


	float max_frequency = 30; //in Hz default 22
	float min_frequency = 3;

	bool batch_load = false;
	int batched_load_remain = 0;
	int batch_load_grand_total = 0;
	int img_load_offset = 0;
	int img_load_max = 50000;

	if (training_set_number>img_load_max){ //manually set the maximum number of images to be loaded once is 60000
		cout<<"Using batch loading"<<endl;
		batch_load_grand_total = training_set_number;
		batch_load = true;
		batched_load_remain = training_set_number - img_load_max;
		training_set_number = img_load_max;
	}
	int input_neuron_num = input_image_w*input_image_l*input_image_channel;
	int input_image_signal_channel_size  = input_image_w*input_image_l;
	int spiking_neuron_num = SPIKING_NEURON_NUM;
	int output_layer_neuron_num = OUTPUT_LAYER_NEURON_NUM;
	int tenpercent_iter = MAX_TIME/10;
	int connection_size = MAX_CONNECTION;
	int syn_timer_max = 25;
	int input_signal_width = 10;	//default 25
	int inhibition_time = 10;	//default 10

	float target_frequency_param = 0.5;
	float target_frequency = 100;
	float *mnist_img = new float[input_neuron_num*training_set_number];
	for(int i=0;i<input_neuron_num*training_set_number;i++) mnist_img[i] = 0;
	string image_file = "dvs_gesture_1bit";//"./NTU_Skeleton/binary_file_2000";//"train_dataset_noisy_cifar";//"fashion-train-images-idx3-ubyte";//"train_dataset_noisy";//"train_dataset_noisy"; //"train-images-idx3-ubyte";
	cout<<endl<<"Image loading"<<endl;
	clock_t load_start = clock();
    std::vector<std::string> folder_list;
    int input_folder_cnt = 0;
	if(input_image_channel==1 || input_image_channel==2){
		//CIFAR_read_image(mnist_img, input_neuron_num, 0, 1);
		//GTVIR_read_image(mnist_img, input_neuron_num, training_set_number);
		//NTU_skeleton_read_image(image_file, mnist_img, training_set_number, img_load_offset);
		//DVS_read_image(image_file, mnist_img, training_set_number);
		MNIST_read_image("train-images-idx3-ubyte", mnist_img, training_set_number);
		//read_polygon("/inverse_polygon/drawings", mnist_img, training_set_number);
		//read_polygon("/rotating_mnist/readMNIST/readMNIST/training/", mnist_img, training_set_number);
		//MNIST_read_image("./rotating_f_mnist/rotating_mnist", mnist_img, training_set_number);
	}else{
		bool learn_imageNet = false;
		if(learn_imageNet){

			ifstream file ("imageNet_folder_list.csv");
			string val;

		    while(file.good()) {
				getline(file, val, ',');
		    	folder_list.push_back(val);
		    }

			imageNET_read_image(folder_list[input_folder_cnt], mnist_img, training_set_number);
		}else{
			CIFAR_read_image(mnist_img, input_neuron_num, training_set_number, 0, 0);
			//KAIST_PED_read_image("", mnist_img , training_set_number);
		}

	}
	clock_t load_end = clock();

	cout<<endl<<"Image loading done"<<", time used is " << (load_end - load_start)/1000 << " (ms)"<<endl;
	//CIFAR_read_image_one_channel(mnist_img, input_image_signal_channel_size, input_int_2, 0);
	//MNIST_read_image(image_file, mnist_img, training_set_number);
	int *mnist_label = new int[training_set_number];
	string image_label_file = "train-labels-idx1-ubyte";
	//CIFAR_read_label(mnist_label, 0);
	//MNIST_read_label(image_label_file, mnist_label, training_set_number);
	//special_function: learn one category
	printf("=0=\n");
	int learn_one_digit = 0;
	int *num_one_digit_img = new int[1];
	if(learn_one_digit){
		MNIST_labeling("abc", 60000, mnist_img, mnist_label, mnist_img, num_one_digit_img, spiking_neuron_num, 1, 5);
		printf("Learning only one digit, number of img: %d\n", num_one_digit_img);
	}

	//int synapse_size = SIZE*SIZE;
	//cout<<SIZE<<endl;
    Neuron *NeuronList = new Neuron[spiking_neuron_num];
    Input_neuron *Input_neuronlist = new Input_neuron[input_neuron_num];
	//unsigned char *synapse_timer = new unsigned char[synapse_size];  //this is the array that stores timer used in STPD. e.g Neuron x --->  Neuron y Spike! In the array index [(x-1)*SIZE+(y-1)]  => 1
	//hiprandState_t *states;
	//float *random_number_list = new float[SIZE];
	float *log_v_host = new float[MAX_TIME];
	float *log_spike_host = new float[total_depth_number];

	float *log_total_spike_host = new float[SIZE];
	for(int i=0; i < SIZE; i++){
		log_total_spike_host[i] = 0;
	}
	int *spike_flag = new int[CNN_total_layer_num];
	for(int i=0; i < CNN_total_layer_num; i++){
		spike_flag[i] = 0;
	}
	for(int i=0; i<total_depth_number; i++) log_spike_host[i] = 0;
	//init_log_v(log_v_host);
	//init_data_log(log_v_host,log_spike_host,log_total_spike_host, MAX_TIME);
	neuron_list_init(NeuronList, spiking_neuron_num);
	input_neuron_list_init(Input_neuronlist, input_neuron_num);
	printf("=1=\n");
	//
	if(resume_learning){
		printf("------RESUME LEARNING-------\n");
		read_neuron_list(NeuronList, 1, "spike_cnn.txt");
		//read_neuron_list(NeuronList, 1, "device2_output_network.txt");
		int start_layer = 3;//the layer that starts to learn
		read_neuron_list_special(NeuronList, (start_layer-1), network_config, "device2_output_network_org.txt"); //duplicate the previous layer
		bool do_reset_weight = true;
		if(do_reset_weight){

			float start_depth = network_config->layer[start_layer].first_depth_id - 0.1;
			float end_depth = network_config->layer[start_layer].last_depth_id + 0.1;

			reset_weight(NeuronList, start_depth, end_depth, 1, spiking_neuron_num);

		}
		//read_neuron_list(NeuronList, 1, "spike_cnn.txt");
	}else{
		read_neuron_list(NeuronList, 1, "spike_cnn.txt");
		//read_neuron_list(NeuronList, 1, "device2_output_network.txt");

	}
	//printf("read out one neuron depth: %f", NeuronList[116000].param[7]);
    //write_neuron_list(NeuronList, "learning_output_confirm.txt", SIZE);
	//check_neuron(NeuronList, 800, 820);


	//Neuron *old_device_neurons;
	//unsigned char *snapse_timer_device;
	float *log_v;
	float *log_spike;
	float *log_spike_default;
	float *log_total_spike;
	int *spike_flag_device;


    printf("2\n");
	//random number function:

    float rand_list_size_to_total_connection_ratio = 1;
	int rand_numb_size = SPIKING_NEURON_NUM*MAX_CONNECTION;

	int SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
    dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
	dim3 print_grid(1);
	dim3 print_block(1);
	dim3 dimBlock_unit( 1, 1 );
	dim3 dimGrid_unit(1, 1);

	int SIZE_PER_SIDE_whole_network = sqrt(spiking_neuron_num)+1;
    dim3 dimBlock_whole_network( ThreadsPerBlock*2, ThreadsPerBlock );
    dim3 dimGrid_whole_network( (SIZE_PER_SIDE_whole_network/dimBlock.x+1), (SIZE_PER_SIDE_whole_network/dimBlock.y+1));
    printf("2.1\n");

	hiprandState_t *states;

//	if (STOCHASTIC_STDP) rand_init<<<dimGrid,dimBlock>>>(time(0), rand_numb_size, states);
//	float *random_number_list = new float[rand_numb_size];
//	float *random_number_list_device;
//	SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
//	dim3 dimBlock_synapse( ThreadsPerBlock, ThreadsPerBlock );
//	dim3 dimGrid_synapse( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));

//	hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
//	hipMemcpy(random_number_list_device,random_number_list,rand_numb_size*sizeof(float),hipMemcpyHostToDevice);
//	if (STOCHASTIC_STDP) random<<<dimGrid_synapse,dimBlock_synapse>>>(random_number_list_device, rand_numb_size, states);

    hiprandGenerator_t gen_uniform;
    float *random_number_list_device;
    if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING || DEVICE_VARIATION){
    	hipMalloc((void **)&states, rand_numb_size * sizeof(hiprandState_t));
		hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
		hiprandCreateGenerator(&gen_uniform, HIPRAND_RNG_PSEUDO_DEFAULT);
		hiprandSetPseudoRandomGeneratorSeed(gen_uniform, time(0));
		hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);
    }


    hiprandGenerator_t gen_normal;
    float *random_number_normal_device;
    float normal_mean = 0;
    float normal_sd = 5.0;
    if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING || DEVICE_VARIATION){
		hipMalloc((void **)&random_number_normal_device,rand_numb_size*sizeof(float));
		hiprandCreateGenerator(&gen_normal, HIPRAND_RNG_PSEUDO_DEFAULT);
		hiprandSetPseudoRandomGeneratorSeed(gen_normal, time(0));
		hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);
    }

//    printf("2.11\n");
    //Setting up input instance matrix:
    float **d_input_instance;
    float **d_convolution_result;
    float **h_input_instance;
    float **h_convolution_result;
    float *probe = new float[1000];
	int instance_array_size = CNN_total_layer_num;
	hipMalloc(&d_input_instance, instance_array_size*sizeof(float *));
	int convolution_result_size = CNN_total_layer_num - 1;
	hipMalloc(&d_convolution_result, convolution_result_size*sizeof(float *));
    h_input_instance = (float**)malloc(instance_array_size * sizeof(float*));
    h_convolution_result = (float**)malloc(convolution_result_size * sizeof(float*));
    CNN_util(network_config, d_input_instance, d_convolution_result, h_input_instance, h_convolution_result, 0);
    printf("2.2\n");
//	float **add = &h_convolution_result[0];
//	printf("Address On GPU: %p\n", add);

    //========Setting up device neuron list============

	Neuron *Neuron_list_device;
    Input_neuron *Input_neuronlist_device;
    hipMalloc((void **)&Neuron_list_device, spiking_neuron_num*sizeof(Neuron));
    hipMalloc((void **)&Input_neuronlist_device, input_neuron_num*sizeof(Input_neuron));
    //hipMalloc((void **)&old_device_neurons, SIZE*sizeof(Neuron));

    //hipMalloc((void **)&states, SIZE * sizeof(hiprandState_t));
    hipMalloc((void **)&log_v, MAX_TIME * sizeof(float));
    hipMalloc((void **)&log_spike, total_depth_number * sizeof(float));
    hipMalloc((void **)&log_spike_default, total_depth_number * sizeof(float));
    //hipMalloc((void **)&log_total_spike, SIZE * sizeof(float));
    gpuErrchk( hipMalloc((void **)&log_total_spike, SIZE * sizeof(float)) );
    hipMalloc((void **)&spike_flag_device, instance_array_size*sizeof(int));
    //rand_init<<<dimGrid,dimBlock>>>(time(0), states);
    printf("2.3\n");
    hipMemcpy(Neuron_list_device,NeuronList,spiking_neuron_num*sizeof(Neuron),hipMemcpyHostToDevice);
    hipMemcpy(Input_neuronlist_device,Input_neuronlist,input_neuron_num*sizeof(Input_neuron),hipMemcpyHostToDevice);
    //hipMemcpy(old_device_neurons,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
    //hipMemcpy(random_number_list_device, random_number_list, SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(log_v,log_v_host,MAX_TIME*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike_default,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
    hipMemcpy(spike_flag_device,spike_flag,instance_array_size*sizeof(int),hipMemcpyHostToDevice);
    printf("3.0\n");
    //cout<<"network size: "<<SIZE<<endl;
    int network_size = SIZE;

    int max_time = MAX_TIME;
    int first_layer_time = max_time*1/6;
    int second_layer_time = max_time*1/3;
    int third_layer_time = max_time*2/3;
    if(CNN_total_layer_num==3){
    	first_layer_time = max_time*1/3;
    	second_layer_time = max_time + 1;
    	third_layer_time = max_time + 1;
    }
    else if (CNN_total_layer_num==2){
    	first_layer_time = max_time;
    	second_layer_time = max_time + 1;
    	third_layer_time = max_time + 1;
    }else if (CNN_total_layer_num==4){
    	first_layer_time = max_time*3/12;
    	second_layer_time = max_time*7/12;
    	third_layer_time = max_time + 1;
    }
    if(resume_learning){
    	first_layer_time = 1;
        if(CNN_total_layer_num==3) second_layer_time = max_time;
        if(CNN_total_layer_num==4) second_layer_time = 2;
        if(CNN_total_layer_num==4){
        	second_layer_time = 2;
        	third_layer_time = 3;
        }
    }

    //hipMemcpy(Neuron_list_device,old_device_neurons,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
    //first change raw img data into frequency
    int mnist_start_index = 0;
    int mnist_end_index = input_neuron_num;
    //change pixel signal to frequency

    MNIST_drive(NeuronList, Input_neuronlist, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, \
    		max_frequency, min_frequency, 1); //change to spike frequency
    std::srand ( unsigned ( std::time(0) ) );
    std::vector<int> myvector;
    for (int i=0; i<training_set_number; ++i) myvector.push_back(i); // 1 2 3 4 5 6 7 8 9

    if(shuffle_image){
    	  std::random_shuffle ( myvector.begin(), myvector.end() );
    }

    hipDeviceSynchronize();

    //data_check(Neuron_list_device,log_total_spike,SIZE,1);
    float *one_mnist_img = new float[input_neuron_num];

    clock_t iter_start, iter_log;
    iter_start = clock();
    int log_interval = MAX_TIME/10;
	bool enable_log_interval = false;
	bool last_layer_teach = false;

	int total_class = 10;
	int num_per_class = 500;
	int frame_per_seq = 50;

	int rotation_num = 2;
	int translation_num = 1;
	int frame_per_class = frame_per_seq*rotation_num*num_per_class;
	int frame_per_rotation = frame_per_seq;
	int frame_per_translation = frame_per_seq*rotation_num*num_per_class*total_class;

    std::vector<int> seq_vector_head;
    std::vector<int> seq_vector;
    for (int i=0; i<training_set_number/frame_per_seq; ++i) seq_vector_head.push_back(i); // 1 2 3 4 5 6 7 8 9
	std::random_shuffle ( seq_vector_head.begin(), seq_vector_head.end() );

	for (int i=0 ; i<training_set_number/frame_per_seq; ++i){
		int begin_index = seq_vector_head[i];
		for (int j=0; j<frame_per_seq; ++j) seq_vector.push_back(10*begin_index+j);

	}

    //read_filter_GPU_one_layer<<<1, 1>>>(d_network_config, h_filter_array[0], 1);
    //read_filter_GPU<<<1, 1>>>(d_network_config, d_filter_array);

//    int reiter_run = 1;

    int time = 0;
    int training_img_index = 0;

    //============now load all convolution settings===========
	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
		if (layer_iter==0) {
			cout<<endl<<"Setting up the first layer"<<endl;
			convolution_kernel_setup(convolution_settings, network_config, layer_iter);
		}else{

			if (layer_iter!=(CNN_total_layer_num-1))
			{
				convolution_kernel_setup(convolution_settings, network_config, layer_iter);
				cout<<"Setting up the"<<" layer "<< layer_iter <<endl;
			}
		}
	}
    if(resume_learning){
		copy_filter_to_cuDNN(Neuron_list_device, d_network_config, d_filter_array, spiking_neuron_num);
		hipDeviceSynchronize();
    }
	float start_depth = network_config->layer[1].first_depth_id - 0.1;
	float end_depth = network_config->layer[1].last_depth_id + 0.1;
	cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
	change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, learning_1st_layer_1st_layer_threshold);
	//return;
	while (time<max_time){

		//if(time==first_layer_time)MNIST_drive(NeuronList, Input_neuronlist, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, max_frequency*2, min_frequency, 1);
    	if(DEVICE_VARIATION){
            hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);
    	}
    	if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING){
    	    hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);
    	}

    	if(time%log_interval == 0 && enable_log_interval){
    	    hipMemcpy(NeuronList,Neuron_list_device,spiking_neuron_num*sizeof(Neuron),hipMemcpyDeviceToHost);
    		printf("NN data copy complete\n");
    		string interval_file_name = "device2_output_at_iter_" + to_string(time) + ".txt";
    		//string interval_weight_file_name = to_string(time);
    	    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, (to_string(time)+"_"));
    		if (time>0) {
    			write_neuron_list(NeuronList, interval_file_name, spiking_neuron_num);
    		    hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
    			filter_util(network_config, NeuronList, network_size, input_neuron_num, h_filter_array, d_filter_array, to_string(time), 1);	//write filter to file
    		}
    	}

    	if(time==first_layer_time){

    	    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );
    	    ofstream myfile ((index_prefix+"first_stage_device2_spike_of_neuron_out.csv"));
    	    if (myfile.is_open()){
    	    	//myfile << "This is a new test\n";
    	    	for(int i=0; i < SIZE; i++){
    	    		//printf("_%f_", log_v_host[i]);
    	    		myfile << log_total_spike_host[i] << ", ";
    	    	}
    	    	myfile.close();
    	    }

        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, learning_2nd_layer_1st_layer_threshold);
//    		cout<<"Changing param of long-term neuron, start: "<< start_depth+32<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+32, end_depth, 5, -1.6);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+32, end_depth, 4, 0.16);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+32, end_depth, 0, -0.001);
    		//change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+32, end_depth, -56.2);

        	start_depth = network_config->layer[2].first_depth_id - 0.1;
        	end_depth = network_config->layer[2].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, learning_2nd_layer_2nd_layer_threshold);


    		//training_time_each_img = training_time_each_img*1.1;
    		hipDeviceSynchronize();
    	}else if(time==second_layer_time){

    	    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );
    	    ofstream myfile ((index_prefix+"second_stage_device2_spike_of_neuron_out.csv"));
    	    if (myfile.is_open()){
    	    	//myfile << "This is a new test\n";
    	    	for(int i=0; i < SIZE; i++){
    	    		//printf("_%f_", log_v_host[i]);
    	    		myfile << log_total_spike_host[i] << ", ";
    	    	}
    	    	myfile.close();
    	    }

        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, learning_3rd_layer_1st_layer_threshold);


        	start_depth = network_config->layer[2].first_depth_id - 0.1;
        	end_depth = network_config->layer[2].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, learning_3rd_layer_2nd_layer_threshold);
//    		cout<<"Changing param, start: "<< start_depth+32<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+64, end_depth, 5, -1.6);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+64, end_depth, 4, 0.16);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+64, end_depth, 0, -0.001);

        	start_depth = network_config->layer[3].first_depth_id - 0.1;
        	end_depth = network_config->layer[3].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, learning_3rd_layer_3rd_layer_threshold);
//    		if(last_layer_teach)change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -64.0);

    		//training_time_each_img = training_time_each_img*1.6;
    		if(last_layer_teach){
    			myvector = seq_vector;
        		training_img_index = 0;
    			training_time_each_img = input_int;
    		}
    		hipDeviceSynchronize();
    	}else if(time==third_layer_time){
        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -68.2);


        	start_depth = network_config->layer[3].first_depth_id - 0.1;
        	end_depth = network_config->layer[3].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -71.2);
    		cout<<"Changing param, start: "<< start_depth+32<<" end: "<<end_depth<<endl;
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 5, -1.6);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 4, 0.16);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 0, -0.001);

        	start_depth = network_config->layer[4].first_depth_id - 0.1;
        	end_depth = network_config->layer[4].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -71.0);

    		cout<<"Parameter Changing complete.\n";
    		training_time_each_img = training_time_each_img*1.3;
    		hipDeviceSynchronize();
    	}

    	if(time%tenpercent_iter == 0){
    		iter_log = clock();
    		cout<<to_string(10*(time/tenpercent_iter))<<"% done, time used is: " << (iter_log - iter_start)/1000 << " (ms)" << endl;
    	}
    	//fault below here:

    	if(time%training_time_each_img==0){//at the beginning of each img's training, load into
    		//cout<<"Image Load Iter: "<<time<<endl;




			int locate_index = myvector[training_img_index];
//			cout<<"loading index: "<<locate_index<<endl;
    		for(int i=0;i<input_neuron_num;i++){
    			one_mnist_img[i] = mnist_img[locate_index*input_neuron_num+i];
    		}
//    	    for (int y=0; y<28; ++y) {
//    	    	    for (int x=0; x<28; ++x) {
//    	    	      std::cout << ((one_mnist_img[y*28+x] <= 1.1)? ' ' : '*');
//    	    	      //std::cout << int(one_mnist_img[y*28+x]) << ' ';
//    	    	    }
//    	    	    std::cout << std::endl;
//    	    }
    		MNIST_drive(Neuron_list_device, Input_neuronlist_device, one_mnist_img, input_neuron_num, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
    		//MNIST_drive(old_device_neurons, one_mnist_img, network_size,training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
    		training_img_index ++;
    		bool one_iter=false;
    		if(training_img_index>training_set_number-1){


        		if(batch_load && batched_load_remain>0){
        			if (batched_load_remain>img_load_max){
        				img_load_offset += training_set_number;
        				training_set_number = img_load_max;
        			}else{
        				img_load_offset += training_set_number;
        				training_set_number = batched_load_remain;
        			}

    				batched_load_remain -= training_set_number;

    				if(batched_load_remain<=0){
    					training_set_number = img_load_max;
    					batched_load_remain = batch_load_grand_total - training_set_number;
    					img_load_offset = 0;
    				}
    				myvector.clear();
    			    for (int i=0; i<training_set_number; ++i)myvector.push_back(i); // 1 2 3 4 5 6 7 8 9
    				//NTU_skeleton_read_image(image_file, mnist_img, training_set_number, img_load_offset);
    			    DVS_read_image_8bit(image_file, mnist_img, training_set_number);
    			    MNIST_drive(NeuronList, Input_neuronlist, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, \
    			    		max_frequency, min_frequency, 1); //change to spike frequency
    			    cout<<"Next batch loaded, total number: "<<training_set_number<<", remaining data: "<<batched_load_remain<<endl;
        		}

    			training_img_index = 0;

    			if(shuffle_image) std::random_shuffle ( myvector.begin(), myvector.end() );
//    			one_iter = true;
    		}

    		if(last_layer_teach&&time>=second_layer_time){
	    		if (one_iter) {
	            	start_depth = network_config->layer[3].first_depth_id - 0.1;
	            	end_depth = network_config->layer[3].last_depth_id + 0.1;
	        		cout<<"One Iter ended. Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//	        		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -64.0);

	    			last_layer_teach = false;
	    		}
    			if(locate_index%frame_per_seq==0){
    				int class_index = locate_index/frame_per_class;

    	        	float start_depth = network_config->layer[3].first_depth_id - 0.1;
    	        	float end_depth = network_config->layer[3].last_depth_id + 0.1;
		    		reset_all_state<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);
//    	    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -61.0);
	        		change_state<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, 1);//change index[5] to 1, which means no potentiation



    	        	start_depth = network_config->layer[3].first_depth_id - 0.1+class_index;
    	        	end_depth = network_config->layer[3].first_depth_id + 0.1+class_index;

//    	    		cout<<"Changing threshold at training index: "<<locate_index<<" class index: "<<class_index<<", start: "<< start_depth<<" end: "<<end_depth<<endl;
//    	    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.0);

    	        	cout<<"Changing state at training index: "<<locate_index<<" class index: "<<class_index<<", start: "<< start_depth<<" end: "<<end_depth<<endl;
	        		change_state<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, 2);//change index[5] to 2, which means no depression


    			}

    		}
    		hipDeviceSynchronize();

    	}
    	//cout<<"One IMG loaded"<<endl;
    	//enter spiking neuron simulation:
    	int one_layer_neuron_num = 0;
    	if(time<first_layer_time){
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, true);
					synapse_drive_cnn_v2(Neuron_list_device, Input_neuronlist_device, network_config, d_network_config, d_filter_array, layer_iter, \
							spiking_neuron_num, input_neuron_num, syn_timer_max, connection_size, random_number_list_device, random_number_normal_device, states, -1.0, -1.0, log_total_spike);//STDP
				}
			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {
	    		lateral_inhibition_depth_wise_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, input_image_channel, input_image_channel+network_config->layer[1].depth, inhibition_time, d_network_config, log_spike, total_depth_number);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition){

	    	}
	    	else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 1, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}
    	}else if(time<second_layer_time){
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==2){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 4*input_float, time, true);
					synapse_drive_cnn_v2(Neuron_list_device, Input_neuronlist_device, network_config, d_network_config, d_filter_array, layer_iter, spiking_neuron_num, input_neuron_num, \
							syn_timer_max, connection_size, random_number_list_device, random_number_normal_device, states, -1.0, -1.0, log_total_spike);//STDP
				}

			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {
	    		lateral_inhibition_depth_wise_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, input_image_channel+network_config->layer[1].depth, input_image_channel+network_config->layer[1].depth+network_config->layer[2].depth, inhibition_time, d_network_config, log_spike, total_depth_number);
	    	}else if(forced_lateral_inhibition_at_last_layer && CNN_total_layer_num==3){//if this is the last layer, use lateral_inhibition
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition && CNN_total_layer_num!=3){

	    	}
	    	else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}

    	}else if(time<third_layer_time){
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==2){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, \
							h_convolution_result, probe);
				}else if(layer_iter==3){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 35*input_float, time, true);
					synapse_drive_cnn_v2(Neuron_list_device, Input_neuronlist_device, network_config, d_network_config, d_filter_array, layer_iter, spiking_neuron_num, input_neuron_num, \
							syn_timer_max, connection_size, random_number_list_device, random_number_normal_device, states, -1.0, -1.0, log_total_spike);//STDP
				}

			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {
	    		lateral_inhibition_depth_wise_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, input_image_channel+network_config->layer[1].depth+network_config->layer[2].depth, input_image_channel+network_config->layer[1].depth+network_config->layer[2].depth+network_config->layer[3].depth, inhibition_time, d_network_config, log_spike, total_depth_number);
	    	}else if(forced_lateral_inhibition_at_last_layer && CNN_total_layer_num==3){//if this is the last layer, use lateral_inhibition
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition && CNN_total_layer_num!=3){

	    	}
	    	else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}

    	}else{
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1 ){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.5*input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==2){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.3*input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==3){
					bool last_layer_inhib = !last_layer_teach;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 1*input_float, time, true);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
					synapse_drive_cnn_v2(Neuron_list_device, Input_neuronlist_device, network_config, d_network_config, d_filter_array, layer_iter, spiking_neuron_num, input_neuron_num, syn_timer_max, connection_size, random_number_list_device, random_number_normal_device, states, -1.0, -1.0, log_total_spike);//STDP
				}

			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {
	    		lateral_inhibition_depth_wise_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, input_image_channel+network_config->layer[1].depth+network_config->layer[2].depth+network_config->layer[3].depth, input_image_channel+network_config->layer[1].depth+network_config->layer[2].depth+network_config->layer[3].depth+network_config->layer[4].depth, inhibition_time, d_network_config, log_spike, total_depth_number);
	    	}else if(forced_lateral_inhibition_at_last_layer && CNN_total_layer_num==4){//if this is the last layer, use lateral_inhibition
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition && CNN_total_layer_num!=3){

	    	}
	    	else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}

    	}
    	time ++;
    }
    //spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, 2);
	//hipDeviceSynchronize();

	filter_util(network_config, Neuron_list_device, spiking_neuron_num, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 2);
    hipMemcpy(NeuronList,Neuron_list_device,spiking_neuron_num*sizeof(Neuron),hipMemcpyDeviceToHost);
    hipMemcpy(log_v_host,log_v,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(log_spike_host,log_spike,total_depth_number*sizeof(float),hipMemcpyDeviceToHost);
    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );


    //print out the synapse conductance data
    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2);

    ofstream myfile ((index_prefix+"device2_spike_of_neuron_out.csv"));
    if (myfile.is_open()){
    	//myfile << "This is a new test\n";
    	for(int i=0; i < SIZE; i++){
    		//printf("_%f_", log_v_host[i]);
    		myfile << log_total_spike_host[i] << ", ";
    	}
    	myfile.close();
    }

    ofstream myfile_p ((index_prefix+"probe.csv"));
    if (myfile_p.is_open()){
    	//myfile << "This is a new test\n";
    	for(int i=0; i < 1000; i++){
    		//printf("_%f_", log_v_host[i]);
    		myfile_p << probe[i] << ", ";
    	}
    	myfile_p.close();
    }

//
//    ofstream myfile_0 ((index_prefix+"device2_out_v.csv"));
//    if (myfile_0.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_0 << log_v_host[i] << ", ";
//    	}
//    	myfile.close();
//    }
//
//    ofstream myfile_2 ((index_prefix+"device2_spike_of_one.csv"));
//    if (myfile_2.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_2 << log_spike_host[i] << ", ";
//    	}
//    	myfile_2.close();
//    }



    hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
	filter_util(network_config, NeuronList, network_size, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 1);	//write filter to file
    write_neuron_list(NeuronList, (index_prefix+"device2_output_network.txt"), spiking_neuron_num);
    data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, "");
    //===clean up===
    //delete[] random_number_list;
    delete[] log_v_host;
	delete[] NeuronList;
	delete[] log_spike_host;
	delete[] log_total_spike_host;
	delete[] mnist_img;
	delete[] NeuronList_temp;
	delete[] one_mnist_img;
	delete[] probe;
//	delete[] random_number_list;
	delete[] mnist_label;
	delete[] spike_flag;
	delete[] num_one_digit_img;
	//hipFree(states);
	hipFree(log_v);
	hipFree(log_spike);
	hipFree(log_total_spike);
	hipFree(Neuron_list_device);
	//hipFree(old_device_neurons);
	hipFree(random_number_list_device);
	hipFree(d_network_config);
	hipFree(states);
	hipFree(spike_flag_device);
	hipFree(log_spike_default);

}

void run_time_sequence(string index_prefix, float input_float, float input_float_2, int input_int, int input_int_2, string input_img){

	int resume_learning = 0;
	CNN_struct *network_config = new CNN_struct;
	network_config_generator(3, network_config);
	Neuron *NeuronList_temp = new Neuron[1];
	CNN_struct *d_network_config;
	hipMalloc((void **)&d_network_config,sizeof(CNN_struct));
	hipMemcpy(d_network_config,network_config,sizeof(CNN_struct),hipMemcpyHostToDevice);
	int total_depth_number = 0;
	for(int i=0;i<CNN_total_layer_num; i++){
		total_depth_number = total_depth_number + network_config->layer[i].depth;
		cout<<"depth number: "<<network_config->layer[i].depth<<endl;
	}

	cout<<endl<<"Total depth number: "<<total_depth_number<<endl;
	float **h_filter_array;
	float **d_filter_array;
	int filter_array_size = CNN_total_layer_num-1;
	hipMalloc(&d_filter_array, filter_array_size*sizeof(float *));
	h_filter_array = (float**)malloc(filter_array_size * sizeof(float*));
	filter_util(network_config, NeuronList_temp, 0,0,  h_filter_array, d_filter_array, index_prefix, 0);

//===========END of CNN special setting-up phase============

	Convolution_setting_struct *convolution_settings = new Convolution_setting_struct[CNN_total_layer_num];

	namespace fs = boost::filesystem;
	string cur_dir = "/home/xshe6/Documents/sc2data/28k/clustered/"+index_prefix.substr(0,3);
	fs::path Path(cur_dir);
	fs::directory_iterator end_iter;
	int total_game_cnt = 0;
	for (fs::directory_iterator iter(Path); iter != end_iter; ++iter){
		if(iter->path().extension() == ".csv"){
			total_game_cnt ++;
		}
	}
	cout<<endl<<"Total game in folder: "<<total_game_cnt<<endl;
	//set parameters
	int seq_length = 1000;
	int repeat_time = 1;
	int training_set_number = total_game_cnt;
	int training_time_each_img = input_int;
	int calculated_total_time = training_time_each_img*seq_length*training_set_number*repeat_time;
	#undef MAX_TIME
	#define MAX_TIME calculated_total_time
	printf("==Training Total Iter: %d==", MAX_TIME);
	int total_neuron_num = 0;
	int total_spiking_num = 0;
	for(int i=0;i<CNN_total_layer_num;i++){
		total_neuron_num += network_config->layer[i].neuron_num;
		if(i!=0)
		total_spiking_num += network_config->layer[i].neuron_num;
	}
	total_spiking_num += 10;
	total_neuron_num += 10;
	//total_neuron_num = 20000;
	cout<<endl<<"total neuron num: "<<total_neuron_num<<endl;
	cout<<"total spiking neuron num: "<<total_spiking_num<<endl;
	#undef SIZE
	#define SIZE total_neuron_num
	#undef SPIKING_NEURON_NUM
	#define SPIKING_NEURON_NUM total_spiking_num


	float max_frequency = 22; //in Hz default 22
	float min_frequency = 1;

	int input_neuron_num = input_image_w*input_image_l*input_image_channel;
	int input_image_signal_channel_size  = input_image_w*input_image_l;
	int spiking_neuron_num = SPIKING_NEURON_NUM;
	int output_layer_neuron_num = OUTPUT_LAYER_NEURON_NUM;
	int tenpercent_iter = total_game_cnt/10;
	int connection_size = MAX_CONNECTION;
	int syn_timer_max = 25;
	int input_signal_width = 10;	//default 25
	int inhibition_time = 10;	//default 10

	float target_frequency_param = 0.5;
	float target_frequency = 100;


	float *mnist_img = new float[seq_length*training_set_number];
	for(int mi=0; mi<seq_length*training_set_number; mi++) mnist_img[mi] = (rand()%100);
	//for(int i=0;i<input_neuron_num*training_set_number;i++) mnist_img[i] = 0;
	string image_file = "train-images-idx3-ubyte";//"train_dataset_noisy_cifar";//"fashion-train-images-idx3-ubyte";//"train_dataset_noisy";//"train_dataset_noisy"; //"train-images-idx3-ubyte";
	cout<<endl<<"Image loading"<<endl;
	if(input_image_channel==1){
		//CIFAR_read_image(mnist_img, input_neuron_num, 0, 1);
		//GTVIR_read_image(mnist_img, input_neuron_num, training_set_number);
		//MNIST_read_image(image_file, mnist_img, training_set_number);
		read_sine_seq("train_sine.csv", mnist_img, training_set_number);

	}else if(input_image_channel==3){
		CIFAR_read_image(mnist_img, input_neuron_num, training_set_number, 0, 0);
		//KAIST_PED_read_image("", mnist_img , training_set_number);

	}


	//cout<<endl<<"Image loading done"<<endl;
	//CIFAR_read_image_one_channel(mnist_img, input_image_signal_channel_size, input_int_2, 0);
	//MNIST_read_image(image_file, mnist_img, training_set_number);
	int *mnist_label = new int[training_set_number];
	string image_label_file = "train-labels-idx1-ubyte";
	//CIFAR_read_label(mnist_label, 0);
	//MNIST_read_label(image_label_file, mnist_label, training_set_number);
	//special_function: learn one category
	printf("=0=\n");

	//int synapse_size = SIZE*SIZE;
	//cout<<SIZE<<endl;
    Neuron *NeuronList = new Neuron[spiking_neuron_num];
    Input_neuron *Input_neuronlist = new Input_neuron[input_neuron_num];
	//unsigned char *synapse_timer = new unsigned char[synapse_size];  //this is the array that stores timer used in STPD. e.g Neuron x --->  Neuron y Spike! In the array index [(x-1)*SIZE+(y-1)]  => 1
	//hiprandState_t *states;
	//float *random_number_list = new float[SIZE];
	float *log_v_host = new float[MAX_TIME];
	float *log_spike_host = new float[total_depth_number];

	float *log_total_spike_host = new float[SIZE];
	for(int i=0; i < SIZE; i++){
		log_total_spike_host[i] = 0;
	}
	int *spike_flag = new int[CNN_total_layer_num];
	for(int i=0; i < CNN_total_layer_num; i++){
		spike_flag[i] = 0;
	}
	for(int i=0; i<total_depth_number; i++) log_spike_host[i] = 0;
	//init_log_v(log_v_host);
	//init_data_log(log_v_host,log_spike_host,log_total_spike_host, MAX_TIME);
	neuron_list_init(NeuronList, spiking_neuron_num);
	input_neuron_list_init(Input_neuronlist, input_neuron_num);
	printf("=1=\n");
	//
	if(resume_learning){
		printf("RESUME LEARNING\n");
		read_neuron_list(NeuronList, 1, "device2_output_network.txt");
	}else{
		read_neuron_list(NeuronList, 1, "spike_cnn.txt");
	}
    //write_neuron_list(NeuronList, "learning_output_confirm.txt", SIZE);
	//check_neuron(NeuronList, 800, 820);


	//Neuron *old_device_neurons;
	//unsigned char *snapse_timer_device;
	float *log_v;
	float *log_spike;
	float *log_spike_default;
	float *log_total_spike;
	int *spike_flag_device;


    printf("2\n");
	//random number function:

    float rand_list_size_to_total_connection_ratio = 1;
	int rand_numb_size = SPIKING_NEURON_NUM*MAX_CONNECTION;

	int SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
    dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
	dim3 print_grid(1);
	dim3 print_block(1);
	dim3 dimBlock_unit( 1, 1 );
	dim3 dimGrid_unit(1, 1);
    printf("2.1\n");

	hiprandState_t *states;
	hipMalloc((void **)&states, rand_numb_size * sizeof(hiprandState_t));

    hiprandGenerator_t gen_uniform;
    float *random_number_list_device;
    hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
    hiprandCreateGenerator(&gen_uniform, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen_uniform, time(0));
    hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);


    hiprandGenerator_t gen_normal;
    float *random_number_normal_device;
    float normal_mean = 0;
    float normal_sd = 5.0;
    hipMalloc((void **)&random_number_normal_device,rand_numb_size*sizeof(float));
    hiprandCreateGenerator(&gen_normal, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen_normal, time(0));
    hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);


//    printf("2.11\n");
    //Setting up input instance matrix:
    float **d_input_instance;
    float **d_convolution_result;
    float **h_input_instance;
    float **h_convolution_result;

    float *probe = new float[1000];
	int instance_array_size = CNN_total_layer_num;
	hipMalloc(&d_input_instance, instance_array_size*sizeof(float *));
	int convolution_result_size = CNN_total_layer_num - 1;
	hipMalloc(&d_convolution_result, convolution_result_size*sizeof(float *));
    h_input_instance = (float**)malloc(instance_array_size * sizeof(float*));
    h_convolution_result = (float**)malloc(convolution_result_size * sizeof(float*));
    CNN_util(network_config, d_input_instance, d_convolution_result, h_input_instance, h_convolution_result, 0);
    printf("2.2\n");
//	float **add = &h_convolution_result[0];
//	printf("Address On GPU: %p\n", add);

    //========Setting up device neuron list============

	Neuron *Neuron_list_device;
    Input_neuron *Input_neuronlist_device;
    hipMalloc((void **)&Neuron_list_device, spiking_neuron_num*sizeof(Neuron));
    hipMalloc((void **)&Input_neuronlist_device, input_neuron_num*sizeof(Input_neuron));
    //hipMalloc((void **)&old_device_neurons, SIZE*sizeof(Neuron));

    //hipMalloc((void **)&states, SIZE * sizeof(hiprandState_t));
    hipMalloc((void **)&log_v, MAX_TIME * sizeof(float));
    hipMalloc((void **)&log_spike, total_depth_number * sizeof(float));
    hipMalloc((void **)&log_spike_default, total_depth_number * sizeof(float));
    //hipMalloc((void **)&log_total_spike, SIZE * sizeof(float));
    gpuErrchk( hipMalloc((void **)&log_total_spike, SIZE * sizeof(float)) );
    hipMalloc((void **)&spike_flag_device, instance_array_size*sizeof(int));
    //rand_init<<<dimGrid,dimBlock>>>(time(0), states);
    printf("2.3\n");
    hipMemcpy(Neuron_list_device,NeuronList,spiking_neuron_num*sizeof(Neuron),hipMemcpyHostToDevice);
    hipMemcpy(Input_neuronlist_device,Input_neuronlist,input_neuron_num*sizeof(Input_neuron),hipMemcpyHostToDevice);
    //hipMemcpy(old_device_neurons,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
    //hipMemcpy(random_number_list_device, random_number_list, SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(log_v,log_v_host,MAX_TIME*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike_default,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
    hipMemcpy(spike_flag_device,spike_flag,instance_array_size*sizeof(int),hipMemcpyHostToDevice);
    printf("3\n");
    //cout<<"network size: "<<SIZE<<endl;
    int network_size = SIZE;

    int max_time = MAX_TIME;


    //hipMemcpy(Neuron_list_device,old_device_neurons,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
    //first change raw img data into frequency
    int mnist_start_index = 0;
    int mnist_end_index = input_neuron_num;
    //change pixel signal to frequency

    std::srand ( unsigned ( std::time(0) ) );
    std::vector<int> myvector;
    for (int i=0; i<training_set_number; ++i) myvector.push_back(i); // 1 2 3 4 5 6 7 8 9

    if(shuffle_image){
    	  std::random_shuffle ( myvector.begin(), myvector.end() );
    }

    hipDeviceSynchronize();

    //data_check(Neuron_list_device,log_total_spike,SIZE,1);
    float *one_mnist_img = new float[seq_length*input_image_channel];
	float *MNIST_stimulus_freq_device;
	int signal_size = seq_length*input_image_channel;
	hipMalloc((void **)&MNIST_stimulus_freq_device, signal_size*sizeof(float));


    clock_t iter_start, iter_log;
    iter_start = clock();
    int log_interval = total_game_cnt/10;

    int reiter_run = 1;

    int time = 0;
    int training_img_index = 0;

    //============now load all convolution settings===========
	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
		if (layer_iter==0) {
			convolution_kernel_setup(convolution_settings, network_config, layer_iter);
		}else{
			if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel_setup(convolution_settings, network_config, layer_iter);
		}
	}
	bool enable_log_interval = 0;
    bool teaching_mod = True;
	fs::directory_iterator iter(Path);// iter != end_iter; ++iter)

    for (int img_iter=0; img_iter<training_set_number*repeat_time; img_iter++){//at the beginning of each img's training, load into
    		string game_log_path = iter->path().string();
//    		cout<<"reading: "<<game_log_path<<endl;
    		read_sc2(game_log_path, one_mnist_img, seq_length*input_image_channel);
    		hipMemcpy(MNIST_stimulus_freq_device, one_mnist_img,signal_size*sizeof(float),hipMemcpyHostToDevice);
			int locate_index = myvector[training_img_index];
    		for(int i=0;i<seq_length*input_image_channel;i++){
    			//cout<<one_mnist_img[i]<<" ";
    		}
    		MNIST_drive(Neuron_list_device, Input_neuronlist_device, one_mnist_img, input_neuron_num, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 2, 0);

    		if(training_img_index>=training_set_number-1){
    			fs::directory_iterator iter(Path);
    			training_img_index = 0;
    		}
    		//cout<<endl<<"at img: "<<img_iter<<endl;
    		for (int game_time_iter=1; game_time_iter<seq_length; game_time_iter++){

        		//cout<<" "<<pixel_iter;
    			int input_start_index = (game_time_iter-1)*input_image_channel;
    			int teaching_start_index =  (game_time_iter)*input_image_channel;
    			if(one_mnist_img[teaching_start_index]<0) break;
    			int teaching_target;
//    			if (input_start_index>118) break;
    			for (int action_iter=0; action_iter<input_image_channel; action_iter++){
//    				cout<<"index "<<teaching_start_index+action_iter<<" value is: "<<one_mnist_img[teaching_start_index+action_iter]<<" ";
    				if(one_mnist_img[teaching_start_index+action_iter]>one_mnist_img[input_start_index+action_iter]){
    					teaching_target=action_iter;
//    					cout<<endl<<"teaching target: "<<teaching_target<<" old_value: "<<one_mnist_img[input_start_index+action_iter]<<" start_value: "<<one_mnist_img[teaching_start_index+action_iter]<<endl;
    					break;
    				}
    			}


//    			if(teaching_target>input_target){
//    				teaching_target = 1;
//    			}else{
//    				teaching_target = 0;
//    			}

    			MNIST_drive(Neuron_list_device, Input_neuronlist_device, MNIST_stimulus_freq_device, input_neuron_num, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 1, input_start_index);

    			//for debugging
//    			MNIST_drive(Neuron_list_device, Input_neuronlist_device, MNIST_stimulus_freq_device, input_neuron_num, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 4, 0);
    			//

    			if(!teaching_mod){
    				//cout<<(game_time_iter)<<" ";
//    				if(!(game_time_iter==input_image_w*input_image_l)) continue;
    			}

    			for (int learn_timer=0; learn_timer<training_time_each_img; learn_timer++){
    		    	if(DEVICE_VARIATION){
    		            hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);
    		    	}
    		    	if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING){
    		    	    hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);
    		    	}


    				for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
    		    		int convolution_result_index = layer_iter - 1;
    		    		if (layer_iter==0) {
    		    			convolution_result_index = 0;
    		    			spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, \
    		    					d_input_instance, layer_iter, network_size, input_neuron_num, \
    		    					log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, 0, teaching_mod);
    		    			if(!teaching_mod)convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
    		    		}else{
    		    			spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
    		    					layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, teaching_target, teaching_mod);
    		    			//if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
    						synapse_drive_cnn_v2(Neuron_list_device, Input_neuronlist_device, network_config, d_network_config, d_filter_array, layer_iter, spiking_neuron_num, input_neuron_num, syn_timer_max, \
    								connection_size, random_number_list_device, random_number_normal_device, states, -1.0, -1.0, log_total_spike);//STDP
    		    		}

    		    	}

    	    		if(!teaching_mod) lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 1, inhibition_time, d_network_config, spike_flag_device);
    		    	time ++;
    		}
    	}
//    	cout<<endl<<"====="<<endl<<endl;
    	++iter;
    	training_img_index ++;


    	if(training_img_index%tenpercent_iter == 0){
    		iter_log = clock();
    		cout<<to_string(10*(training_img_index/tenpercent_iter))<<"% done, time used is: " << (iter_log - iter_start)/1000 << " (ms)" << endl;
    	}

    	if(training_img_index%log_interval == 0 && enable_log_interval){
    		hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
    		printf("NN data copy complete\n");
    		string interval_file_name = "device2_output_at_iter_" + to_string(time) + ".txt";
    	    data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, (to_string(time)+"_"));
    		//if (time>0) write_neuron_list(NeuronList, interval_file_name, network_size);
    	}
    }


    //spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, 2);
	//hipDeviceSynchronize();

	filter_util(network_config, Neuron_list_device, spiking_neuron_num, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 2);
    hipMemcpy(NeuronList,Neuron_list_device,spiking_neuron_num*sizeof(Neuron),hipMemcpyDeviceToHost);
    hipMemcpy(log_v_host,log_v,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(log_spike_host,log_spike,total_depth_number*sizeof(float),hipMemcpyDeviceToHost);
    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );


    //print out the synapse conductance data
    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2);

    ofstream myfile ((index_prefix+"device2_spike_of_neuron_out.csv"));
    if (myfile.is_open()){
    	//myfile << "This is a new test\n";
    	for(int i=0; i < SIZE; i++){
    		//printf("_%f_", log_v_host[i]);
    		myfile << log_total_spike_host[i] << ", ";
    	}
    	myfile.close();
    }

    ofstream myfile_p ((index_prefix+"probe.csv"));
    if (myfile_p.is_open()){
    	//myfile << "This is a new test\n";
    	for(int i=0; i < 1000; i++){
    		//printf("_%f_", log_v_host[i]);
    		myfile_p << probe[i] << ", ";
    	}
    	myfile_p.close();
    }



    //hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
	filter_util(network_config, NeuronList, network_size, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 1);	//write filter to file
    write_neuron_list(NeuronList, (index_prefix+"device2_output_network.txt"), spiking_neuron_num);
    data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, "");
    //===clean up===
    //delete[] random_number_list;
    delete[] log_v_host;
	delete[] NeuronList;
	delete[] log_spike_host;
	delete[] log_total_spike_host;
	delete[] mnist_img;
	delete[] NeuronList_temp;
	delete[] one_mnist_img;
	delete[] probe;
//	delete[] random_number_list;
	delete[] mnist_label;
	delete[] spike_flag;

	//hipFree(states);
	hipFree(log_v);
	hipFree(log_spike);
	hipFree(log_total_spike);
	hipFree(Neuron_list_device);
	//hipFree(old_device_neurons);
	hipFree(random_number_list_device);
	hipFree(d_network_config);
	hipFree(states);
	hipFree(spike_flag_device);
	hipFree(log_spike_default);
}

void run_sc2(string index_prefix, float input_float, float input_float_2, int input_int, int input_int_2, string input_img){
	/*
	int training_set_number = 1;
	int size_per_img = input_image_w * input_image_l*input_image_channel;
	float *mnist_img = new float[size_per_img*training_set_number];
	string image_file = "train-images-idx3-ubyte"; //"train-images-idx3-ubyte";
	MNIST_read_image(image_file, mnist_img, training_set_number);
	int *mnist_label = new int[training_set_number];
	string image_label_file = "train-labels-idx1-ubyte";
	MNIST_read_label(image_label_file, mnist_label, training_set_number);

	float *filter;
	float *output = new float[size_per_img*training_set_number];

	convolution_kernel(mnist_img, filter, output);
	img_util(output, "test_output.jpg", 0);
	*/
	int resume_learning = 0;
	CNN_struct *network_config = new CNN_struct;
	network_config_generator(3, network_config);
	Neuron *NeuronList_temp = new Neuron[1];
	CNN_struct *d_network_config;
	hipMalloc((void **)&d_network_config,sizeof(CNN_struct));
	hipMemcpy(d_network_config,network_config,sizeof(CNN_struct),hipMemcpyHostToDevice);
	int total_depth_number = 0;
	for(int i=0;i<CNN_total_layer_num; i++){
		total_depth_number = total_depth_number + network_config->layer[i].depth;
		cout<<"depth number: "<<network_config->layer[i].depth<<endl;
	}

	cout<<endl<<"Total depth number: "<<total_depth_number<<endl;
	float **h_filter_array;
	float **d_filter_array;
	int filter_array_size = CNN_total_layer_num-1;
	hipMalloc(&d_filter_array, filter_array_size*sizeof(float *));
	h_filter_array = (float**)malloc(filter_array_size * sizeof(float*));
	filter_util(network_config, NeuronList_temp, 0,0,  h_filter_array, d_filter_array, index_prefix, 0);

	/*
	img_util(mnist_img, "tensorflow_small.png", 1);
	img_util(mnist_img, "test_output_-1.png", 0);

	float *output = new float[size_per_img*training_set_number];

	int image_bytes = input_image_channel * input_image_l * input_image_w * sizeof(float);

	float* convolution_device_input{nullptr};
	hipMalloc(&convolution_device_input, image_bytes);
	hipMemcpy(convolution_device_input, mnist_img, image_bytes, hipMemcpyHostToDevice);

	int filter_in_channel = input_image_channel;
	int filter_out_channel = input_image_channel;
	int filter_height = 3;
	int filter_width = 3;
	const float kernel_template[3][3] = {
	{1, 1, 1},
	{1, -8, 1},
	{1, 1, 1}
	};
	float h_kernel[filter_in_channel][filter_out_channel][filter_height][filter_width];
	for (int kernel = 0; kernel < filter_in_channel; ++kernel) {
		for (int channel = 0; channel < filter_out_channel; ++channel) {
		  for (int row = 0; row < filter_height; ++row) {
			for (int column = 0; column < filter_width; ++column) {
			  h_kernel[kernel][channel][row][column] = kernel_template[row][column];
			}
		  }
		}
	}
	float* filter{nullptr};
	hipMalloc(&filter, sizeof(h_kernel));
	hipMemcpy(filter, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);


	convolution_kernel(convolution_device_input, filter, output);
	img_util(output, "test_output_1.png", 0);

	hipFree(filter);
	hipFree(convolution_device_input);

	*/

//===========END of CNN special setting-up phase============

	Convolution_setting_struct *convolution_settings = new Convolution_setting_struct[CNN_total_layer_num];

	namespace fs = boost::filesystem;
	string cur_dir = "/home/xshe6/Documents/sc2data/28k/clustered/"+index_prefix.substr(0,3);  //changed format only displays building at one time step, not the accumulated list
	fs::path Path(cur_dir);
	fs::directory_iterator end_iter;
	int total_game_cnt = 0;
	for (fs::directory_iterator iter(Path); iter != end_iter; ++iter){
		if(iter->path().extension() == ".csv"){
			total_game_cnt ++;
		}
	}
	cout<<endl<<"Total game in folder: "<<total_game_cnt<<endl;
	//set parameters
	int repeat_time = 10;
	int training_set_number = total_game_cnt;
	int training_time_each_img = input_int;
	int calculated_total_time = training_time_each_img*training_set_number*repeat_time;


	#undef MAX_TIME
	#define MAX_TIME calculated_total_time
	printf("==Training Total Iter: %d==", MAX_TIME);
	int total_neuron_num = 0;
	int total_spiking_num = 0;
	for(int i=0;i<CNN_total_layer_num;i++){
		total_neuron_num += network_config->layer[i].neuron_num;
		if(i!=0)
		total_spiking_num += network_config->layer[i].neuron_num;
	}
	total_spiking_num += 10;
	total_neuron_num += 10;
	//total_neuron_num = 20000;
	cout<<endl<<"total neuron num: "<<total_neuron_num<<endl;
	cout<<"total spiking neuron num: "<<total_spiking_num<<endl;
	#undef SIZE
	#define SIZE total_neuron_num
	#undef SPIKING_NEURON_NUM
	#define SPIKING_NEURON_NUM total_spiking_num


	float max_frequency = 100; //in Hz default 22
	float min_frequency = 1;

	int training_folder_number = 800;
	int input_neuron_num = input_image_w*input_image_l*input_image_channel;
	int input_image_signal_channel_size  = input_image_w*input_image_l;
	int spiking_neuron_num = SPIKING_NEURON_NUM;
	int output_layer_neuron_num = OUTPUT_LAYER_NEURON_NUM;
	int tenpercent_iter = MAX_TIME/10;
	int connection_size = MAX_CONNECTION;
	int syn_timer_max = 25;
	int input_signal_width = 3;	//default 25
	int inhibition_time = 10;	//default 10

	float target_frequency_param = 0.5;
	float target_frequency = 100;
	float *mnist_img = new float[input_neuron_num*training_set_number];
//	for(int mi=0; mi<input_neuron_num*training_set_number; mi++) mnist_img[mi] = (rand()%100);
	//for(int i=0;i<input_neuron_num*training_set_number;i++) mnist_img[i] = 0;
	string image_file = "train-images-idx3-ubyte";//"train_dataset_noisy_cifar";//"fashion-train-images-idx3-ubyte";//"train_dataset_noisy";//"train_dataset_noisy"; //"train-images-idx3-ubyte";
	cout<<endl<<"Image loading"<<endl;
	clock_t load_start = clock();
    std::vector<std::string> folder_list;
    int input_folder_cnt = 0;
	bool learn_imageNet = false;

	clock_t load_end = clock();

	cout<<endl<<"Image loading done"<<", time used is " << (load_end - load_start)/1000 << " (ms)"<<endl;

	//CIFAR_read_image_one_channel(mnist_img, input_image_signal_channel_size, input_int_2, 0);
	//MNIST_read_image(image_file, mnist_img, training_set_number);
	int *mnist_label = new int[training_set_number];
	string image_label_file = "train-labels-idx1-ubyte";
	//CIFAR_read_label(mnist_label, 0);
	//MNIST_read_label(image_label_file, mnist_label, training_set_number);
	//special_function: learn one category
	printf("=0=\n");
	int learn_one_digit = 0;
	int *num_one_digit_img = new int[1];
	if(learn_one_digit){
		MNIST_labeling("abc", 60000, mnist_img, mnist_label, mnist_img, num_one_digit_img, spiking_neuron_num, 1, 5);
		//printf("Learning only one digit, number of img: %d\n", num_one_digit_img[0]);
	}

	//int synapse_size = SIZE*SIZE;
	//cout<<SIZE<<endl;
    Neuron *NeuronList = new Neuron[spiking_neuron_num];
    Input_neuron *Input_neuronlist = new Input_neuron[input_neuron_num];
	//unsigned char *synapse_timer = new unsigned char[synapse_size];  //this is the array that stores timer used in STPD. e.g Neuron x --->  Neuron y Spike! In the array index [(x-1)*SIZE+(y-1)]  => 1
	//hiprandState_t *states;
	//float *random_number_list = new float[SIZE];
	float *log_v_host = new float[MAX_TIME];
	float *log_spike_host = new float[total_depth_number];

	float *log_total_spike_host = new float[SIZE];
	for(int i=0; i < SIZE; i++){
		log_total_spike_host[i] = 0;
	}
	int *spike_flag = new int[CNN_total_layer_num];
	for(int i=0; i < CNN_total_layer_num; i++){
		spike_flag[i] = 0;
	}
	for(int i=0; i<total_depth_number; i++) log_spike_host[i] = 0;
	//init_log_v(log_v_host);
	//init_data_log(log_v_host,log_spike_host,log_total_spike_host, MAX_TIME);
	neuron_list_init(NeuronList, spiking_neuron_num);
	input_neuron_list_init(Input_neuronlist, input_neuron_num);
	printf("=1=\n");
	//
	if(resume_learning){
		printf("RESUME LEARNING\n");
		read_neuron_list(NeuronList, 1, "device2_output_network.txt");
	}else{
		read_neuron_list(NeuronList, 1, "spike_cnn.txt");
	}
    //write_neuron_list(NeuronList, "learning_output_confirm.txt", SIZE);
	//check_neuron(NeuronList, 800, 820);


	//Neuron *old_device_neurons;
	//unsigned char *snapse_timer_device;
	float *log_v;
	float *log_spike;
	float *log_spike_default;
	float *log_total_spike;
	int *spike_flag_device;


    printf("2\n");
	//random number function:

    float rand_list_size_to_total_connection_ratio = 1;
	int rand_numb_size = SPIKING_NEURON_NUM*MAX_CONNECTION;

	int SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
    dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
	dim3 print_grid(1);
	dim3 print_block(1);
	dim3 dimBlock_unit( 1, 1 );
	dim3 dimGrid_unit(1, 1);
    printf("2.1\n");

	hiprandState_t *states;
	hipMalloc((void **)&states, rand_numb_size * sizeof(hiprandState_t));
//	if (STOCHASTIC_STDP) rand_init<<<dimGrid,dimBlock>>>(time(0), rand_numb_size, states);
//	float *random_number_list = new float[rand_numb_size];
//	float *random_number_list_device;
//	SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
//	dim3 dimBlock_synapse( ThreadsPerBlock, ThreadsPerBlock );
//	dim3 dimGrid_synapse( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));

//	hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
//	hipMemcpy(random_number_list_device,random_number_list,rand_numb_size*sizeof(float),hipMemcpyHostToDevice);
//	if (STOCHASTIC_STDP) random<<<dimGrid_synapse,dimBlock_synapse>>>(random_number_list_device, rand_numb_size, states);

    hiprandGenerator_t gen_uniform;
    float *random_number_list_device;
    hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
    hiprandCreateGenerator(&gen_uniform, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen_uniform, time(0));
    hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);


    hiprandGenerator_t gen_normal;
    float *random_number_normal_device;
    float normal_mean = 0;
    float normal_sd = 5.0;
    hipMalloc((void **)&random_number_normal_device,rand_numb_size*sizeof(float));
    hiprandCreateGenerator(&gen_normal, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen_normal, time(0));
    hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);


//    printf("2.11\n");
    //Setting up input instance matrix:
    float **d_input_instance;
    float **d_convolution_result;
    float **h_input_instance;
    float **h_convolution_result;

    float *probe = new float[1000];
	int instance_array_size = CNN_total_layer_num;
	hipMalloc(&d_input_instance, instance_array_size*sizeof(float *));
	int convolution_result_size = CNN_total_layer_num - 1;
	hipMalloc(&d_convolution_result, convolution_result_size*sizeof(float *));
    h_input_instance = (float**)malloc(instance_array_size * sizeof(float*));
    h_convolution_result = (float**)malloc(convolution_result_size * sizeof(float*));
    CNN_util(network_config, d_input_instance, d_convolution_result, h_input_instance, h_convolution_result, 0);
    printf("2.2\n");
//	float **add = &h_convolution_result[0];
//	printf("Address On GPU: %p\n", add);

    //========Setting up device neuron list============

	Neuron *Neuron_list_device;
    Input_neuron *Input_neuronlist_device;
    hipMalloc((void **)&Neuron_list_device, spiking_neuron_num*sizeof(Neuron));
    hipMalloc((void **)&Input_neuronlist_device, input_neuron_num*sizeof(Input_neuron));
    //hipMalloc((void **)&old_device_neurons, SIZE*sizeof(Neuron));

    //hipMalloc((void **)&states, SIZE * sizeof(hiprandState_t));
    hipMalloc((void **)&log_v, MAX_TIME * sizeof(float));
    hipMalloc((void **)&log_spike, total_depth_number * sizeof(float));
    hipMalloc((void **)&log_spike_default, total_depth_number * sizeof(float));
    //hipMalloc((void **)&log_total_spike, SIZE * sizeof(float));
    gpuErrchk( hipMalloc((void **)&log_total_spike, SIZE * sizeof(float)) );
    hipMalloc((void **)&spike_flag_device, instance_array_size*sizeof(int));
    //rand_init<<<dimGrid,dimBlock>>>(time(0), states);
    printf("2.3\n");
    hipMemcpy(Neuron_list_device,NeuronList,spiking_neuron_num*sizeof(Neuron),hipMemcpyHostToDevice);
    hipMemcpy(Input_neuronlist_device,Input_neuronlist,input_neuron_num*sizeof(Input_neuron),hipMemcpyHostToDevice);
    //hipMemcpy(old_device_neurons,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
    //hipMemcpy(random_number_list_device, random_number_list, SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(log_v,log_v_host,MAX_TIME*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike_default,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
    hipMemcpy(spike_flag_device,spike_flag,instance_array_size*sizeof(int),hipMemcpyHostToDevice);
    printf("3\n");
    //cout<<"network size: "<<SIZE<<endl;
    int network_size = SIZE;

    int max_time = MAX_TIME;


    //hipMemcpy(Neuron_list_device,old_device_neurons,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
    //first change raw img data into frequency
    int mnist_start_index = 0;
    int mnist_end_index = input_neuron_num;
    //change pixel signal to frequency

    MNIST_drive(NeuronList, Input_neuronlist, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 1);
    std::srand ( unsigned ( std::time(0) ) );
    std::vector<int> myvector;
    for (int i=0; i<training_set_number; ++i) myvector.push_back(i); // 1 2 3 4 5 6 7 8 9

    if(shuffle_image){
    	  std::random_shuffle ( myvector.begin(), myvector.end() );
    }

    hipDeviceSynchronize();

    //data_check(Neuron_list_device,log_total_spike,SIZE,1);
    float *one_mnist_img = new float[input_neuron_num];

    clock_t iter_start, iter_log;
    iter_start = clock();
    int log_interval = MAX_TIME/10;
    //read_filter_GPU_one_layer<<<1, 1>>>(d_network_config, h_filter_array[0], 1);
    //read_filter_GPU<<<1, 1>>>(d_network_config, d_filter_array);

    int reiter_run = 1;

    int time = 0;
    int training_img_index = 0;

    //============now load all convolution settings===========
	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
		if (layer_iter==0) {
			convolution_kernel_setup(convolution_settings, network_config, layer_iter);
		}else{
			if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel_setup(convolution_settings, network_config, layer_iter);
		}
	}
	bool enable_log_interval = false;

	fs::directory_iterator iter(Path);// iter != end_iter; ++iter)
	int game_cnt = 0;
	while (time<max_time){
    	//cout<<endl<<" It: "<<time<<endl;
        //random<<<dimGrid,dimBlock>>>(random_number_list_device, states);
    	//first create an array of 1 MNIST image
//    	if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING){
//            random<<<dimGrid_synapse,dimBlock_synapse>>>(random_number_list_device, rand_numb_size, states);
//    	}

    	if(DEVICE_VARIATION){
            hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);
    	}
    	if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING){
    	    hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);
    	}

    	if(time%log_interval == 0 && enable_log_interval){
    		hipMemcpy(NeuronList,Neuron_list_device,SIZE*sizeof(Neuron),hipMemcpyDeviceToHost);
    		printf("NN data copy complete\n");
    		string interval_file_name = "device2_output_at_iter_" + to_string(time) + ".txt";
    		//data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, (to_string(time)+"_"));
    		//if (time>0) write_neuron_list(NeuronList, interval_file_name, network_size);
    	}

    	if(time%tenpercent_iter == 0){
    		iter_log = clock();
    		cout<<to_string(10*(time/tenpercent_iter))<<"% done, time used is: " << (iter_log - iter_start)/1000 << " (ms)" << endl;
    	}
    	//fault below here:

    	if(time%training_time_each_img==0){//at the beginning of each img's training, load into
    		//cout<<"Image Load Iter: "<<time<<endl;

    		if(game_cnt==total_game_cnt){
    			game_cnt = 0;
    			iter = fs::directory_iterator(Path);
    		}

    		string game_log_path = iter->path().string();
    		//cout<<"Reading: "<<game_log_path<<endl;
			delete[] one_mnist_img;
			one_mnist_img = new float[input_neuron_num];
			//cout<<"loading index: "<<locate_index<<endl;
    		read_sc2_3(game_log_path, one_mnist_img, input_neuron_num);
        	++iter;
        	game_cnt++;
    		//for(int iii=0; iii<input_neuron_num; iii++) cout<<one_mnist_img[iii]<<" ";
    		MNIST_drive(Neuron_list_device, Input_neuronlist_device, one_mnist_img, input_neuron_num, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
    		//MNIST_drive(old_device_neurons, one_mnist_img, network_size,training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
    		training_img_index ++;
    		if(training_img_index>=training_set_number-1){
    			if(learn_imageNet&&(input_folder_cnt<training_folder_number-1)){
    				input_folder_cnt++;
    				imageNET_read_image(folder_list[input_folder_cnt], mnist_img, training_set_number);
    				if(input_folder_cnt==training_folder_number-1) input_folder_cnt = 0;
    			}
    	    	if(shuffle_image) std::random_shuffle ( myvector.begin(), myvector.end() );
    			training_img_index = 0;
    		}

    	}
    	//cout<<"One IMG loaded"<<endl;

    	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
    		int convolution_result_index = layer_iter - 1;
    		if (layer_iter==0) {
    			convolution_result_index = 0;
    			spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
    					layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
    			convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
    		}else{
    			spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
    					layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, true);
    			if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				synapse_drive_cnn_v2(Neuron_list_device, Input_neuronlist_device, network_config, d_network_config, d_filter_array, layer_iter, spiking_neuron_num, input_neuron_num, syn_timer_max, connection_size, random_number_list_device, random_number_normal_device, states, -1.0, -1.0, log_total_spike);//STDP
    		}

    	}
    	//=================TRY WITH LAYER wise inhibition=====================
    	//hipDeviceSynchronize();
    	if(depth_wise_inhibition) {
    		//lateral_inhibition_depth_wise_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, input_image_channel, inhibition_time, d_network_config, log_spike, total_depth_number);
    	}else{
    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 1, inhibition_time, d_network_config, spike_flag_device);
    	}
//    	hipMemcpy(spike_flag, spike_flag_device, CNN_total_layer_num*sizeof(int),hipMemcpyDeviceToHost);
//    	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
//    		//if (layer_iter==0) printf("Layer[%d] SpikeFlag: %d\n", layer_iter, spike_flag[layer_iter]);
//            if(spike_flag[layer_iter]>0){//use lateral inhibition
//            	spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, layer_iter, network_config, 4);
//            }
//    		spike_flag[layer_iter] = 0;
//    	}
//    	hipMemcpy(spike_flag_device,spike_flag,CNN_total_layer_num*sizeof(int),hipMemcpyHostToDevice);
//    	hipMemcpy(log_spike,log_spike_default,total_depth_number*sizeof(float),hipMemcpyDeviceToDevice);	//set the log_spike to default value

	//=================TRY WITH NO LAYERAL INHIBITION, MAY BE WRONG=====================
    	//printf("network_size: %d", network_size);
    	//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 3); //lateral inhibition
	//==================================================================================
		if(HOMEOSTASIS_ENABLE){
			if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
				//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
			}
		}
        //hipDeviceSynchronize();


        //if any neuron spikes, run inhibition
	//hipMemcpy(spike_flag, spike_flag_device, CNN_total_layer_num*sizeof(int),hipMemcpyDeviceToHost);
	//printf("AtTime:%d_spike_flag_is:%d\n",time,spike_flag[0]);
        //if(spike_flag[0]>0){//use lateral inhibition
        	//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0);
        	//spike_flag[0] = 0;
    	    	//hipMemcpy(spike_flag_device,spike_flag,sizeof(int),hipMemcpyHostToDevice);
        //}

        //hipMemcpy(old_device_neurons,Neuron_list_device,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
        //hipDeviceSynchronize();
    	time ++;
    }
    //spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, 2);
	//hipDeviceSynchronize();

	filter_util(network_config, Neuron_list_device, SIZE, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 2);
    hipMemcpy(NeuronList,Neuron_list_device,spiking_neuron_num*sizeof(Neuron),hipMemcpyDeviceToHost);
    hipMemcpy(log_v_host,log_v,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(log_spike_host,log_spike,total_depth_number*sizeof(float),hipMemcpyDeviceToHost);
    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );


    //print out the synapse conductance data
    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2);

    ofstream myfile ((index_prefix+"device2_spike_of_neuron_out.csv"));
    if (myfile.is_open()){
    	//myfile << "This is a new test\n";
    	for(int i=0; i < SIZE; i++){
    		//printf("_%f_", log_v_host[i]);
    		myfile << log_total_spike_host[i] << ", ";
    	}
    	myfile.close();
    }

    ofstream myfile_p ((index_prefix+"probe.csv"));
    if (myfile_p.is_open()){
    	//myfile << "This is a new test\n";
    	for(int i=0; i < 1000; i++){
    		//printf("_%f_", log_v_host[i]);
    		myfile_p << probe[i] << ", ";
    	}
    	myfile_p.close();
    }

//
//    ofstream myfile_0 ((index_prefix+"device2_out_v.csv"));
//    if (myfile_0.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_0 << log_v_host[i] << ", ";
//    	}
//    	myfile.close();
//    }
//
//    ofstream myfile_2 ((index_prefix+"device2_spike_of_one.csv"));
//    if (myfile_2.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_2 << log_spike_host[i] << ", ";
//    	}
//    	myfile_2.close();
//    }



    //hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
	filter_util(network_config, NeuronList, network_size, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 1);	//write filter to file
    write_neuron_list(NeuronList, (index_prefix+"device2_output_network.txt"), spiking_neuron_num);
    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, "");
    //===clean up===
    //delete[] random_number_list;
    delete[] log_v_host;
	delete[] NeuronList;
	delete[] log_spike_host;
	delete[] log_total_spike_host;
	delete[] mnist_img;
	delete[] NeuronList_temp;
	delete[] one_mnist_img;
	delete[] probe;
//	delete[] random_number_list;
	delete[] mnist_label;
	delete[] spike_flag;
	delete[] num_one_digit_img;
	//hipFree(states);
	hipFree(log_v);
	hipFree(log_spike);
	hipFree(log_total_spike);
	hipFree(Neuron_list_device);
	//hipFree(old_device_neurons);
	hipFree(random_number_list_device);
	hipFree(random_number_normal_device);
	hipFree(d_network_config);
	hipFree(states);
	hipFree(spike_flag_device);
	hipFree(log_spike_default);
	hipFree(d_filter_array);
	hipFree(d_input_instance);
	hipFree(d_convolution_result);
	hipFree(Input_neuronlist_device);
}

