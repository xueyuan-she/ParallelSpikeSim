#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <vector>
#include <string>
#include "header.h"
#include <stdlib.h>
#include <streambuf>
#include <sstream>
#include <fstream>
#include <math.h>
#include "CImg.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include "cifar10_reader.hpp"
#include <boost/filesystem.hpp>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      //if (abort) exit(code);
   }
}


#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return;}} while(0)

long read_label_file_NTU(string image_file, vector<int> &label, vector<long> &frame_num){

	cout<<"reading labels from file: "<<image_file<<endl;
	ifstream file (image_file.c_str());
	if(!file.is_open()) throw std::runtime_error("Could not open file");
	string val, word;
	int line_cnt = 0;
	//int one_col;
	bool flag = true;
	while(getline(file, val)){

		std::stringstream ss(val);
		while(getline(ss, word, ',')) {
			if (flag){
				label.push_back(stoi(word));
				flag = false;
			}else{
				frame_num.push_back(stoi(word));
				flag = true;
				if(line_cnt>0 && frame_num[line_cnt]==frame_num[line_cnt-1]) cout<<"Warning: Duplicate Frame Nums";
				line_cnt ++ ;


			}


//			cout<<stoi(word)<<' ';
		}

		//cout<<val<<endl;
		//std::stringstream ss(val);
		//if(val.at(val.length()-1)=='\n') cout<<"gg";
		//cout<<val[0]<<'~';
		//cout<<stof(val);
//		while (ss>>one_col){
//		if(val=='\n'){
//			line_cnt ++;
//			continue;
//		}

//		if (flag){
//			label[line_cnt] = stoi(val);
//			flag = false;
//		}else{
//			frame_num[line_cnt] = stoi(val);
//			flag = true;
//		}

//
//			if(ss.peek()==',') ss.ignore();
//		}
//		line_cnt ++;
		//cout<<image[i]<<" ";
	}

	bool check_content = false;
	if(check_content){
		cout<<"Check read label: "<<endl;
		for(int i=0; i<line_cnt; i++){
			cout<<label[i]<<", "<<frame_num[i]<<endl;
		}

	}

	file.close();
	cout<<"A total of "<<line_cnt<<" lines of labels are read"<<endl;

	return frame_num[line_cnt-1];
}





void run_cnn_multilayer_inference(string index_prefix, float input_float, float input_float_2, int input_int, int input_int_2, string input_img){
	cout << "Running CNN Multilayer Inference" << endl << endl;
	cout<<"Functions: \n"<<"0. One image inference\n"<<"1. load a folder\n"<<"2. One folder separate run\n" \
			<< "3. For NTU, read a label file\n";
	cout << endl;
	cout<<"Function Select: ";
	int mode_select;
	cin >> mode_select;

	switch (mode_select){
		case 0:
		{
			cout<<"One image inference selected"<<endl;
			cout<<endl;
		}
		break;
		case 1:
		{
			printf("Case 1 selected");
			cout<<"How many iterations for each image:";
			cin>>input_int;
		}
		break;
		case 2:
		{
			printf("Case 2 selected");
			cout<<"How many iterations for each image:";
			cin>>input_int;
		}
		break;
		case 3:
		{
			printf("Case 3 selected");
			cout<<"How many iterations for each image:";
			cin>>input_int;
		}
		break;
	}



	/*
	int training_set_number = 1;
	int size_per_img = input_image_w * input_image_l*input_image_channel;
	float *mnist_img = new float[size_per_img*training_set_number];
	string image_file = "train-images-idx3-ubyte"; //"train-images-idx3-ubyte";
	MNIST_read_image(image_file, mnist_img, training_set_number);
	int *mnist_label = new int[training_set_number];
	string image_label_file = "train-labels-idx1-ubyte";
	MNIST_read_label(image_label_file, mnist_label, training_set_number);

	float *filter;
	float *output = new float[size_per_img*training_set_number];

	convolution_kernel(mnist_img, filter, output);
	img_util(output, "test_output.jpg", 0);
	*/
	int resume_learning = 0;
	CNN_struct *network_config = new CNN_struct;
	network_config_generator(3, network_config);
	Neuron *NeuronList_temp = new Neuron[1];
	CNN_struct *d_network_config;
	hipMalloc((void **)&d_network_config,sizeof(CNN_struct));
	hipMemcpy(d_network_config,network_config,sizeof(CNN_struct),hipMemcpyHostToDevice);
	int total_depth_number = 0;
	for(int i=0;i<CNN_total_layer_num; i++){
		total_depth_number = total_depth_number + network_config->layer[i].depth;
		cout<<"depth number: "<<network_config->layer[i].depth<<endl;
	}

	cout<<endl<<"Total depth number: "<<total_depth_number<<endl;
	float **h_filter_array;
	float **d_filter_array;
	int filter_array_size = CNN_total_layer_num-1;
	hipMalloc(&d_filter_array, filter_array_size*sizeof(float *));
	h_filter_array = (float**)malloc(filter_array_size * sizeof(float*));
	filter_util(network_config, NeuronList_temp, 0,0,  h_filter_array, d_filter_array, index_prefix, 0);

	/*
	img_util(mnist_img, "tensorflow_small.png", 1);
	img_util(mnist_img, "test_output_-1.png", 0);

	float *output = new float[size_per_img*training_set_number];

	int image_bytes = input_image_channel * input_image_l * input_image_w * sizeof(float);

	float* convolution_device_input{nullptr};
	hipMalloc(&convolution_device_input, image_bytes);
	hipMemcpy(convolution_device_input, mnist_img, image_bytes, hipMemcpyHostToDevice);

	int filter_in_channel = input_image_channel;
	int filter_out_channel = input_image_channel;
	int filter_height = 3;
	int filter_width = 3;
	const float kernel_template[3][3] = {
	{1, 1, 1},
	{1, -8, 1},
	{1, 1, 1}
	};
	float h_kernel[filter_in_channel][filter_out_channel][filter_height][filter_width];
	for (int kernel = 0; kernel < filter_in_channel; ++kernel) {
		for (int channel = 0; channel < filter_out_channel; ++channel) {
		  for (int row = 0; row < filter_height; ++row) {
			for (int column = 0; column < filter_width; ++column) {
			  h_kernel[kernel][channel][row][column] = kernel_template[row][column];
			}
		  }
		}
	}
	float* filter{nullptr};
	hipMalloc(&filter, sizeof(h_kernel));
	hipMemcpy(filter, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);


	convolution_kernel(convolution_device_input, filter, output);
	img_util(output, "test_output_1.png", 0);

	hipFree(filter);
	hipFree(convolution_device_input);

	*/

//===========END of CNN special setting-up phase============

	Convolution_setting_struct *convolution_settings = new Convolution_setting_struct[CNN_total_layer_num];

	//set parameters
	int per_run_img_num=6;
	int training_set_number = 23;
	vector<int> labels;
	vector<long> frame_nums;
	int checked_label_idx = 0;

	if(mode_select==1){
		cout<<"how many images to read: ";
		cin>>training_set_number;
	}
	else if(mode_select==0){
		training_set_number=1;
	}else if(mode_select==2){
		cout<<"how many images in total: ";
		cin>>training_set_number;
		cout<<"how many images in per run: ";
		cin>>per_run_img_num;
	}else if(mode_select==3){
		string label_file;
		cout<<"label file directory: ";
		cin>>label_file;

		training_set_number = read_label_file_NTU(label_file, labels, frame_nums);
		bool check_content = 0;
		if(check_content){
			cout<<"Check read label: "<<endl;
			for(int i=0; i<labels.size(); i++){
				cout<<labels[i]<<", "<<frame_nums[i]<<endl;
			}
		}
		cout<<"total number of labels: "<<labels.size()<<", limit? (0 for no limit)"<<endl;
		int inf_limit;
		cin>>inf_limit;

		if(inf_limit>0){

			training_set_number = frame_nums[inf_limit-1];

		}
		cout<<"Training Set Number: "<<training_set_number<<endl;
		cout<<"==++==:"<<frame_nums[462]<<frame_nums[463]<<endl;

		//return;
	}



	int training_time_each_img = input_int;
	int calculated_total_time = training_time_each_img*training_set_number;
	#undef MAX_TIME
	#define MAX_TIME calculated_total_time
	printf("==Training Total Iter: %d==", MAX_TIME);
	int total_neuron_num = 0;
	int total_spiking_num = 0;
	for(int i=0;i<CNN_total_layer_num;i++){
		total_neuron_num += network_config->layer[i].neuron_num;
		if(i!=0) total_spiking_num += network_config->layer[i].neuron_num;
	}
	total_spiking_num += 10;
	total_neuron_num += 10;
	//total_neuron_num = 20000;
	cout<<endl<<"total neuron num: "<<total_neuron_num<<endl;
	cout<<"total spiking neuron num: "<<total_spiking_num<<endl;
	#undef SIZE
	#define SIZE total_neuron_num
	#undef SPIKING_NEURON_NUM
	#define SPIKING_NEURON_NUM total_spiking_num


	float max_frequency = 60; //in Hz default 22
	float min_frequency = 3;

	int input_neuron_num = input_image_w*input_image_l*input_image_channel;
	int input_image_signal_channel_size  = input_image_w*input_image_l;
	int spiking_neuron_num = SPIKING_NEURON_NUM;
	int output_layer_neuron_num = OUTPUT_LAYER_NEURON_NUM;
	int tenpercent_iter = MAX_TIME/10;
	int connection_size = MAX_CONNECTION;
	int syn_timer_max = 25;
	int input_signal_width = 3;	//default 25
	int inhibition_time = 10;	//default 10

	float target_frequency_param = 0.5;
	float target_frequency = 100;

	bool batch_load = false;
	int batched_load_remain = 0;
	int batch_load_grand_total = 0;
	int img_load_offset = 0;
	int img_load_max = 50000;

	if (training_set_number>img_load_max){ //manually set the maximum number of images to be loaded once is 60000
		cout<<"Using batch loading"<<endl;
		batch_load_grand_total = training_set_number;
		batch_load = true;
		batched_load_remain = training_set_number - img_load_max;
		training_set_number = img_load_max;
	}

	float *mnist_img = new float[input_neuron_num*training_set_number];
	for(int i=0;i<input_neuron_num*training_set_number;i++) mnist_img[i] = 0;
	string image_file = "train-images-idx3-ubyte";//"train_dataset_noisy_cifar";//"fashion-train-images-idx3-ubyte";//"train_dataset_noisy";//"train_dataset_noisy"; //"train-images-idx3-ubyte";

	if(mode_select==0){
		cout<<"image directory: ";
		cin>>image_file;
	}else if(mode_select==1){
		cout<<"image folder directory: ";
		cin>>image_file;
	}else if(mode_select==2){
		cout<<"image folder directory: ";
		cin>>image_file;
	}else if(mode_select==3){
		cout<<"image folder directory: ";
		cin>>image_file;
	}

	cout<<endl<<"Image loading"<<endl;
	clock_t load_start = clock();
    std::vector<std::string> folder_list;
    int input_folder_cnt = 0;
	if(input_image_channel==1 || input_image_channel==2){
		//CIFAR_read_image(mnist_img, input_neuron_num, 0, 1);
		//GTVIR_read_image(mnist_img, input_neuron_num, training_set_number);
		//MNIST_read_image(image_file, mnist_img, training_set_number);
		//read_polygon("/inverse_polygon/drawings", mnist_img, training_set_number);
		if(mode_select==0)read_one_image(image_file, mnist_img, training_set_number);//"/home/xshe6/Documents/CUDA/Spike_CNN/Debug/inverse_polygon/drawings/Slide15.png"
		else if(mode_select==1)read_polygon(image_file, mnist_img, training_set_number); //"/inverse_polygon/drawings"
		else if(mode_select==2){
			//NTU_skeleton_read_image(image_file, mnist_img, training_set_number, img_load_offset);//DVS_read_image(image_file, mnist_img, training_set_number);//MNIST_read_image(image_file, mnist_img, training_set_number);//read_polygon(image_file, mnist_img, training_set_number); //"/inverse_polygon/drawings"
			cout<<"Overwrote to read MNIST images";
			MNIST_read_image("train-images-idx3-ubyte", mnist_img, training_set_number);
		}
		// for dvs_gesture_1bit, 100 images per sequence
		else if(mode_select==3) NTU_skeleton_read_image(image_file, mnist_img, training_set_number, img_load_offset);



	}else{
		bool learn_imageNet = false;
		if(learn_imageNet){

			ifstream file ("imageNet_folder_list.csv");
			string val;

		    while(file.good()) {
				getline(file, val, ',');
		    	folder_list.push_back(val);
		    }

			imageNET_read_image(folder_list[input_folder_cnt], mnist_img, training_set_number);
		}else{

			CIFAR_read_image(mnist_img, input_neuron_num, training_set_number, 1, 0);	//set 4th param as 0 for training, 1 for test data
			//KAIST_PED_read_image("", mnist_img , training_set_number);
		}
	}
	clock_t load_end = clock();

	cout<<endl<<"Image loading done"<<", time used is " << (load_end - load_start)/1000 << " (ms)"<<endl;
	//CIFAR_read_image_one_channel(mnist_img, input_image_signal_channel_size, input_int_2, 0);
	//MNIST_read_image(image_file, mnist_img, training_set_number);
	int *mnist_label = new int[training_set_number];
	string image_label_file = "train-labels-idx1-ubyte";
	//CIFAR_read_label(mnist_label, 0);
	//MNIST_read_label(image_label_file, mnist_label, training_set_number);
	//special_function: learn one category
	printf("=0=\n");
	int learn_one_digit = 0;
	int *num_one_digit_img = new int[1];
	if(learn_one_digit){
		MNIST_labeling("abc", 60000, mnist_img, mnist_label, mnist_img, num_one_digit_img, spiking_neuron_num, 1, 5);
		printf("Learning only one digit, number of img: %d\n", num_one_digit_img[0]);
	}

	//int synapse_size = SIZE*SIZE;
	//cout<<SIZE<<endl;
    Neuron *NeuronList = new Neuron[spiking_neuron_num];
    Input_neuron *Input_neuronlist = new Input_neuron[input_neuron_num];
	//unsigned char *synapse_timer = new unsigned char[synapse_size];  //this is the array that stores timer used in STPD. e.g Neuron x --->  Neuron y Spike! In the array index [(x-1)*SIZE+(y-1)]  => 1
	//hiprandState_t *states;
	//float *random_number_list = new float[SIZE];
	float *log_v_host = new float[MAX_TIME];
	float *log_spike_host = new float[total_depth_number];

	float *log_total_spike_host = new float[SIZE];
	for(int i=0; i < SIZE; i++){
		log_total_spike_host[i] = 0;
	}
	int *spike_flag = new int[CNN_total_layer_num];
	for(int i=0; i < CNN_total_layer_num; i++){
		spike_flag[i] = 0;
	}
	for(int i=0; i<total_depth_number; i++) log_spike_host[i] = 0;
	//init_log_v(log_v_host);
	//init_data_log(log_v_host,log_spike_host,log_total_spike_host, MAX_TIME);
	neuron_list_init(NeuronList, spiking_neuron_num);
	input_neuron_list_init(Input_neuronlist, input_neuron_num);
	printf("=1=\n");
	//
	if(resume_learning){
		printf("RESUME LEARNING\n");
		read_neuron_list(NeuronList, 1, "device2_output_network.txt");
		//read_neuron_list(NeuronList, 1, "spike_cnn.txt");
	}else{
		read_neuron_list(NeuronList, 1, "device2_output_network.txt");
		//read_neuron_list(NeuronList, 1, "device2_output_network.txt");
	}
	bool nomalize_weight = false;
	if(nomalize_weight){
		for(int i=0; i<3; i++){
			float start_depth = network_config->layer[3].first_depth_id - 0.1+i;
			float end_depth = network_config->layer[3].first_depth_id + 0.1+i;

			normalize_weight(NeuronList, start_depth, end_depth, 1, spiking_neuron_num);
		}
	}
    //write_neuron_list(NeuronList, "learning_output_confirm.txt", SIZE);
	//check_neuron(NeuronList, 800, 820);


	//Neuron *old_device_neurons;
	//unsigned char *snapse_timer_device;
	float *log_v;
	float *log_spike;
	float *log_spike_default;
	float *log_total_spike;
	int *spike_flag_device;


    printf("2\n");
	//random number function:

    float rand_list_size_to_total_connection_ratio = 1;
	int rand_numb_size = SPIKING_NEURON_NUM*MAX_CONNECTION;

	int SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
    dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
	dim3 print_grid(1);
	dim3 print_block(1);
	dim3 dimBlock_unit( 1, 1 );
	dim3 dimGrid_unit(1, 1);

	int SIZE_PER_SIDE_whole_network = sqrt(spiking_neuron_num)+1;
    dim3 dimBlock_whole_network( ThreadsPerBlock*2, ThreadsPerBlock );
    dim3 dimGrid_whole_network( (SIZE_PER_SIDE_whole_network/dimBlock.x+1), (SIZE_PER_SIDE_whole_network/dimBlock.y+1));
    printf("2.1\n");

	hiprandState_t *states;

//	if (STOCHASTIC_STDP) rand_init<<<dimGrid,dimBlock>>>(time(0), rand_numb_size, states);
//	float *random_number_list = new float[rand_numb_size];
//	float *random_number_list_device;
//	SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
//	dim3 dimBlock_synapse( ThreadsPerBlock, ThreadsPerBlock );
//	dim3 dimGrid_synapse( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));

//	hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
//	hipMemcpy(random_number_list_device,random_number_list,rand_numb_size*sizeof(float),hipMemcpyHostToDevice);
//	if (STOCHASTIC_STDP) random<<<dimGrid_synapse,dimBlock_synapse>>>(random_number_list_device, rand_numb_size, states);

    hiprandGenerator_t gen_uniform;
    float *random_number_list_device;
    if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING || DEVICE_VARIATION){
    	hipMalloc((void **)&states, rand_numb_size * sizeof(hiprandState_t));
		hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
		hiprandCreateGenerator(&gen_uniform, HIPRAND_RNG_PSEUDO_DEFAULT);
		hiprandSetPseudoRandomGeneratorSeed(gen_uniform, time(0));
		hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);
    }


    hiprandGenerator_t gen_normal;
    float *random_number_normal_device;
    float normal_mean = 0;
    float normal_sd = 5.0;
    if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING || DEVICE_VARIATION){
		hipMalloc((void **)&random_number_normal_device,rand_numb_size*sizeof(float));
		hiprandCreateGenerator(&gen_normal, HIPRAND_RNG_PSEUDO_DEFAULT);
		hiprandSetPseudoRandomGeneratorSeed(gen_normal, time(0));
		hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);
    }


    printf("2.11\n");
    //Setting up input instance matrix:
    float **d_input_instance;
    float **d_convolution_result;
    float **h_input_instance;
    float **h_convolution_result;
    float *probe = new float[1000];
	int instance_array_size = CNN_total_layer_num;
	hipMalloc(&d_input_instance, instance_array_size*sizeof(float *));
	int convolution_result_size = CNN_total_layer_num - 1;
	hipMalloc(&d_convolution_result, convolution_result_size*sizeof(float *));
    h_input_instance = (float**)malloc(instance_array_size * sizeof(float*));
    h_convolution_result = (float**)malloc(convolution_result_size * sizeof(float*));
    CNN_util(network_config, d_input_instance, d_convolution_result, h_input_instance, h_convolution_result, 0);
    printf("2.2\n");
//	float **add = &h_convolution_result[0];
//	printf("Address On GPU: %p\n", add);

    //========Setting up device neuron list============

	Neuron *Neuron_list_device;
    Input_neuron *Input_neuronlist_device;
    hipMalloc((void **)&Neuron_list_device, spiking_neuron_num*sizeof(Neuron));
    hipMalloc((void **)&Input_neuronlist_device, input_neuron_num*sizeof(Input_neuron));
    //hipMalloc((void **)&old_device_neurons, SIZE*sizeof(Neuron));

    //hipMalloc((void **)&states, SIZE * sizeof(hiprandState_t));
    hipMalloc((void **)&log_v, MAX_TIME * sizeof(float));
    hipMalloc((void **)&log_spike, total_depth_number * sizeof(float));
    hipMalloc((void **)&log_spike_default, total_depth_number * sizeof(float));
    //hipMalloc((void **)&log_total_spike, SIZE * sizeof(float));
    gpuErrchk( hipMalloc((void **)&log_total_spike, SIZE * sizeof(float)) );
    hipMalloc((void **)&spike_flag_device, instance_array_size*sizeof(int));
    //rand_init<<<dimGrid,dimBlock>>>(time(0), states);
    printf("2.3\n");
    hipMemcpy(Neuron_list_device,NeuronList,spiking_neuron_num*sizeof(Neuron),hipMemcpyHostToDevice);
    hipMemcpy(Input_neuronlist_device,Input_neuronlist,input_neuron_num*sizeof(Input_neuron),hipMemcpyHostToDevice);
    //hipMemcpy(old_device_neurons,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
    //hipMemcpy(random_number_list_device, random_number_list, SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(log_v,log_v_host,MAX_TIME*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike_default,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
    hipMemcpy(spike_flag_device,spike_flag,instance_array_size*sizeof(int),hipMemcpyHostToDevice);
    printf("3\n");
    //cout<<"network size: "<<SIZE<<endl;
    int network_size = SIZE;

    int max_time = MAX_TIME;
    int first_layer_time = 1;
    int second_layer_time = first_layer_time+(max_time-first_layer_time)*2/3;
    int third_layer_time = first_layer_time+(max_time-first_layer_time)*2/3;
    if(CNN_total_layer_num==3) {
    	second_layer_time = max_time + 1;
    	third_layer_time = max_time + 1;
    }
    if(CNN_total_layer_num==5){
    	first_layer_time = 1;
    	second_layer_time = 2;
    	third_layer_time = 3;
    }
    if(CNN_total_layer_num==4){
    	first_layer_time = 1;
    	second_layer_time = 2;
    	third_layer_time = max_time+1;
    }
    if (CNN_total_layer_num==2){
    	float start_depth = network_config->layer[1].first_depth_id - 0.1;
    	float end_depth = network_config->layer[1].last_depth_id + 0.1;
		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
		update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
		update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
		update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -66.2);
    	first_layer_time = max_time + 1;
    	second_layer_time = max_time + 1;
    	third_layer_time = max_time + 1;
    }
    if(resume_learning){
    	first_layer_time = 100;
    	second_layer_time = max_time;
    }

    //hipMemcpy(Neuron_list_device,old_device_neurons,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
    //first change raw img data into frequency
    int mnist_start_index = 0;
    int mnist_end_index = input_neuron_num;
    //change pixel signal to frequency

    MNIST_drive(NeuronList, Input_neuronlist, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 1);
    std::srand ( unsigned ( std::time(0) ) );
    std::vector<int> myvector;
    for (int i=0; i<training_set_number; ++i) myvector.push_back(i); // 1 2 3 4 5 6 7 8 9

    //for inference, don't shuffle
	#undef shuffle_image
	#define shuffle_image 0

    if(shuffle_image){
    	  std::random_shuffle ( myvector.begin(), myvector.end() );
    }

    hipDeviceSynchronize();

    //data_check(Neuron_list_device,log_total_spike,SIZE,1);
    float *one_mnist_img = new float[input_neuron_num];

    clock_t iter_start, iter_log;
    iter_start = clock();
    int log_interval = MAX_TIME/10;
	bool enable_log_interval = false;
	bool mass_log_spike_out = false;
    //read_filter_GPU_one_layer<<<1, 1>>>(d_network_config, h_filter_array[0], 1);
    //read_filter_GPU<<<1, 1>>>(d_network_config, d_filter_array);

//    int reiter_run = 1;

    int time = 0;
    int training_img_index = 0;
    int training_img_grand_total = 0;

    //============now load all convolution settings===========
	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
		if (layer_iter==0) {
			convolution_kernel_setup(convolution_settings, network_config, layer_iter);
		}else{
			if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel_setup(convolution_settings, network_config, layer_iter);
		}
	}
	copy_filter_to_cuDNN(Neuron_list_device, d_network_config, d_filter_array, spiking_neuron_num);
    hipDeviceSynchronize();

	while (time<=max_time){
		//if(time==first_layer_time)MNIST_drive(NeuronList, Input_neuronlist, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, max_frequency*2, min_frequency, 1);
    	if(DEVICE_VARIATION){
            hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);
    	}
    	if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING){
    	    hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);
    	}

    	if(time%log_interval == 0 && enable_log_interval){
    	    hipMemcpy(NeuronList,Neuron_list_device,spiking_neuron_num*sizeof(Neuron),hipMemcpyDeviceToHost);
    		printf("NN data copy complete\n");
    		string interval_file_name = "device2_output_at_iter_" + to_string(time) + ".txt";
    		//string interval_weight_file_name = to_string(time);
    	    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, (to_string(time)+"_"));
    		if (time>0) {
    			write_neuron_list(NeuronList, interval_file_name, spiking_neuron_num);
    		    hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
    			filter_util(network_config, NeuronList, network_size, input_neuron_num, h_filter_array, d_filter_array, to_string(time), 1);	//write filter to file
    		}
    	}

    	if(time==first_layer_time){

        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -60.2);
//    		cout<<"Changing param of long-term neuron, start: "<< start_depth+32<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+32, end_depth, 5, -1.6);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+32, end_depth, 4, 0.16);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+32, end_depth, 0, -0.001);
    		//change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+32, end_depth, -56.2);

        	start_depth = network_config->layer[2].first_depth_id - 0.1;
        	end_depth = network_config->layer[2].last_depth_id + 0.1;
//    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.0);


    		hipDeviceSynchronize();
    	}else if(time==second_layer_time){
        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -65.0);


        	start_depth = network_config->layer[2].first_depth_id - 0.1;
        	end_depth = network_config->layer[2].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -65.0);
//    		cout<<"Changing param, start: "<< start_depth+32<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+64, end_depth, 5, -1.6);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+64, end_depth, 4, 0.16);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+64, end_depth, 0, -0.001);

        	start_depth = network_config->layer[3].first_depth_id - 0.1;
        	end_depth = network_config->layer[3].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -65.0);

    		cout<<"Parameter Changing complete.\n";
    		hipDeviceSynchronize();
    	}else if(time==third_layer_time){
        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -60.2);

        	start_depth = network_config->layer[2].first_depth_id - 0.1;
        	end_depth = network_config->layer[2].last_depth_id + 0.1;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -60.0);

        	start_depth = network_config->layer[3].first_depth_id - 0.1;
        	end_depth = network_config->layer[3].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -60.2);
//    		cout<<"Changing param, start: "<< start_depth+32<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 5, -1.6);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 4, 0.16);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 0, -0.001);

        	start_depth = network_config->layer[4].first_depth_id - 0.1;
        	end_depth = network_config->layer[4].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.0);

    		cout<<"Parameter Changing complete.\n";
    		hipDeviceSynchronize();
    	}

    	if(time%tenpercent_iter == 0){
    		iter_log = clock();
    		cout<<to_string(10*(time/tenpercent_iter))<<"% done, time used is: " << (iter_log - iter_start)/1000 << " (ms)" << endl;
    	}
    	//fault below here:

    	if(time%training_time_each_img==0){//at the beginning of each img's training, load into
    		//cout<<"Image Load Iter: "<<time<<endl;
    		//cout<<";;;"<<training_img_index<<"\n";
    		//if(training_img_index>41445) cout<<training_img_index<<", "<<checked_label_idx<<"!";
    		if(mode_select==2 && time!=0)
    		{
				if((training_img_index)%per_run_img_num==0){//print current spike numbers and reset neurons
				    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );
				    ofstream myfile;
				    myfile.open((index_prefix+"inf_multirun_output_spike.csv"), std::ios_base::app);
				    if (myfile.is_open()){
				    	//myfile << "This is a new test\n";
				    	//cout<<"Checking number of neuron spike at image index "<<training_img_index<<endl;

				    	//log last layer
				    	int log_start = 0;
				    	for (int layer_i=1; layer_i<CNN_total_layer_num-1; layer_i++) log_start+=network_config->layer[layer_i].neuron_num;
				    	for(int i=(log_start); i < network_config->layer[CNN_total_layer_num-1].neuron_num+log_start ; i++){
				    		//printf("_%f_", log_v_host[i]);
							myfile << log_total_spike_host[i] << ", ";
//							cout<<log_total_spike_host[i]<<" ";

//								if( (i>=network_config->layer[1].neuron_num) && (i<network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
//									myfile << log_total_spike_host[i] << ", ";
//								}
				    	}

				    	//log any layer
//				    	int log_start = 0;
//				    	int log_end = 0;
//				    	int layer_to_log = 3;
//				    	for (int layer_i=1; layer_i<layer_to_log; layer_i++) log_start+=network_config->layer[layer_i].neuron_num;
//				    	for (int layer_i=1; layer_i<layer_to_log+1; layer_i++) log_end+=network_config->layer[layer_i].neuron_num;
//				    	for(int i=(log_start); i < log_end; i++){
//							myfile << log_total_spike_host[i] << ", ";
//				    	}

//				    	for(int i=0; i < spiking_neuron_num; i++){
//				    		//printf("_%f_", log_v_host[i]);
//				    		if(CNN_total_layer_num==3){
//								if(i>=network_config->layer[1].neuron_num){
//									myfile << log_total_spike_host[i] << ", ";
////									cout<<log_total_spike_host[i]<<" ";
//								}
//				    		}else if(CNN_total_layer_num==4){
//								if(i>=(network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
//									myfile << log_total_spike_host[i] << ", ";
//									cout<<log_total_spike_host[i]<<" ";
//								}
////								if( (i>=network_config->layer[1].neuron_num) && (i<network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
////									myfile << log_total_spike_host[i] << ", ";
////								}
//				    		}
//				    	}
				    	myfile<<endl;
				    	myfile.close();
				    }
				    if (mass_log_spike_out){
						ofstream myfile_2(("./spike_log/mass_logging/" + to_string(training_img_index) + "inf_out_device2_spike_of_neuron_out.csv"));
						if (myfile_2.is_open()){
							//myfile << "This is a new test\n";
					//    	cout<<"Checking number of neuron spike:\n";
							for(int i=0; i < spiking_neuron_num; i++){
								//printf("_%f_", log_v_host[i]);
								myfile_2 << log_total_spike_host[i] << ", ";
					//    		if(i>=network_config->layer[1].neuron_num) cout<<log_total_spike_host[i]<<" ";

							}
							myfile_2.close();
						}
				    }
//					for(int i=0; i < SIZE; i++){
//						log_total_spike_host[i] = 0;
//					}
					std::fill(log_total_spike_host, log_total_spike_host+SIZE, 0);
				    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
				    //print spike numbers done
//				    hipMemcpy(Neuron_list_device,NeuronList,spiking_neuron_num*sizeof(Neuron),hipMemcpyHostToDevice);
//		        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
//		        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
//		    		reset_all_state<<<dimBlock_whole_network, dimGrid_whole_network>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);
//		        	start_depth = network_config->layer[2].first_depth_id - 0.1;
//		        	end_depth = network_config->layer[2].last_depth_id + 0.1;
//		    		reset_all_state<<<dimBlock_whole_network, dimGrid_whole_network>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);

		    		//change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.0);
		    		hipDeviceSynchronize();
				}
    		}else if(mode_select==3 && time!=0)
    		{
				if((training_img_grand_total)==frame_nums[checked_label_idx]){//print current spike numbers and reset neurons
					cout<<"No."<<checked_label_idx<<"checking spike out at frame: "<< training_img_grand_total \
							<<", next record frame: "<<frame_nums[checked_label_idx+1]<<endl;
					long last_sequence_ending;
					if (checked_label_idx==0) last_sequence_ending = 0;
					else last_sequence_ending = frame_nums[checked_label_idx-1];
					float this_sequence_length = frame_nums[checked_label_idx] - last_sequence_ending;
					float length_normalizer = this_sequence_length/100;
					//cout<<"Length: "<<length_normalizer<<", this sequence length: "<<this_sequence_length<<endl;
					checked_label_idx ++;
				    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );
				    ofstream myfile;
				    myfile.open((index_prefix+"inf_multirun_output_spike.csv"), std::ios_base::app);
				    if (myfile.is_open()){
				    	//myfile << "This is a new test\n";
				    	//cout<<"Checking number of neuron spike at image index "<<training_img_index<<endl;
				    	int log_start = 0;
				    	for (int layer_i=1; layer_i<CNN_total_layer_num-1; layer_i++) log_start+=network_config->layer[layer_i].neuron_num;
				    	for(int i=(log_start); i < network_config->layer[CNN_total_layer_num-1].neuron_num+log_start ; i++){
				    		//printf("_%f_", log_v_host[i]);
							myfile << log_total_spike_host[i]/length_normalizer << ", ";
//							cout<<log_total_spike_host[i]<<" ";

//								if( (i>=network_config->layer[1].neuron_num) && (i<network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
//									myfile << log_total_spike_host[i] << ", ";
//								}
				    	}

//				    	for(int i=0; i < spiking_neuron_num; i++){
//				    		//printf("_%f_", log_v_host[i]);
//				    		if(CNN_total_layer_num==3){
//								if(i>=network_config->layer[1].neuron_num){
//									myfile << log_total_spike_host[i] << ", ";
////									cout<<log_total_spike_host[i]<<" ";
//								}
//				    		}else if(CNN_total_layer_num==4){
//								if(i>=(network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
//									myfile << log_total_spike_host[i] << ", ";
//									cout<<log_total_spike_host[i]<<" ";
//								}
////								if( (i>=network_config->layer[1].neuron_num) && (i<network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
////									myfile << log_total_spike_host[i] << ", ";
////								}
//				    		}
//				    	}
				    	myfile<<endl;
				    	myfile.close();
				    }
				    if (mass_log_spike_out){
						ofstream myfile_2(("./spike_log/mass_logging/" + to_string(training_img_grand_total) + "inf_out_device2_spike_of_neuron_out.csv"));
						if (myfile_2.is_open()){
							//myfile << "This is a new test\n";
					//    	cout<<"Checking number of neuron spike:\n";
							for(int i=0; i < spiking_neuron_num; i++){
								//printf("_%f_", log_v_host[i]);
								myfile_2 << log_total_spike_host[i] << ", ";
					//    		if(i>=network_config->layer[1].neuron_num) cout<<log_total_spike_host[i]<<" ";

							}
							myfile_2.close();
						}
				    }
//					for(int i=0; i < SIZE; i++){
//						log_total_spike_host[i] = 0;
//					}
					std::fill(log_total_spike_host, log_total_spike_host+SIZE, 0);
				    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
				    //print spike numbers done
//				    hipMemcpy(Neuron_list_device,NeuronList,spiking_neuron_num*sizeof(Neuron),hipMemcpyHostToDevice);
//		        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
//		        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
//		    		reset_all_state<<<dimBlock_whole_network, dimGrid_whole_network>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);
//		        	start_depth = network_config->layer[2].first_depth_id - 0.1;
//		        	end_depth = network_config->layer[2].last_depth_id + 0.1;
//		    		reset_all_state<<<dimBlock_whole_network, dimGrid_whole_network>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);

		    		//change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.0);
		    		hipDeviceSynchronize();
				}
    		}
//        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
//        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
//    		cout<<"ressetting everything";
//    		reset_all_state<<<dimBlock_whole_network, dimGrid_whole_network>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);
//        	start_depth = network_config->layer[2].first_depth_id - 0.1;
//        	end_depth = network_config->layer[2].last_depth_id + 0.1;
//    		reset_all_state<<<dimBlock_whole_network, dimGrid_whole_network>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);

			int locate_index = myvector[training_img_index];
			//cout<<"loading index: "<<locate_index<<endl;
    		for(int i=0;i<input_neuron_num;i++){
    			one_mnist_img[i] = mnist_img[locate_index*input_neuron_num+i];
    		}
//    	    for (int y=0; y<28; ++y) {
//    	    	    for (int x=0; x<28; ++x) {
//    	    	      std::cout << ((one_mnist_img[y*28+x] <= 1.1)? ' ' : '*');
//    	    	      //std::cout << int(one_mnist_img[y*28+x]) << ' ';
//    	    	    }
//    	    	    std::cout << std::endl;
//    	    }
    		MNIST_drive(Neuron_list_device, Input_neuronlist_device, one_mnist_img, input_neuron_num, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
    		//MNIST_drive(old_device_neurons, one_mnist_img, network_size,training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
    		training_img_index ++;
    		training_img_grand_total ++;
    		if(training_img_index>training_set_number-1){
        		if(batch_load && batched_load_remain>0){
        			if (batched_load_remain>img_load_max){
        				img_load_offset += training_set_number;
        				training_set_number = img_load_max;
        			}else{
        				img_load_offset += training_set_number;
        				training_set_number = batched_load_remain;
        			}

    				batched_load_remain -= training_set_number;

    				if(batched_load_remain<=0){
    					training_set_number = img_load_max;
    					batched_load_remain = batch_load_grand_total - training_set_number;
    					img_load_offset = 0;
    				}
    				myvector.clear();
    			    for (int i=0; i<training_set_number; ++i)myvector.push_back(i); // 1 2 3 4 5 6 7 8 9
    				NTU_skeleton_read_image(image_file, mnist_img, training_set_number, img_load_offset);
    			    MNIST_drive(NeuronList, Input_neuronlist, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, \
    			    		max_frequency, min_frequency, 1); //change to spike frequency
    			    cout<<"Next batch loaded, total number: "<<training_set_number<<", remaining data: "<<batched_load_remain<<endl;
        		}

    			training_img_index = 0;

    			if(shuffle_image) std::random_shuffle ( myvector.begin(), myvector.end() );
//    			one_iter = true;
    		}


    	}
    	//cout<<"One IMG loaded"<<endl;
    	//enter spiking neuron simulation:
    	int one_layer_neuron_num = 0;
    	if(time<first_layer_time){
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, 0, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, 0, false);
				}
			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {
//	    		lateral_inhibition_depth_wise_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, input_image_channel, inhibition_time, d_network_config, log_spike, total_depth_number);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition){

	    	}
	    	else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 1, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}
    	}else if(time<second_layer_time){
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, 0, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, 0, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==2){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, 0, false);
					//synapse_drive_cnn_v2(Neuron_list_device, Input_neuronlist_device, network_config, d_network_config, d_filter_array, layer_iter, spiking_neuron_num, input_neuron_num, syn_timer_max, connection_size, random_number_list_device, random_number_normal_device, states, -1.0, -1.0);//STDP
				}

			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {
//	    		lateral_inhibition_depth_wise_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, input_image_channel+network_config->layer[1].depth, inhibition_time, d_network_config, log_spike, total_depth_number);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition&& CNN_total_layer_num!=3){

	    	}else if(forced_lateral_inhibition_at_last_layer && CNN_total_layer_num==3){//if this is the last layer, use lateral_inhibition
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}

    	}else{
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, 0, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.5*input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==2){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.5*input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==3){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.5*input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==4){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, true);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}

			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {
//	    		lateral_inhibition_depth_wise_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, input_image_channel+network_config->layer[1].depth, inhibition_time, d_network_config, log_spike, total_depth_number);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition&& CNN_total_layer_num!=3){

	    	}else if(forced_lateral_inhibition_at_last_layer && CNN_total_layer_num==3){//if this is the last layer, use lateral_inhibition
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}

    	}
    	time ++;
    }
    //spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, 2);
	//hipDeviceSynchronize();



	filter_util(network_config, Neuron_list_device, spiking_neuron_num, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 2);
    hipMemcpy(NeuronList,Neuron_list_device,spiking_neuron_num*sizeof(Neuron),hipMemcpyDeviceToHost);

    hipMemcpy(log_v_host,log_v,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(log_spike_host,log_spike,total_depth_number*sizeof(float),hipMemcpyDeviceToHost);
    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );


    //print out the synapse conductance data
    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2);

    ofstream myfile ((index_prefix+"inf_out_device2_spike_of_neuron_out.csv"));
    if (myfile.is_open()){
    	//myfile << "This is a new test\n";
    	cout<<"Checking number of neuron spike:\n";
    	for(int i=0; i < spiking_neuron_num; i++){
    		//printf("_%f_", log_v_host[i]);
//    		myfile << log_total_spike_host[i] << ", ";
//    		if(i>=network_config->layer[1].neuron_num) cout<<log_total_spike_host[i]<<" ";

    	}
    	myfile.close();
    }

//    ofstream myfile_p ((index_prefix+"probe.csv"));
//    if (myfile_p.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < 1000; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_p << probe[i] << ", ";
//    	}
//    	myfile_p.close();
//    }

//
//    ofstream myfile_0 ((index_prefix+"device2_out_v.csv"));
//    if (myfile_0.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_0 << log_v_host[i] << ", ";
//    	}
//    	myfile.close();
//    }
//
//    ofstream myfile_2 ((index_prefix+"device2_spike_of_one.csv"));
//    if (myfile_2.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_2 << log_spike_host[i] << ", ";
//    	}
//    	myfile_2.close();
//    }



    hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
	//filter_util(network_config, NeuronList, network_size, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 1);	//write filter to file
    write_neuron_list(NeuronList, ("inf_out_device2_output_network.txt"), spiking_neuron_num);
    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, "");
    //===clean up===
    //delete[] random_number_list;
    delete[] log_v_host;
	delete[] NeuronList;
	delete[] log_spike_host;
	delete[] log_total_spike_host;
	delete[] mnist_img;
	delete[] NeuronList_temp;
	delete[] one_mnist_img;
	delete[] probe;
//	delete[] random_number_list;
	delete[] mnist_label;
	delete[] spike_flag;
	delete[] num_one_digit_img;
	//hipFree(states);
	hipFree(log_v);
	hipFree(log_spike);
	hipFree(log_total_spike);
	hipFree(Neuron_list_device);
	//hipFree(old_device_neurons);
	hipFree(random_number_list_device);
	hipFree(d_network_config);
	hipFree(states);
	hipFree(spike_flag_device);
	hipFree(log_spike_default);



}

void run_autotune_inference(string index_prefix, float input_float, float input_float_2, int input_int, int input_int_2, string input_img){
	cout << "Running CNN Multilayer Inference" << endl << endl;
	cout<<"Functions: \n"<<"0. One image inference\n"<<"1. load a folder\n"<<"2. One folder separate run\n" \
			<< "3. For NTU, read a label file\n";
	cout << endl;
	cout<<"Function Select: ";
	int mode_select;
	cin >> mode_select;

	switch (mode_select){
		case 0:
		{
			cout<<"One image inference selected"<<endl;
			cout<<endl;
		}
		break;
		case 1:
		{
			printf("Case 1 selected");
			cout<<"How many iterations for each image:";
			cin>>input_int;
		}
		break;
		case 2:
		{
			printf("Case 2 selected");
			cout<<"How many iterations for each image:";
			cin>>input_int;
		}
		break;
		case 3:
		{
			printf("Case 3 selected");
			cout<<"How many iterations for each image:";
			cin>>input_int;
		}
		break;
	}

	float inference_1st_layer_1st_layer_threshold;

	float inference_2nd_layer_1st_layer_threshold;
	float inference_2nd_layer_2nd_layer_threshold;

	float inference_3rd_layer_1st_layer_threshold;
	float inference_3rd_layer_2nd_layer_threshold;
	float inference_3rd_layer_3rd_layer_threshold;

	cout<<"Load input"<<endl;

	cout<<"inference_1st_layer_1st_layer_threshold";
	cin >> inference_1st_layer_1st_layer_threshold;
	cout<<"inference_2nd_layer_1st_layer_threshold";
	cin >> inference_2nd_layer_1st_layer_threshold;
	cout<<"inference_2nd_layer_2nd_layer_threshold";
	cin >> inference_2nd_layer_2nd_layer_threshold;

	cout<<"inference_3rd_layer_1st_layer_threshold";
	cin >> inference_3rd_layer_1st_layer_threshold;
	cout<<"inference_3rd_layer_2nd_layer_threshold";
	cin >> inference_3rd_layer_2nd_layer_threshold;
	cout<<"inference_3rd_layer_3rd_layer_threshold";
	cin >> inference_3rd_layer_3rd_layer_threshold;

	cout<<"loaded input: "<<inference_1st_layer_1st_layer_threshold<<' '<< inference_2nd_layer_1st_layer_threshold<<' '<<inference_2nd_layer_2nd_layer_threshold<<' ';
	cout<<inference_3rd_layer_1st_layer_threshold<<' '<<inference_3rd_layer_2nd_layer_threshold<<' '<<inference_3rd_layer_3rd_layer_threshold<<endl;
	cout<<"Input Loading Done\n"<<endl;


	/*
	int training_set_number = 1;
	int size_per_img = input_image_w * input_image_l*input_image_channel;
	float *mnist_img = new float[size_per_img*training_set_number];
	string image_file = "train-images-idx3-ubyte"; //"train-images-idx3-ubyte";
	MNIST_read_image(image_file, mnist_img, training_set_number);
	int *mnist_label = new int[training_set_number];
	string image_label_file = "train-labels-idx1-ubyte";
	MNIST_read_label(image_label_file, mnist_label, training_set_number);

	float *filter;
	float *output = new float[size_per_img*training_set_number];

	convolution_kernel(mnist_img, filter, output);
	img_util(output, "test_output.jpg", 0);
	*/
	int resume_learning = 0;
	CNN_struct *network_config = new CNN_struct;
	network_config_generator(3, network_config);
	Neuron *NeuronList_temp = new Neuron[1];
	CNN_struct *d_network_config;
	hipMalloc((void **)&d_network_config,sizeof(CNN_struct));
	hipMemcpy(d_network_config,network_config,sizeof(CNN_struct),hipMemcpyHostToDevice);
	int total_depth_number = 0;
	for(int i=0;i<CNN_total_layer_num; i++){
		total_depth_number = total_depth_number + network_config->layer[i].depth;
		cout<<"depth number: "<<network_config->layer[i].depth<<endl;
	}

	cout<<endl<<"Total depth number: "<<total_depth_number<<endl;
	float **h_filter_array;
	float **d_filter_array;
	int filter_array_size = CNN_total_layer_num-1;
	hipMalloc(&d_filter_array, filter_array_size*sizeof(float *));
	h_filter_array = (float**)malloc(filter_array_size * sizeof(float*));
	filter_util(network_config, NeuronList_temp, 0,0,  h_filter_array, d_filter_array, index_prefix, 0);

	/*
	img_util(mnist_img, "tensorflow_small.png", 1);
	img_util(mnist_img, "test_output_-1.png", 0);

	float *output = new float[size_per_img*training_set_number];

	int image_bytes = input_image_channel * input_image_l * input_image_w * sizeof(float);

	float* convolution_device_input{nullptr};
	hipMalloc(&convolution_device_input, image_bytes);
	hipMemcpy(convolution_device_input, mnist_img, image_bytes, hipMemcpyHostToDevice);

	int filter_in_channel = input_image_channel;
	int filter_out_channel = input_image_channel;
	int filter_height = 3;
	int filter_width = 3;
	const float kernel_template[3][3] = {
	{1, 1, 1},
	{1, -8, 1},
	{1, 1, 1}
	};
	float h_kernel[filter_in_channel][filter_out_channel][filter_height][filter_width];
	for (int kernel = 0; kernel < filter_in_channel; ++kernel) {
		for (int channel = 0; channel < filter_out_channel; ++channel) {
		  for (int row = 0; row < filter_height; ++row) {
			for (int column = 0; column < filter_width; ++column) {
			  h_kernel[kernel][channel][row][column] = kernel_template[row][column];
			}
		  }
		}
	}
	float* filter{nullptr};
	hipMalloc(&filter, sizeof(h_kernel));
	hipMemcpy(filter, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);


	convolution_kernel(convolution_device_input, filter, output);
	img_util(output, "test_output_1.png", 0);

	hipFree(filter);
	hipFree(convolution_device_input);

	*/

//===========END of CNN special setting-up phase============

	Convolution_setting_struct *convolution_settings = new Convolution_setting_struct[CNN_total_layer_num];

	//set parameters
	int per_run_img_num=6;
	int training_set_number = 23;
	vector<int> labels;
	vector<long> frame_nums;
	int checked_label_idx = 0;

	int use_training_set = 0;
	if(mode_select==1){
		cout<<"how many images to read: ";
		cin>>training_set_number;
	}
	else if(mode_select==0){
		training_set_number=1;
	}else if(mode_select==2){
		cout<<"training set?";
		cin>>use_training_set;
		cout<<"how many images in total: ";
		cin>>training_set_number;
		cout<<"how many images in per run: ";
		cin>>per_run_img_num;
	}else if(mode_select==3){
		string label_file;
		cout<<"label file directory: ";
		cin>>label_file;

		training_set_number = read_label_file_NTU(label_file, labels, frame_nums);
		bool check_content = 0;
		if(check_content){
			cout<<"Check read label: "<<endl;
			for(int i=0; i<labels.size(); i++){
				cout<<labels[i]<<", "<<frame_nums[i]<<endl;
			}
		}
		cout<<"total number of labels: "<<labels.size()<<", limit? (0 for no limit)"<<endl;
		int inf_limit;
		cin>>inf_limit;

		if(inf_limit>0){

			training_set_number = frame_nums[inf_limit-1];

		}
		cout<<"Training Set Number: "<<training_set_number<<endl;
		cout<<"==++==:"<<frame_nums[462]<<frame_nums[463]<<endl;

		//return;
	}



	int training_time_each_img = input_int;
	int calculated_total_time = training_time_each_img*training_set_number;
	#undef MAX_TIME
	#define MAX_TIME calculated_total_time
	printf("==Training Total Iter: %d==", MAX_TIME);
	int total_neuron_num = 0;
	int total_spiking_num = 0;
	for(int i=0;i<CNN_total_layer_num;i++){
		total_neuron_num += network_config->layer[i].neuron_num;
		if(i!=0) total_spiking_num += network_config->layer[i].neuron_num;
	}
	total_spiking_num += 10;
	total_neuron_num += 10;
	//total_neuron_num = 20000;
	cout<<endl<<"total neuron num: "<<total_neuron_num<<endl;
	cout<<"total spiking neuron num: "<<total_spiking_num<<endl;
	#undef SIZE
	#define SIZE total_neuron_num
	#undef SPIKING_NEURON_NUM
	#define SPIKING_NEURON_NUM total_spiking_num


	float max_frequency = 60; //in Hz default 22
	float min_frequency = 3;

	int input_neuron_num = input_image_w*input_image_l*input_image_channel;
	int input_image_signal_channel_size  = input_image_w*input_image_l;
	int spiking_neuron_num = SPIKING_NEURON_NUM;
	int output_layer_neuron_num = OUTPUT_LAYER_NEURON_NUM;
	int tenpercent_iter = MAX_TIME/10;
	int connection_size = MAX_CONNECTION;
	int syn_timer_max = 25;
	int input_signal_width = 3;	//default 25
	int inhibition_time = 10;	//default 10

	float target_frequency_param = 0.5;
	float target_frequency = 100;

	bool batch_load = false;
	int batched_load_remain = 0;
	int batch_load_grand_total = 0;
	int img_load_offset = 0;
	int img_load_max = 50000;

	if (training_set_number>img_load_max){ //manually set the maximum number of images to be loaded once is 60000
		cout<<"Using batch loading"<<endl;
		batch_load_grand_total = training_set_number;
		batch_load = true;
		batched_load_remain = training_set_number - img_load_max;
		training_set_number = img_load_max;
	}

	float *mnist_img = new float[input_neuron_num*training_set_number];
	for(int i=0;i<input_neuron_num*training_set_number;i++) mnist_img[i] = 0;
	string image_file = "train-images-idx3-ubyte";//"train_dataset_noisy_cifar";//"fashion-train-images-idx3-ubyte";//"train_dataset_noisy";//"train_dataset_noisy"; //"train-images-idx3-ubyte";

	if(mode_select==0){
		cout<<"image directory: ";
		cin>>image_file;
	}else if(mode_select==1){
		cout<<"image folder directory: ";
		cin>>image_file;
	}else if(mode_select==2){
		cout<<"image folder directory: ";
		cin>>image_file;
	}else if(mode_select==3){
		cout<<"image folder directory: ";
		cin>>image_file;
	}

	cout<<endl<<"Image loading"<<endl;
	clock_t load_start = clock();
    std::vector<std::string> folder_list;
    int input_folder_cnt = 0;
	if(input_image_channel==1 || input_image_channel==2){
		//CIFAR_read_image(mnist_img, input_neuron_num, 0, 1);
		//GTVIR_read_image(mnist_img, input_neuron_num, training_set_number);
		//MNIST_read_image(image_file, mnist_img, training_set_number);
		//read_polygon("/inverse_polygon/drawings", mnist_img, training_set_number);
		if(mode_select==0)read_one_image(image_file, mnist_img, training_set_number);//"/home/xshe6/Documents/CUDA/Spike_CNN/Debug/inverse_polygon/drawings/Slide15.png"
		else if(mode_select==1)read_polygon(image_file, mnist_img, training_set_number); //"/inverse_polygon/drawings"
		else if(mode_select==2) {
			//NTU_skeleton_read_image(image_file, mnist_img, training_set_number, img_load_offset);//DVS_read_image(image_file, mnist_img, training_set_number);//MNIST_read_image(image_file, mnist_img, training_set_number);//read_polygon(image_file, mnist_img, training_set_number); //"/inverse_polygon/drawings"
			cout<<"Overwrote to read MNIST images";
			if(use_training_set){
				MNIST_read_image("train-images-idx3-ubyte", mnist_img, training_set_number);
			}else{
				MNIST_read_image("t10k-images-idx3-ubyte", mnist_img, training_set_number);
			}
		}
		else if(mode_select==3) NTU_skeleton_read_image(image_file, mnist_img, training_set_number, img_load_offset);

	}else{
		bool learn_imageNet = false;
		if(learn_imageNet){

			ifstream file ("imageNet_folder_list.csv");
			string val;

		    while(file.good()) {
				getline(file, val, ',');
		    	folder_list.push_back(val);
		    }

			imageNET_read_image(folder_list[input_folder_cnt], mnist_img, training_set_number);
		}else{
			CIFAR_read_image(mnist_img, input_neuron_num, training_set_number, 0, 0);
			//KAIST_PED_read_image("", mnist_img , training_set_number);
		}

	}
	clock_t load_end = clock();

	cout<<endl<<"Image loading done"<<", time used is " << (load_end - load_start)/1000 << " (ms)"<<endl;
	//CIFAR_read_image_one_channel(mnist_img, input_image_signal_channel_size, input_int_2, 0);
	//MNIST_read_image(image_file, mnist_img, training_set_number);
	int *mnist_label = new int[training_set_number];
	string image_label_file = "train-labels-idx1-ubyte";
	//CIFAR_read_label(mnist_label, 0);
	//MNIST_read_label(image_label_file, mnist_label, training_set_number);
	//special_function: learn one category
	printf("=0=\n");
	int learn_one_digit = 0;
	int *num_one_digit_img = new int[1];
	if(learn_one_digit){
		MNIST_labeling("abc", 60000, mnist_img, mnist_label, mnist_img, num_one_digit_img, spiking_neuron_num, 1, 5);
		printf("Learning only one digit, number of img: %d\n", num_one_digit_img[0]);
	}

	//int synapse_size = SIZE*SIZE;
	//cout<<SIZE<<endl;
    Neuron *NeuronList = new Neuron[spiking_neuron_num];
    Input_neuron *Input_neuronlist = new Input_neuron[input_neuron_num];
	//unsigned char *synapse_timer = new unsigned char[synapse_size];  //this is the array that stores timer used in STPD. e.g Neuron x --->  Neuron y Spike! In the array index [(x-1)*SIZE+(y-1)]  => 1
	//hiprandState_t *states;
	//float *random_number_list = new float[SIZE];
	float *log_v_host = new float[MAX_TIME];
	float *log_spike_host = new float[total_depth_number];

	float *log_total_spike_host = new float[SIZE];
	for(int i=0; i < SIZE; i++){
		log_total_spike_host[i] = 0;
	}
	int *spike_flag = new int[CNN_total_layer_num];
	for(int i=0; i < CNN_total_layer_num; i++){
		spike_flag[i] = 0;
	}
	for(int i=0; i<total_depth_number; i++) log_spike_host[i] = 0;
	//init_log_v(log_v_host);
	//init_data_log(log_v_host,log_spike_host,log_total_spike_host, MAX_TIME);
	neuron_list_init(NeuronList, spiking_neuron_num);
	input_neuron_list_init(Input_neuronlist, input_neuron_num);
	printf("=1=\n");
	//
	if(resume_learning){
		printf("RESUME LEARNING\n");
		read_neuron_list(NeuronList, 1, "device2_output_network.txt");
		//read_neuron_list(NeuronList, 1, "spike_cnn.txt");
	}else{

		read_neuron_list(NeuronList, 1, (index_prefix+"device2_output_network.txt"));
		//read_neuron_list(NeuronList, 1, "device2_output_network.txt");
	}
	bool nomalize_weight = false;
	if(nomalize_weight){
		for(int i=0; i<3; i++){
			float start_depth = network_config->layer[3].first_depth_id - 0.1+i;
			float end_depth = network_config->layer[3].first_depth_id + 0.1+i;

			normalize_weight(NeuronList, start_depth, end_depth, 1, spiking_neuron_num);
		}
	}
    //write_neuron_list(NeuronList, "learning_output_confirm.txt", SIZE);
	//check_neuron(NeuronList, 800, 820);


	//Neuron *old_device_neurons;
	//unsigned char *snapse_timer_device;
	float *log_v;
	float *log_spike;
	float *log_spike_default;
	float *log_total_spike;
	int *spike_flag_device;


    printf("2\n");
	//random number function:

    float rand_list_size_to_total_connection_ratio = 1;
	int rand_numb_size = SPIKING_NEURON_NUM*MAX_CONNECTION;

	int SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
    dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
	dim3 print_grid(1);
	dim3 print_block(1);
	dim3 dimBlock_unit( 1, 1 );
	dim3 dimGrid_unit(1, 1);

	int SIZE_PER_SIDE_whole_network = sqrt(spiking_neuron_num)+1;
    dim3 dimBlock_whole_network( ThreadsPerBlock*2, ThreadsPerBlock );
    dim3 dimGrid_whole_network( (SIZE_PER_SIDE_whole_network/dimBlock.x+1), (SIZE_PER_SIDE_whole_network/dimBlock.y+1));
    printf("2.1\n");

	hiprandState_t *states;

//	if (STOCHASTIC_STDP) rand_init<<<dimGrid,dimBlock>>>(time(0), rand_numb_size, states);
//	float *random_number_list = new float[rand_numb_size];
//	float *random_number_list_device;
//	SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
//	dim3 dimBlock_synapse( ThreadsPerBlock, ThreadsPerBlock );
//	dim3 dimGrid_synapse( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));

//	hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
//	hipMemcpy(random_number_list_device,random_number_list,rand_numb_size*sizeof(float),hipMemcpyHostToDevice);
//	if (STOCHASTIC_STDP) random<<<dimGrid_synapse,dimBlock_synapse>>>(random_number_list_device, rand_numb_size, states);

    hiprandGenerator_t gen_uniform;
    float *random_number_list_device;
    if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING || DEVICE_VARIATION){
    	hipMalloc((void **)&states, rand_numb_size * sizeof(hiprandState_t));
		hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
		hiprandCreateGenerator(&gen_uniform, HIPRAND_RNG_PSEUDO_DEFAULT);
		hiprandSetPseudoRandomGeneratorSeed(gen_uniform, time(0));
		hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);
    }


    hiprandGenerator_t gen_normal;
    float *random_number_normal_device;
    float normal_mean = 0;
    float normal_sd = 5.0;
    if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING || DEVICE_VARIATION){
		hipMalloc((void **)&random_number_normal_device,rand_numb_size*sizeof(float));
		hiprandCreateGenerator(&gen_normal, HIPRAND_RNG_PSEUDO_DEFAULT);
		hiprandSetPseudoRandomGeneratorSeed(gen_normal, time(0));
		hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);
    }


    printf("2.11\n");
    //Setting up input instance matrix:
    float **d_input_instance;
    float **d_convolution_result;
    float **h_input_instance;
    float **h_convolution_result;
    float *probe = new float[1000];
	int instance_array_size = CNN_total_layer_num;
	hipMalloc(&d_input_instance, instance_array_size*sizeof(float *));
	int convolution_result_size = CNN_total_layer_num - 1;
	hipMalloc(&d_convolution_result, convolution_result_size*sizeof(float *));
    h_input_instance = (float**)malloc(instance_array_size * sizeof(float*));
    h_convolution_result = (float**)malloc(convolution_result_size * sizeof(float*));
    CNN_util(network_config, d_input_instance, d_convolution_result, h_input_instance, h_convolution_result, 0);
    printf("2.2\n");
//	float **add = &h_convolution_result[0];
//	printf("Address On GPU: %p\n", add);

    //========Setting up device neuron list============

	Neuron *Neuron_list_device;
    Input_neuron *Input_neuronlist_device;
    hipMalloc((void **)&Neuron_list_device, spiking_neuron_num*sizeof(Neuron));
    hipMalloc((void **)&Input_neuronlist_device, input_neuron_num*sizeof(Input_neuron));
    //hipMalloc((void **)&old_device_neurons, SIZE*sizeof(Neuron));

    //hipMalloc((void **)&states, SIZE * sizeof(hiprandState_t));
    hipMalloc((void **)&log_v, MAX_TIME * sizeof(float));
    hipMalloc((void **)&log_spike, total_depth_number * sizeof(float));
    hipMalloc((void **)&log_spike_default, total_depth_number * sizeof(float));
    //hipMalloc((void **)&log_total_spike, SIZE * sizeof(float));
    gpuErrchk( hipMalloc((void **)&log_total_spike, SIZE * sizeof(float)) );
    hipMalloc((void **)&spike_flag_device, instance_array_size*sizeof(int));
    //rand_init<<<dimGrid,dimBlock>>>(time(0), states);
    printf("2.3\n");
    hipMemcpy(Neuron_list_device,NeuronList,spiking_neuron_num*sizeof(Neuron),hipMemcpyHostToDevice);
    hipMemcpy(Input_neuronlist_device,Input_neuronlist,input_neuron_num*sizeof(Input_neuron),hipMemcpyHostToDevice);
    //hipMemcpy(old_device_neurons,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
    //hipMemcpy(random_number_list_device, random_number_list, SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(log_v,log_v_host,MAX_TIME*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike_default,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
    hipMemcpy(spike_flag_device,spike_flag,instance_array_size*sizeof(int),hipMemcpyHostToDevice);
    printf("3\n");
    //cout<<"network size: "<<SIZE<<endl;
    int network_size = SIZE;

    int max_time = MAX_TIME;
    int first_layer_time = 1;
    int second_layer_time = first_layer_time+(max_time-first_layer_time)*2/3;
    int third_layer_time = first_layer_time+(max_time-first_layer_time)*2/3;
    if(CNN_total_layer_num==3) {
    	second_layer_time = max_time + 1;
    	third_layer_time = max_time + 1;
    }
    if(CNN_total_layer_num==5){
    	first_layer_time = 1;
    	second_layer_time = 2;
    	third_layer_time = 3;
    }
    if(CNN_total_layer_num==4){
    	first_layer_time = 1;
    	second_layer_time = 2;
    	third_layer_time = max_time+1;
    }
    if (CNN_total_layer_num==2){
    	float start_depth = network_config->layer[1].first_depth_id - 0.1;
    	float end_depth = network_config->layer[1].last_depth_id + 0.1;
		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
		update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
		update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
		update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -66.2);
    	first_layer_time = max_time + 1;
    	second_layer_time = max_time + 1;
    	third_layer_time = max_time + 1;
    }
    if(resume_learning){
    	first_layer_time = 100;
    	second_layer_time = max_time;
    }

    //hipMemcpy(Neuron_list_device,old_device_neurons,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
    //first change raw img data into frequency
    int mnist_start_index = 0;
    int mnist_end_index = input_neuron_num;
    //change pixel signal to frequency

    MNIST_drive(NeuronList, Input_neuronlist, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 1);
    std::srand ( unsigned ( std::time(0) ) );
    std::vector<int> myvector;
    for (int i=0; i<training_set_number; ++i) myvector.push_back(i); // 1 2 3 4 5 6 7 8 9

	#undef shuffle_image
	#define shuffle_image 0

    if(shuffle_image){
    	  std::random_shuffle ( myvector.begin(), myvector.end() );
    }

    hipDeviceSynchronize();

    //data_check(Neuron_list_device,log_total_spike,SIZE,1);
    float *one_mnist_img = new float[input_neuron_num];

    clock_t iter_start, iter_log;
    iter_start = clock();
    int log_interval = MAX_TIME/10;
	bool enable_log_interval = false;
	bool mass_log_spike_out = false;
    //read_filter_GPU_one_layer<<<1, 1>>>(d_network_config, h_filter_array[0], 1);
    //read_filter_GPU<<<1, 1>>>(d_network_config, d_filter_array);

//    int reiter_run = 1;

    int time = 0;
    int training_img_index = 0;
    int training_img_grand_total = 0;

    //============now load all convolution settings===========
	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
		if (layer_iter==0) {
			convolution_kernel_setup(convolution_settings, network_config, layer_iter);
		}else{
			if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel_setup(convolution_settings, network_config, layer_iter);
		}
	}
	copy_filter_to_cuDNN(Neuron_list_device, d_network_config, d_filter_array, spiking_neuron_num);
    hipDeviceSynchronize();

	while (time<=max_time){
		//if(time==first_layer_time)MNIST_drive(NeuronList, Input_neuronlist, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, max_frequency*2, min_frequency, 1);
    	if(DEVICE_VARIATION){
            hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);
    	}
    	if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING){
    	    hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);
    	}

    	if(time%log_interval == 0 && enable_log_interval){
    	    hipMemcpy(NeuronList,Neuron_list_device,spiking_neuron_num*sizeof(Neuron),hipMemcpyDeviceToHost);
    		printf("NN data copy complete\n");
    		string interval_file_name = "device2_output_at_iter_" + to_string(time) + ".txt";
    		//string interval_weight_file_name = to_string(time);
    	    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, (to_string(time)+"_"));
    		if (time>0) {
    			write_neuron_list(NeuronList, interval_file_name, spiking_neuron_num);
    		    hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
    			filter_util(network_config, NeuronList, network_size, input_neuron_num, h_filter_array, d_filter_array, to_string(time), 1);	//write filter to file
    		}
    	}

    	if(time==first_layer_time){

        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, inference_2nd_layer_1st_layer_threshold);
//    		cout<<"Changing param of long-term neuron, start: "<< start_depth+32<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+32, end_depth, 5, -1.6);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+32, end_depth, 4, 0.16);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+32, end_depth, 0, -0.001);
    		//change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+32, end_depth, -56.2);

        	start_depth = network_config->layer[2].first_depth_id - 0.1;
        	end_depth = network_config->layer[2].last_depth_id + 0.1;
//    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, inference_2nd_layer_2nd_layer_threshold);


    		hipDeviceSynchronize();
    	}else if(time==second_layer_time){
        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, inference_3rd_layer_1st_layer_threshold);


        	start_depth = network_config->layer[2].first_depth_id - 0.1;
        	end_depth = network_config->layer[2].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, inference_3rd_layer_2nd_layer_threshold);
//    		cout<<"Changing param, start: "<< start_depth+32<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+64, end_depth, 5, -1.6);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+64, end_depth, 4, 0.16);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+64, end_depth, 0, -0.001);

        	start_depth = network_config->layer[3].first_depth_id - 0.1;
        	end_depth = network_config->layer[3].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, inference_3rd_layer_3rd_layer_threshold);

    		cout<<"Parameter Changing complete.\n";
    		hipDeviceSynchronize();
    	}else if(time==third_layer_time){
        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -68.2);


        	start_depth = network_config->layer[3].first_depth_id - 0.1;
        	end_depth = network_config->layer[3].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -71.2);
    		cout<<"Changing param, start: "<< start_depth+32<<" end: "<<end_depth<<endl;
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 5, -1.6);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 4, 0.16);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 0, -0.001);

        	start_depth = network_config->layer[4].first_depth_id - 0.1;
        	end_depth = network_config->layer[4].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -71.0);

    		cout<<"Parameter Changing complete.\n";
    		hipDeviceSynchronize();
    	}

    	if(time%tenpercent_iter == 0){
    		iter_log = clock();
    		cout<<to_string(10*(time/tenpercent_iter))<<"% done, time used is: " << (iter_log - iter_start)/1000 << " (ms)" << endl;
    	}
    	//fault below here:

    	if(time%training_time_each_img==0){//at the beginning of each img's training, load into
    		//cout<<"Image Load Iter: "<<time<<endl;
    		//cout<<";;;"<<training_img_index<<"\n";
    		//if(training_img_index>41445) cout<<training_img_index<<", "<<checked_label_idx<<"!";
    		if(mode_select==2 && time!=0)
    		{
				if((training_img_index)%per_run_img_num==0){//print current spike numbers and reset neurons
				    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );
				    ofstream myfile;

				    if(use_training_set){
				    	myfile.open((index_prefix+"training_inf_multirun_output_spike.csv"), std::ios_base::app);
				    }else{
				    	myfile.open((index_prefix+"test_inf_multirun_output_spike.csv"), std::ios_base::app);
				    }
				    if (myfile.is_open()){
				    	//myfile << "This is a new test\n";
				    	//cout<<"Checking number of neuron spike at image index "<<training_img_index<<endl;
				    	int log_start = 0;
				    	for (int layer_i=1; layer_i<CNN_total_layer_num-1; layer_i++) log_start+=network_config->layer[layer_i].neuron_num;
				    	for(int i=(log_start); i < network_config->layer[CNN_total_layer_num-1].neuron_num+log_start ; i++){
				    		//printf("_%f_", log_v_host[i]);
				    		myfile << log_total_spike_host[i] << ", ";
//							cout<<log_total_spike_host[i]<<" ";

//								if( (i>=network_config->layer[1].neuron_num) && (i<network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
//									myfile << log_total_spike_host[i] << ", ";
//								}
				    	}

//				    	for(int i=0; i < spiking_neuron_num; i++){
//				    		//printf("_%f_", log_v_host[i]);
//				    		if(CNN_total_layer_num==3){
//								if(i>=network_config->layer[1].neuron_num){
//									myfile << log_total_spike_host[i] << ", ";
////									cout<<log_total_spike_host[i]<<" ";
//								}
//				    		}else if(CNN_total_layer_num==4){
//								if(i>=(network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
//									myfile << log_total_spike_host[i] << ", ";
//									cout<<log_total_spike_host[i]<<" ";
//								}
////								if( (i>=network_config->layer[1].neuron_num) && (i<network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
////									myfile << log_total_spike_host[i] << ", ";
////								}
//				    		}
//				    	}
				    	myfile<<endl;
				    	myfile.close();
				    }
				    if (mass_log_spike_out){
						ofstream myfile_2(("./spike_log/mass_logging/" + to_string(training_img_index) + "inf_out_device2_spike_of_neuron_out.csv"));
						if (myfile_2.is_open()){
							//myfile << "This is a new test\n";
					//    	cout<<"Checking number of neuron spike:\n";
							for(int i=0; i < spiking_neuron_num; i++){
								//printf("_%f_", log_v_host[i]);
								myfile_2 << log_total_spike_host[i] << ", ";
					//    		if(i>=network_config->layer[1].neuron_num) cout<<log_total_spike_host[i]<<" ";

							}
							myfile_2.close();
						}
				    }
//					for(int i=0; i < SIZE; i++){
//						log_total_spike_host[i] = 0;
//					}
					std::fill(log_total_spike_host, log_total_spike_host+SIZE, 0);
				    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
				    //print spike numbers done
//				    hipMemcpy(Neuron_list_device,NeuronList,spiking_neuron_num*sizeof(Neuron),hipMemcpyHostToDevice);
//		        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
//		        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
//		    		reset_all_state<<<dimBlock_whole_network, dimGrid_whole_network>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);
//		        	start_depth = network_config->layer[2].first_depth_id - 0.1;
//		        	end_depth = network_config->layer[2].last_depth_id + 0.1;
//		    		reset_all_state<<<dimBlock_whole_network, dimGrid_whole_network>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);

		    		//change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.0);
		    		hipDeviceSynchronize();
				}
    		}else if(mode_select==3 && time!=0)
    		{
				if((training_img_grand_total)==frame_nums[checked_label_idx]){//print current spike numbers and reset neurons
					cout<<"No."<<checked_label_idx<<"checking spike out at frame: "<< training_img_grand_total \
							<<", next record frame: "<<frame_nums[checked_label_idx+1]<<endl;
					long last_sequence_ending;
					if (checked_label_idx==0) last_sequence_ending = 0;
					else last_sequence_ending = frame_nums[checked_label_idx-1];
					float this_sequence_length = frame_nums[checked_label_idx] - last_sequence_ending;
					float length_normalizer = this_sequence_length/100;
					//cout<<"Length: "<<length_normalizer<<", this sequence length: "<<this_sequence_length<<endl;
					checked_label_idx ++;
				    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );
				    ofstream myfile;
				    myfile.open((index_prefix+"inf_multirun_output_spike.csv"), std::ios_base::app);
				    if (myfile.is_open()){
				    	//myfile << "This is a new test\n";
				    	//cout<<"Checking number of neuron spike at image index "<<training_img_index<<endl;
				    	int log_start = 0;
				    	for (int layer_i=1; layer_i<CNN_total_layer_num-1; layer_i++) log_start+=network_config->layer[layer_i].neuron_num;
				    	for(int i=(log_start); i < network_config->layer[CNN_total_layer_num-1].neuron_num+log_start ; i++){
				    		//printf("_%f_", log_v_host[i]);
							myfile << log_total_spike_host[i]/length_normalizer << ", ";
//							cout<<log_total_spike_host[i]<<" ";

//								if( (i>=network_config->layer[1].neuron_num) && (i<network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
//									myfile << log_total_spike_host[i] << ", ";
//								}
				    	}

//				    	for(int i=0; i < spiking_neuron_num; i++){
//				    		//printf("_%f_", log_v_host[i]);
//				    		if(CNN_total_layer_num==3){
//								if(i>=network_config->layer[1].neuron_num){
//									myfile << log_total_spike_host[i] << ", ";
////									cout<<log_total_spike_host[i]<<" ";
//								}
//				    		}else if(CNN_total_layer_num==4){
//								if(i>=(network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
//									myfile << log_total_spike_host[i] << ", ";
//									cout<<log_total_spike_host[i]<<" ";
//								}
////								if( (i>=network_config->layer[1].neuron_num) && (i<network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
////									myfile << log_total_spike_host[i] << ", ";
////								}
//				    		}
//				    	}
				    	myfile<<endl;
				    	myfile.close();
				    }
				    if (mass_log_spike_out){
						ofstream myfile_2(("./spike_log/mass_logging/" + to_string(training_img_grand_total) + "inf_out_device2_spike_of_neuron_out.csv"));
						if (myfile_2.is_open()){
							//myfile << "This is a new test\n";
					//    	cout<<"Checking number of neuron spike:\n";
							for(int i=0; i < spiking_neuron_num; i++){
								//printf("_%f_", log_v_host[i]);
								myfile_2 << log_total_spike_host[i] << ", ";
					//    		if(i>=network_config->layer[1].neuron_num) cout<<log_total_spike_host[i]<<" ";

							}
							myfile_2.close();
						}
				    }
//					for(int i=0; i < SIZE; i++){
//						log_total_spike_host[i] = 0;
//					}
					std::fill(log_total_spike_host, log_total_spike_host+SIZE, 0);
				    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
				    //print spike numbers done
//				    hipMemcpy(Neuron_list_device,NeuronList,spiking_neuron_num*sizeof(Neuron),hipMemcpyHostToDevice);
//		        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
//		        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
//		    		reset_all_state<<<dimBlock_whole_network, dimGrid_whole_network>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);
//		        	start_depth = network_config->layer[2].first_depth_id - 0.1;
//		        	end_depth = network_config->layer[2].last_depth_id + 0.1;
//		    		reset_all_state<<<dimBlock_whole_network, dimGrid_whole_network>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);

		    		//change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.0);
		    		hipDeviceSynchronize();
				}
    		}
//        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
//        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
//    		cout<<"ressetting everything";
//    		reset_all_state<<<dimBlock_whole_network, dimGrid_whole_network>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);
//        	start_depth = network_config->layer[2].first_depth_id - 0.1;
//        	end_depth = network_config->layer[2].last_depth_id + 0.1;
//    		reset_all_state<<<dimBlock_whole_network, dimGrid_whole_network>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);

			int locate_index = myvector[training_img_index];
			//cout<<"loading index: "<<locate_index<<endl;
    		for(int i=0;i<input_neuron_num;i++){
    			one_mnist_img[i] = mnist_img[locate_index*input_neuron_num+i];
    		}
//    	    for (int y=0; y<28; ++y) {
//    	    	    for (int x=0; x<28; ++x) {
//    	    	      std::cout << ((one_mnist_img[y*28+x] <= 1.1)? ' ' : '*');
//    	    	      //std::cout << int(one_mnist_img[y*28+x]) << ' ';
//    	    	    }
//    	    	    std::cout << std::endl;
//    	    }
    		MNIST_drive(Neuron_list_device, Input_neuronlist_device, one_mnist_img, input_neuron_num, training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
    		//MNIST_drive(old_device_neurons, one_mnist_img, network_size,training_set_number, mnist_start_index, mnist_end_index, max_frequency, min_frequency, 0);
    		training_img_index ++;
    		training_img_grand_total ++;
    		if(training_img_index>training_set_number-1){
        		if(batch_load && batched_load_remain>0){
        			if (batched_load_remain>img_load_max){
        				img_load_offset += training_set_number;
        				training_set_number = img_load_max;
        			}else{
        				img_load_offset += training_set_number;
        				training_set_number = batched_load_remain;
        			}

    				batched_load_remain -= training_set_number;

    				if(batched_load_remain<=0){
    					training_set_number = img_load_max;
    					batched_load_remain = batch_load_grand_total - training_set_number;
    					img_load_offset = 0;
    				}
    				myvector.clear();
    			    for (int i=0; i<training_set_number; ++i)myvector.push_back(i); // 1 2 3 4 5 6 7 8 9
    				NTU_skeleton_read_image(image_file, mnist_img, training_set_number, img_load_offset);
    			    MNIST_drive(NeuronList, Input_neuronlist, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, \
    			    		max_frequency, min_frequency, 1); //change to spike frequency
    			    cout<<"Next batch loaded, total number: "<<training_set_number<<", remaining data: "<<batched_load_remain<<endl;
        		}

    			training_img_index = 0;

    			if(shuffle_image) std::random_shuffle ( myvector.begin(), myvector.end() );
//    			one_iter = true;
    		}


    	}
    	//cout<<"One IMG loaded"<<endl;
    	//enter spiking neuron simulation:
    	int one_layer_neuron_num = 0;
    	if(time<first_layer_time){
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, 0, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 1.3*input_float, 0, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, \
							h_convolution_result, probe);
					//synapse_drive_cnn_v2(Neuron_list_device, Input_neuronlist_device, network_config, d_network_config, d_filter_array, layer_iter, \
							spiking_neuron_num, input_neuron_num, syn_timer_max, connection_size, random_number_list_device, random_number_normal_device, states, -1.0, -1.0);//STDP
				}
			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {
//	    		lateral_inhibition_depth_wise_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, input_image_channel, inhibition_time, d_network_config, log_spike, total_depth_number);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition){

	    	}
	    	else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 1, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}
    	}else if(time<second_layer_time){
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, 0, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 2*input_float, 0, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==2){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, 0, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, \
							h_convolution_result, probe);
					//synapse_drive_cnn_v2(Neuron_list_device, Input_neuronlist_device, network_config, d_network_config, d_filter_array, layer_iter, spiking_neuron_num, input_neuron_num, syn_timer_max, connection_size, random_number_list_device, random_number_normal_device, states, -1.0, -1.0);//STDP
				}

			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {
//	    		lateral_inhibition_depth_wise_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, input_image_channel+network_config->layer[1].depth, inhibition_time, d_network_config, log_spike, total_depth_number);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition&& CNN_total_layer_num!=3){

	    	}else if(forced_lateral_inhibition_at_last_layer && CNN_total_layer_num==3){//if this is the last layer, use lateral_inhibition
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}

    	}else{
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, 0, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.5*input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==2){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.5*input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==3){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.3*input_float, time, true);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==4){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.6*input_float, time, true);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}

			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {
//	    		lateral_inhibition_depth_wise_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, input_image_channel+network_config->layer[1].depth, inhibition_time, d_network_config, log_spike, total_depth_number);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition&& CNN_total_layer_num!=3){

	    	}else if(forced_lateral_inhibition_at_last_layer && CNN_total_layer_num==3){//if this is the last layer, use lateral_inhibition
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}

    	}
    	time ++;
    }
    //spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, 2);
	//hipDeviceSynchronize();



//	filter_util(network_config, Neuron_list_device, spiking_neuron_num, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 2);
//    hipMemcpy(NeuronList,Neuron_list_device,spiking_neuron_num*sizeof(Neuron),hipMemcpyDeviceToHost);
//
//    hipMemcpy(log_v_host,log_v,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
//    hipMemcpy(log_spike_host,log_spike,total_depth_number*sizeof(float),hipMemcpyDeviceToHost);
//    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );


    //print out the synapse conductance data
    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2);

//    ofstream myfile ((index_prefix+"inf_out_device2_spike_of_neuron_out.csv"));
//    if (myfile.is_open()){
//    	//myfile << "This is a new test\n";
//    	cout<<"Checking number of neuron spike:\n";
//    	for(int i=0; i < spiking_neuron_num; i++){
//    		//printf("_%f_", log_v_host[i]);
////    		myfile << log_total_spike_host[i] << ", ";
////    		if(i>=network_config->layer[1].neuron_num) cout<<log_total_spike_host[i]<<" ";
//
//    	}
//    	myfile.close();
//    }

//    ofstream myfile_p ((index_prefix+"probe.csv"));
//    if (myfile_p.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < 1000; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_p << probe[i] << ", ";
//    	}
//    	myfile_p.close();
//    }

//
//    ofstream myfile_0 ((index_prefix+"device2_out_v.csv"));
//    if (myfile_0.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_0 << log_v_host[i] << ", ";
//    	}
//    	myfile.close();
//    }
//
//    ofstream myfile_2 ((index_prefix+"device2_spike_of_one.csv"));
//    if (myfile_2.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_2 << log_spike_host[i] << ", ";
//    	}
//    	myfile_2.close();
//    }



    //hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
	//filter_util(network_config, NeuronList, network_size, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 1);	//write filter to file
    //write_neuron_list(NeuronList, ("inf_out_device2_output_network.txt"), spiking_neuron_num);
    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, "");
    //===clean up===
    //delete[] random_number_list;
    delete[] log_v_host;
	delete[] NeuronList;
	delete[] log_spike_host;
	delete[] log_total_spike_host;
	delete[] mnist_img;
	delete[] NeuronList_temp;
	delete[] one_mnist_img;
	delete[] probe;
//	delete[] random_number_list;
	delete[] mnist_label;
	delete[] spike_flag;
	delete[] num_one_digit_img;
	//hipFree(states);
	hipFree(log_v);
	hipFree(log_spike);
	hipFree(log_total_spike);
	hipFree(Neuron_list_device);
	//hipFree(old_device_neurons);
	hipFree(random_number_list_device);
	hipFree(d_network_config);
	hipFree(states);
	hipFree(spike_flag_device);
	hipFree(log_spike_default);



}

void run_event_based_inference(string index_prefix, float input_float, float input_float_2, int input_int, int input_int_2, string input_img){
	cout << "Running CNN Multilayer Inference" << endl << endl;
	cout<<"Functions: \n"<<"0. load from regular event file\n";
	cout << endl;
	cout<<"Function Select: ";
	int mode_select;
	cin >> mode_select;

	switch (mode_select){
		case 0:
		{
			cout<<"-load from regular event file- selected"<<endl;
			cout<<endl;
		}
		break;
		case 1:
		{
			printf("Case 1 selected, no such option");
			return;
		}
		break;
		case 2:
		{
			printf("Case 2 selected, no such option");
			return;
		}
		break;
		case 3:
		{
			printf("Case 3 selected, no such option");
			return;
		}
		break;
	}



	/*
	int training_set_number = 1;
	int size_per_img = input_image_w * input_image_l*input_image_channel;
	float *mnist_img = new float[size_per_img*training_set_number];
	string image_file = "train-images-idx3-ubyte"; //"train-images-idx3-ubyte";
	MNIST_read_image(image_file, mnist_img, training_set_number);
	int *mnist_label = new int[training_set_number];
	string image_label_file = "train-labels-idx1-ubyte";
	MNIST_read_label(image_label_file, mnist_label, training_set_number);

	float *filter;
	float *output = new float[size_per_img*training_set_number];

	convolution_kernel(mnist_img, filter, output);
	img_util(output, "test_output.jpg", 0);
	*/
	int resume_learning = 0;
	CNN_struct *network_config = new CNN_struct;
	network_config_generator(3, network_config);
	Neuron *NeuronList_temp = new Neuron[1];
	CNN_struct *d_network_config;
	hipMalloc((void **)&d_network_config,sizeof(CNN_struct));
	hipMemcpy(d_network_config,network_config,sizeof(CNN_struct),hipMemcpyHostToDevice);
	int total_depth_number = 0;
	for(int i=0;i<CNN_total_layer_num; i++){
		total_depth_number = total_depth_number + network_config->layer[i].depth;
		cout<<"depth number: "<<network_config->layer[i].depth<<endl;
	}

	cout<<endl<<"Total depth number: "<<total_depth_number<<endl;
	float **h_filter_array;
	float **d_filter_array;
	int filter_array_size = CNN_total_layer_num-1;
	hipMalloc(&d_filter_array, filter_array_size*sizeof(float *));
	h_filter_array = (float**)malloc(filter_array_size * sizeof(float*));
	filter_util(network_config, NeuronList_temp, 0,0,  h_filter_array, d_filter_array, index_prefix, 0);

	/*
	img_util(mnist_img, "tensorflow_small.png", 1);
	img_util(mnist_img, "test_output_-1.png", 0);

	float *output = new float[size_per_img*training_set_number];

	int image_bytes = input_image_channel * input_image_l * input_image_w * sizeof(float);

	float* convolution_device_input{nullptr};
	hipMalloc(&convolution_device_input, image_bytes);
	hipMemcpy(convolution_device_input, mnist_img, image_bytes, hipMemcpyHostToDevice);

	int filter_in_channel = input_image_channel;
	int filter_out_channel = input_image_channel;
	int filter_height = 3;
	int filter_width = 3;
	const float kernel_template[3][3] = {
	{1, 1, 1},
	{1, -8, 1},
	{1, 1, 1}
	};
	float h_kernel[filter_in_channel][filter_out_channel][filter_height][filter_width];
	for (int kernel = 0; kernel < filter_in_channel; ++kernel) {
		for (int channel = 0; channel < filter_out_channel; ++channel) {
		  for (int row = 0; row < filter_height; ++row) {
			for (int column = 0; column < filter_width; ++column) {
			  h_kernel[kernel][channel][row][column] = kernel_template[row][column];
			}
		  }
		}
	}
	float* filter{nullptr};
	hipMalloc(&filter, sizeof(h_kernel));
	hipMemcpy(filter, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);


	convolution_kernel(convolution_device_input, filter, output);
	img_util(output, "test_output_1.png", 0);

	hipFree(filter);
	hipFree(convolution_device_input);

	*/

//===========END of CNN special setting-up phase============

	Convolution_setting_struct *convolution_settings = new Convolution_setting_struct[CNN_total_layer_num];

	//set parameters
	int per_run_img_num=6;
	float total_event_to_infer = 23;
	vector<int> labels;
	vector<long> frame_nums;
	int checked_label_idx = 0;

	if(mode_select==0){
		cout<<"how many million events to run: ";
		cin>>total_event_to_infer;
	}
	else if(mode_select==1){

	}else if(mode_select==2){

	}else if(mode_select==3){

	}

    static int img_load_max  = 10000000;

	int time_per_event = input_int;
	int calculated_total_time = time_per_event*total_event_to_infer*1000000;
	#undef MAX_TIME
	#define MAX_TIME calculated_total_time
	printf("==Training Total Iter: %d==", MAX_TIME);
	int total_neuron_num = 0;
	int total_spiking_num = 0;
	for(int i=0;i<CNN_total_layer_num;i++){
		total_neuron_num += network_config->layer[i].neuron_num;
		if(i!=0) total_spiking_num += network_config->layer[i].neuron_num;
	}
	total_spiking_num += 10;
	total_neuron_num += 10;
	//total_neuron_num = 20000;
	cout<<endl<<"total neuron num: "<<total_neuron_num<<endl;
	cout<<"total spiking neuron num: "<<total_spiking_num<<endl;
	#undef SIZE
	#define SIZE total_neuron_num
	#undef SPIKING_NEURON_NUM
	#define SPIKING_NEURON_NUM total_spiking_num


	float max_frequency = 100; //in Hz default 22
	float min_frequency = 10;

	int input_neuron_num = input_image_w*input_image_l*input_image_channel;
	int input_image_signal_channel_size  = input_image_w*input_image_l;
	int spiking_neuron_num = SPIKING_NEURON_NUM;
	int output_layer_neuron_num = OUTPUT_LAYER_NEURON_NUM;
	int tenpercent_iter = MAX_TIME/10;
	int connection_size = MAX_CONNECTION;
	int syn_timer_max = 25;
	int input_signal_width = 10;	//default 25
	int inhibition_time = 10;	//default 10

	float target_frequency_param = 0.5;
	float target_frequency = 100;


	Event_Camera_Input *events_host = new Event_Camera_Input[img_load_max];
	Event_Camera_Input *events_GPU;
	hipMalloc((void **)&events_GPU,img_load_max*sizeof(Event_Camera_Input));

	int current_input_file_id = 8;
	int input_file_id_max = 20;
	string image_file = "";
	if (current_input_file_id<10) {
		image_file = "/hdd2/extra_home/xshe6/Event_camera/event_based/user0" + to_string(current_input_file_id) + "_event_based.csv";
	}
	else{
		image_file = "/hdd2/extra_home/xshe6/Event_camera/event_based/user" + to_string(current_input_file_id) + "_event_based.csv";
	}
	//string image_file = "/hdd2/extra_home/xshe6/Event_camera/event_based/user01_event_based.csv";//"dvs_gesture_event_based_test.csv";

	cout<<endl<<"Image loading"<<endl;
	clock_t load_start = clock();
    std::vector<std::string> folder_list;


    int input_folder_cnt = 0;
    int current_total_read_event = 0;
    int current_file_total_line = 0;
    int current_file_target_line = 0;
	if(input_image_channel==1 || input_image_channel==2){
		current_file_total_line = IBM_DVS128_event_based_count_line(image_file);
		current_total_read_event = IBM_DVS128_event_based_one_line(image_file, events_host, current_file_total_line, current_file_target_line);
		if (current_total_read_event>=img_load_max) current_total_read_event=img_load_max;
		current_file_target_line++;
	    hipMemcpy(events_GPU,events_host,img_load_max*sizeof(Event_Camera_Input),hipMemcpyHostToDevice);
	}else{
		printf("Input channel error.");
		return;
	}
	//current_total_read_event = 20;
	clock_t load_end = clock();
	cout<<"Line: "<<current_file_target_line<<"/"<<current_file_total_line<<", total loaded:"<< current_total_read_event<<endl;
	cout<<endl<<"Image loading done"<<", time used is " << (load_end - load_start)/1000 << " (ms)"<<endl;
	//CIFAR_read_image_one_channel(mnist_img, input_image_signal_channel_size, input_int_2, 0);
	//MNIST_read_image(image_file, mnist_img, training_set_number);


	//CIFAR_read_label(mnist_label, 0);
	//MNIST_read_label(image_label_file, mnist_label, training_set_number);
	//special_function: learn one category
	printf("=0=\n");
	int learn_one_digit = 0;
	int *num_one_digit_img = new int[1];


	//int synapse_size = SIZE*SIZE;
	//cout<<SIZE<<endl;
    Neuron *NeuronList = new Neuron[spiking_neuron_num];
    Input_neuron *Input_neuronlist = new Input_neuron[input_neuron_num];
	//unsigned char *synapse_timer = new unsigned char[synapse_size];  //this is the array that stores timer used in STPD. e.g Neuron x --->  Neuron y Spike! In the array index [(x-1)*SIZE+(y-1)]  => 1
	//hiprandState_t *states;
	//float *random_number_list = new float[SIZE];
	float *log_v_host = new float[MAX_TIME];
	float *log_spike_host = new float[total_depth_number];

	float *log_total_spike_host = new float[SIZE];
	for(int i=0; i < SIZE; i++){
		log_total_spike_host[i] = 0;
	}
	int *spike_flag = new int[CNN_total_layer_num];
	for(int i=0; i < CNN_total_layer_num; i++){
		spike_flag[i] = 0;
	}
	for(int i=0; i<total_depth_number; i++) log_spike_host[i] = 0;
	//init_log_v(log_v_host);
	//init_data_log(log_v_host,log_spike_host,log_total_spike_host, MAX_TIME);
	neuron_list_init(NeuronList, spiking_neuron_num);
	input_neuron_list_init(Input_neuronlist, input_neuron_num);
	printf("=1=\n");
	//
	if(resume_learning){
		printf("RESUME LEARNING\n");
		read_neuron_list(NeuronList, 1, "device2_output_network.txt");
		//read_neuron_list(NeuronList, 1, "spike_cnn.txt");
	}else{
		read_neuron_list(NeuronList, 1, "device2_output_network.txt");
		//read_neuron_list(NeuronList, 1, "device2_output_network.txt");
	}
	bool nomalize_weight = false;
	if(nomalize_weight){
		for(int i=0; i<3; i++){
			float start_depth = network_config->layer[3].first_depth_id - 0.1+i;
			float end_depth = network_config->layer[3].first_depth_id + 0.1+i;

			normalize_weight(NeuronList, start_depth, end_depth, 1, spiking_neuron_num);
		}
	}
    //write_neuron_list(NeuronList, "learning_output_confirm.txt", SIZE);
	//check_neuron(NeuronList, 800, 820);


	//Neuron *old_device_neurons;
	//unsigned char *snapse_timer_device;
	float *log_v;
	float *log_spike;
	float *log_spike_default;
	float *log_total_spike;
	int *spike_flag_device;


    printf("2\n");
	//random number function:

    float rand_list_size_to_total_connection_ratio = 1;
	int rand_numb_size = SPIKING_NEURON_NUM*MAX_CONNECTION;

	int SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
    dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
	dim3 print_grid(1);
	dim3 print_block(1);
	dim3 dimBlock_unit( 1, 1 );
	dim3 dimGrid_unit(1, 1);

	int SIZE_PER_SIDE_whole_network = sqrt(spiking_neuron_num)+1;
    dim3 dimBlock_whole_network( ThreadsPerBlock*2, ThreadsPerBlock );
    dim3 dimGrid_whole_network( (SIZE_PER_SIDE_whole_network/dimBlock.x+1), (SIZE_PER_SIDE_whole_network/dimBlock.y+1));
    printf("2.1\n");

	hiprandState_t *states;

//	if (STOCHASTIC_STDP) rand_init<<<dimGrid,dimBlock>>>(time(0), rand_numb_size, states);
//	float *random_number_list = new float[rand_numb_size];
//	float *random_number_list_device;
//	SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
//	dim3 dimBlock_synapse( ThreadsPerBlock, ThreadsPerBlock );
//	dim3 dimGrid_synapse( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));

//	hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
//	hipMemcpy(random_number_list_device,random_number_list,rand_numb_size*sizeof(float),hipMemcpyHostToDevice);
//	if (STOCHASTIC_STDP) random<<<dimGrid_synapse,dimBlock_synapse>>>(random_number_list_device, rand_numb_size, states);

    hiprandGenerator_t gen_uniform;
    float *random_number_list_device;
    if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING || DEVICE_VARIATION){
    	hipMalloc((void **)&states, rand_numb_size * sizeof(hiprandState_t));
		hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
		hiprandCreateGenerator(&gen_uniform, HIPRAND_RNG_PSEUDO_DEFAULT);
		hiprandSetPseudoRandomGeneratorSeed(gen_uniform, time(0));
		hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);
    }


    hiprandGenerator_t gen_normal;
    float *random_number_normal_device;
    float normal_mean = 0;
    float normal_sd = 5.0;
    if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING || DEVICE_VARIATION){
		hipMalloc((void **)&random_number_normal_device,rand_numb_size*sizeof(float));
		hiprandCreateGenerator(&gen_normal, HIPRAND_RNG_PSEUDO_DEFAULT);
		hiprandSetPseudoRandomGeneratorSeed(gen_normal, time(0));
		hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);
    }


    printf("2.11\n");
    //Setting up input instance matrix:
    float **d_input_instance;
    float **d_convolution_result;
    float **h_input_instance;
    float **h_convolution_result;
    float *probe = new float[1000];
	int instance_array_size = CNN_total_layer_num;
	hipMalloc(&d_input_instance, instance_array_size*sizeof(float *));
	int convolution_result_size = CNN_total_layer_num - 1;
	hipMalloc(&d_convolution_result, convolution_result_size*sizeof(float *));
    h_input_instance = (float**)malloc(instance_array_size * sizeof(float*));
    h_convolution_result = (float**)malloc(convolution_result_size * sizeof(float*));
    CNN_util(network_config, d_input_instance, d_convolution_result, h_input_instance, h_convolution_result, 0);
    printf("2.2\n");
//	float **add = &h_convolution_result[0];
//	printf("Address On GPU: %p\n", add);

    //========Setting up device neuron list============

	Neuron *Neuron_list_device;
    Input_neuron *Input_neuronlist_device;
    hipMalloc((void **)&Neuron_list_device, spiking_neuron_num*sizeof(Neuron));
    hipMalloc((void **)&Input_neuronlist_device, input_neuron_num*sizeof(Input_neuron));
    //hipMalloc((void **)&old_device_neurons, SIZE*sizeof(Neuron));

    //hipMalloc((void **)&states, SIZE * sizeof(hiprandState_t));
    hipMalloc((void **)&log_v, MAX_TIME * sizeof(float));
    hipMalloc((void **)&log_spike, total_depth_number * sizeof(float));
    hipMalloc((void **)&log_spike_default, total_depth_number * sizeof(float));
    //hipMalloc((void **)&log_total_spike, SIZE * sizeof(float));
    gpuErrchk( hipMalloc((void **)&log_total_spike, SIZE * sizeof(float)) );
    hipMalloc((void **)&spike_flag_device, instance_array_size*sizeof(int));
    //rand_init<<<dimGrid,dimBlock>>>(time(0), states);
    printf("2.3\n");
    hipMemcpy(Neuron_list_device,NeuronList,spiking_neuron_num*sizeof(Neuron),hipMemcpyHostToDevice);
    hipMemcpy(Input_neuronlist_device,Input_neuronlist,input_neuron_num*sizeof(Input_neuron),hipMemcpyHostToDevice);
    //hipMemcpy(old_device_neurons,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
    //hipMemcpy(random_number_list_device, random_number_list, SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(log_v,log_v_host,MAX_TIME*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike_default,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
    hipMemcpy(spike_flag_device,spike_flag,instance_array_size*sizeof(int),hipMemcpyHostToDevice);
    printf("3\n");
    //cout<<"network size: "<<SIZE<<endl;
    int network_size = SIZE;

    int max_time = MAX_TIME;
    int first_layer_time = 1;
    int second_layer_time = first_layer_time+(max_time-first_layer_time)*2/3;
    int third_layer_time = first_layer_time+(max_time-first_layer_time)*2/3;
    if(CNN_total_layer_num==3) {
    	second_layer_time = max_time + 1;
    	third_layer_time = max_time + 1;
    }
    if(CNN_total_layer_num==5){
    	first_layer_time = 1;
    	second_layer_time = 2;
    	third_layer_time = 3;
    }
    if(CNN_total_layer_num==4){
    	first_layer_time = 1;
    	second_layer_time = 2;
    	third_layer_time = max_time+1;
    }
    if (CNN_total_layer_num==2){
    	float start_depth = network_config->layer[1].first_depth_id - 0.1;
    	float end_depth = network_config->layer[1].last_depth_id + 0.1;
		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
		update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
		update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
		update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -66.2);
    	first_layer_time = max_time + 1;
    	second_layer_time = max_time + 1;
    	third_layer_time = max_time + 1;
    }
    if(resume_learning){
    	first_layer_time = 100;
    	second_layer_time = max_time;
    }

    //hipMemcpy(Neuron_list_device,old_device_neurons,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
    //first change raw img data into frequency
    int mnist_start_index = 0;
    int mnist_end_index = input_neuron_num;
    //change pixel signal to frequency

    std::srand ( unsigned ( std::time(0) ) );


    //for inference, don't shuffle
	#undef shuffle_image
	#define shuffle_image 0



    hipDeviceSynchronize();

    //data_check(Neuron_list_device,log_total_spike,SIZE,1);
    float *one_mnist_img = new float[input_neuron_num];

    clock_t iter_start, iter_log;
    iter_start = clock();
    int log_interval = MAX_TIME/10;
	bool enable_log_interval = false;
	bool mass_log_spike_out = false;
    //read_filter_GPU_one_layer<<<1, 1>>>(d_network_config, h_filter_array[0], 1);
    //read_filter_GPU<<<1, 1>>>(d_network_config, d_filter_array);

//    int reiter_run = 1;

    int time = 0;
    int training_img_index = 0;
    int training_img_grand_total = 0;

    //============now load all convolution settings===========
	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
		if (layer_iter==0) {
			convolution_kernel_setup(convolution_settings, network_config, layer_iter);
		}else{
			if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel_setup(convolution_settings, network_config, layer_iter);
		}
	}
	copy_filter_to_cuDNN(Neuron_list_device, d_network_config, d_filter_array, spiking_neuron_num);
	cout<<"Filter copy complete"<<endl;
    hipDeviceSynchronize();
	int event_count = 0;
	while (time<=max_time){

		//if(time==first_layer_time)MNIST_drive(NeuronList, Input_neuronlist, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, max_frequency*2, min_frequency, 1);
    	if(DEVICE_VARIATION){
            hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);
    	}
    	if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING){
    	    hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);
    	}

    	if(time%log_interval == 0 && enable_log_interval){
    	    hipMemcpy(NeuronList,Neuron_list_device,spiking_neuron_num*sizeof(Neuron),hipMemcpyDeviceToHost);
    		printf("NN data copy complete\n");
    		string interval_file_name = "device2_output_at_iter_" + to_string(time) + ".txt";
    		//string interval_weight_file_name = to_string(time);
    	    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, (to_string(time)+"_"));
    		if (time>0) {
    			write_neuron_list(NeuronList, interval_file_name, spiking_neuron_num);
    		    hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
    			filter_util(network_config, NeuronList, network_size, input_neuron_num, h_filter_array, d_filter_array, to_string(time), 1);	//write filter to file
    		}
    	}

    	if(time==first_layer_time){

        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.2);

    		hipDeviceSynchronize();
    	}else if(time==second_layer_time){
        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -64);


        	start_depth = network_config->layer[2].first_depth_id - 0.1;
        	end_depth = network_config->layer[2].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63);

        	start_depth = network_config->layer[3].first_depth_id - 0.1;
        	end_depth = network_config->layer[3].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.0);
			cout<<"Changing param, start: "<< start_depth+32<<" end: "<<end_depth<<endl;
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
    		cout<<"Parameter Changing complete.\n";
    		hipDeviceSynchronize();
    	}else if(time==third_layer_time){
        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -60.2);

        	start_depth = network_config->layer[2].first_depth_id - 0.1;
        	end_depth = network_config->layer[2].last_depth_id + 0.1;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -66.0);

        	start_depth = network_config->layer[3].first_depth_id - 0.1;
        	end_depth = network_config->layer[3].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.2);
//    		cout<<"Changing param, start: "<< start_depth+32<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 5, -1.6);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 4, 0.16);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 0, -0.001);

        	start_depth = network_config->layer[4].first_depth_id - 0.1;
        	end_depth = network_config->layer[4].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.0);

    		cout<<"Parameter Changing complete.\n";
    		hipDeviceSynchronize();
    	}

//    	if(time%100==0)cout<<time<<" ";

    	if(time%tenpercent_iter == 0){
    		iter_log = clock();
    		cout<<to_string(10*(time/tenpercent_iter))<<"% done, time used is: " << (iter_log - iter_start)/1000 << " (ms)" << endl;
    	}
    	//fault below here:

    	if(time%time_per_event){//at the beginning of each img's training, load into
    		//cout<<"Image Load Iter: "<<time<<endl;
    		//cout<<";;;"<<training_img_index<<"\n";
    		//if(training_img_index>41445) cout<<training_img_index<<", "<<checked_label_idx<<"!";
    		if(mode_select==0 && time!=0)
    		{
    			event_count++;
    			bool log_this_spike = False;
	    		while(events_host[event_count].valid==False && event_count<current_total_read_event){
	    			event_count++;
	    		}
	    		if (event_count>=current_total_read_event){
	    			//cout<<endl<<"Image loading"<<endl;

	    			if (current_file_target_line>=current_file_total_line){
	    				current_file_target_line = 0;
		    			current_input_file_id ++;

		    			if(current_input_file_id>input_file_id_max) current_input_file_id = 1;

		    			if (current_input_file_id<10) {
		    				image_file = "/hdd2/extra_home/xshe6/Event_camera/event_based/user0" + to_string(current_input_file_id) + "_event_based.csv";
		    			}
		    			else{
		    				image_file = "/hdd2/extra_home/xshe6/Event_camera/event_based/user" + to_string(current_input_file_id) + "_event_based.csv";
		    			}

		    			cout<<"Target File: "<<image_file<<endl;
		    			current_file_total_line = IBM_DVS128_event_based_count_line(image_file);
	    			}

	    			log_this_spike = True;
	    		    current_total_read_event = 0;
	    			current_total_read_event = IBM_DVS128_event_based_one_line(image_file, events_host, current_file_total_line, current_file_target_line);
	    			//current_total_read_event = 20;
	    			if (current_total_read_event>=img_load_max) current_total_read_event=img_load_max;
	    			current_file_target_line++;

					cout<<"Line: "<<current_file_target_line<<"/"<<current_file_total_line<<", total loaded:"<< current_total_read_event<<endl;
					gpuErrchk( hipMemcpy(events_GPU,events_host,img_load_max*sizeof(Event_Camera_Input),hipMemcpyHostToDevice) );
	    			event_count=0;
	    		}

				if(log_this_spike){//print current spike numbers and reset neurons
				    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );
				    ofstream myfile;
				    myfile.open((index_prefix+"inf_multirun_output_spike.csv"), std::ios_base::app);
				    if (myfile.is_open()){
				    	//myfile << "This is a new test\n";
				    	//cout<<"Checking number of neuron spike at image index "<<training_img_index<<endl;

				    	//log last layer
				    	int log_start = 0;
				    	for (int layer_i=1; layer_i<CNN_total_layer_num-1; layer_i++) log_start+=network_config->layer[layer_i].neuron_num;
				    	for(int i=(log_start); i < network_config->layer[CNN_total_layer_num-1].neuron_num+log_start ; i++){
				    		//printf("_%f_", log_v_host[i]);
							myfile << log_total_spike_host[i] << ", ";
//							cout<<log_total_spike_host[i]<<" ";

//								if( (i>=network_config->layer[1].neuron_num) && (i<network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
//									myfile << log_total_spike_host[i] << ", ";
//								}
				    	}

				    	//log any layer
//				    	int log_start = 0;
//				    	int log_end = 0;
//				    	int layer_to_log = 3;
//				    	for (int layer_i=1; layer_i<layer_to_log; layer_i++) log_start+=network_config->layer[layer_i].neuron_num;
//				    	for (int layer_i=1; layer_i<layer_to_log+1; layer_i++) log_end+=network_config->layer[layer_i].neuron_num;
//				    	for(int i=(log_start); i < log_end; i++){
//							myfile << log_total_spike_host[i] << ", ";
//				    	}

//				    	for(int i=0; i < spiking_neuron_num; i++){
//				    		//printf("_%f_", log_v_host[i]);
//				    		if(CNN_total_layer_num==3){
//								if(i>=network_config->layer[1].neuron_num){
//									myfile << log_total_spike_host[i] << ", ";
////									cout<<log_total_spike_host[i]<<" ";
//								}
//				    		}else if(CNN_total_layer_num==4){
//								if(i>=(network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
//									myfile << log_total_spike_host[i] << ", ";
//									cout<<log_total_spike_host[i]<<" ";
//								}
////								if( (i>=network_config->layer[1].neuron_num) && (i<network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
////									myfile << log_total_spike_host[i] << ", ";
////								}
//				    		}
//				    	}
				    	myfile<<endl;
				    	myfile.close();
				    }
				    if (mass_log_spike_out){
						ofstream myfile_2(("./spike_log/mass_logging/" + to_string(training_img_index) + "inf_out_device2_spike_of_neuron_out.csv"));
						if (myfile_2.is_open()){
							//myfile << "This is a new test\n";
					//    	cout<<"Checking number of neuron spike:\n";
							for(int i=0; i < spiking_neuron_num; i++){
								//printf("_%f_", log_v_host[i]);
								myfile_2 << log_total_spike_host[i] << ", ";
					//    		if(i>=network_config->layer[1].neuron_num) cout<<log_total_spike_host[i]<<" ";

							}
							myfile_2.close();
						}
				    }
//					for(int i=0; i < SIZE; i++){
//						log_total_spike_host[i] = 0;
//					}
					std::fill(log_total_spike_host, log_total_spike_host+SIZE, 0);
				    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
				    //print spike numbers done
//				    hipMemcpy(Neuron_list_device,NeuronList,spiking_neuron_num*sizeof(Neuron),hipMemcpyHostToDevice);
//		        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
//		        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
//		    		reset_all_state<<<dimBlock_whole_network, dimGrid_whole_network>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);
//		        	start_depth = network_config->layer[2].first_depth_id - 0.1;
//		        	end_depth = network_config->layer[2].last_depth_id + 0.1;
//		    		reset_all_state<<<dimBlock_whole_network, dimGrid_whole_network>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);

		    		//change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.0);
		    		hipDeviceSynchronize();
				}
    		}
    	}
    	//cout<<"One IMG loaded"<<endl;
    	//enter spiking neuron simulation:
    	int one_layer_neuron_num = 0;
    	if(time<first_layer_time){
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main_event_based(Neuron_list_device, Input_neuronlist_device, events_GPU, event_count, network_config , d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.7*input_float, time, true);
				}
			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {

	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition){

	    	}
	    	else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 1, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}
    	}else if(time<second_layer_time){
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main_event_based(Neuron_list_device, Input_neuronlist_device, events_GPU, event_count, network_config, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.5*input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==2){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 2*input_float, time, true);
				}

			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {

	    	}else if(forced_lateral_inhibition_at_last_layer && CNN_total_layer_num==3){//if this is the last layer, use lateral_inhibition
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition && CNN_total_layer_num!=3){

	    	}
	    	else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}

    	}else{
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main_event_based(Neuron_list_device, Input_neuronlist_device, events_GPU, event_count, network_config, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.5*input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==2){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.5*input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, \
							h_convolution_result, probe);
				}else if(layer_iter==3){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 2*input_float, time, true);
				}

			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {

	    	}else if(forced_lateral_inhibition_at_last_layer && CNN_total_layer_num==3){//if this is the last layer, use lateral_inhibition
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition && CNN_total_layer_num!=3){

	    	}
	    	else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}

    	}
    	time ++;
    }
    //spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, 2);
	//hipDeviceSynchronize();



	filter_util(network_config, Neuron_list_device, spiking_neuron_num, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 2);
    hipMemcpy(NeuronList,Neuron_list_device,spiking_neuron_num*sizeof(Neuron),hipMemcpyDeviceToHost);

    hipMemcpy(log_v_host,log_v,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(log_spike_host,log_spike,total_depth_number*sizeof(float),hipMemcpyDeviceToHost);
    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );


    //print out the synapse conductance data
    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2);

    ofstream myfile ((index_prefix+"inf_out_device2_spike_of_neuron_out.csv"));
    if (myfile.is_open()){
    	//myfile << "This is a new test\n";
    	cout<<"Checking number of neuron spike:\n";
    	for(int i=0; i < spiking_neuron_num; i++){
    		//printf("_%f_", log_v_host[i]);
//    		myfile << log_total_spike_host[i] << ", ";
//    		if(i>=network_config->layer[1].neuron_num) cout<<log_total_spike_host[i]<<" ";

    	}
    	myfile.close();
    }

//    ofstream myfile_p ((index_prefix+"probe.csv"));
//    if (myfile_p.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < 1000; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_p << probe[i] << ", ";
//    	}
//    	myfile_p.close();
//    }

//
//    ofstream myfile_0 ((index_prefix+"device2_out_v.csv"));
//    if (myfile_0.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_0 << log_v_host[i] << ", ";
//    	}
//    	myfile.close();
//    }
//
//    ofstream myfile_2 ((index_prefix+"device2_spike_of_one.csv"));
//    if (myfile_2.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_2 << log_spike_host[i] << ", ";
//    	}
//    	myfile_2.close();
//    }



    hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
	//filter_util(network_config, NeuronList, network_size, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 1);	//write filter to file
    write_neuron_list(NeuronList, ("inf_out_device2_output_network.txt"), spiking_neuron_num);
    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, "");
    //===clean up===
    //delete[] random_number_list;
    delete[] log_v_host;
	delete[] NeuronList;
	delete[] log_spike_host;
	delete[] log_total_spike_host;
	delete[] NeuronList_temp;
	delete[] one_mnist_img;
	delete[] probe;
//	delete[] random_number_list;
	delete[] spike_flag;
	delete[] num_one_digit_img;
	//hipFree(states);
	hipFree(log_v);
	hipFree(log_spike);
	hipFree(log_total_spike);
	hipFree(Neuron_list_device);
	//hipFree(old_device_neurons);
	hipFree(random_number_list_device);
	hipFree(d_network_config);
	hipFree(states);
	hipFree(spike_flag_device);
	hipFree(log_spike_default);



}


void run_event_based_inference_hsnn(string index_prefix, float input_float, float input_float_2, int input_int, int input_int_2, string input_img){
	cout << "Running CNN Multilayer Inference" << endl << endl;
	cout<<"Functions: \n"<<"0. load from regular event file\n";
	cout << endl;
	cout<<"Function Select: ";
	int mode_select;
	cin >> mode_select;

	switch (mode_select){
		case 0:
		{
			cout<<"-load from regular event file- selected"<<endl;
			cout<<endl;
		}
		break;
		case 1:
		{
			printf("Case 1 selected, no such option");
			return;
		}
		break;
		case 2:
		{
			printf("Case 2 selected, no such option");
			return;
		}
		break;
		case 3:
		{
			printf("Case 3 selected, no such option");
			return;
		}
		break;
	}



	/*
	int training_set_number = 1;
	int size_per_img = input_image_w * input_image_l*input_image_channel;
	float *mnist_img = new float[size_per_img*training_set_number];
	string image_file = "train-images-idx3-ubyte"; //"train-images-idx3-ubyte";
	MNIST_read_image(image_file, mnist_img, training_set_number);
	int *mnist_label = new int[training_set_number];
	string image_label_file = "train-labels-idx1-ubyte";
	MNIST_read_label(image_label_file, mnist_label, training_set_number);

	float *filter;
	float *output = new float[size_per_img*training_set_number];

	convolution_kernel(mnist_img, filter, output);
	img_util(output, "test_output.jpg", 0);
	*/
	int resume_learning = 0;
	CNN_struct *network_config = new CNN_struct;
	int depth_list[3] = {32,64,32};
	hsnn_config_generator(depth_list, network_config);

	Neuron *NeuronList_temp = new Neuron[1];
	CNN_struct *d_network_config;
	hipMalloc((void **)&d_network_config,sizeof(CNN_struct));
	hipMemcpy(d_network_config,network_config,sizeof(CNN_struct),hipMemcpyHostToDevice);
	int total_depth_number = 0;
	for(int i=0;i<CNN_total_layer_num; i++){
		total_depth_number = total_depth_number + network_config->layer[i].depth;
		cout<<"depth number: "<<network_config->layer[i].depth<<endl;
	}

	cout<<endl<<"Total depth number: "<<total_depth_number<<endl;
	float **h_filter_array;
	float **d_filter_array;
	int filter_array_size = CNN_total_layer_num-1;
	hipMalloc(&d_filter_array, filter_array_size*sizeof(float *));
	h_filter_array = (float**)malloc(filter_array_size * sizeof(float*));
	filter_util(network_config, NeuronList_temp, 0,0,  h_filter_array, d_filter_array, index_prefix, 0);

	/*
	img_util(mnist_img, "tensorflow_small.png", 1);
	img_util(mnist_img, "test_output_-1.png", 0);

	float *output = new float[size_per_img*training_set_number];

	int image_bytes = input_image_channel * input_image_l * input_image_w * sizeof(float);

	float* convolution_device_input{nullptr};
	hipMalloc(&convolution_device_input, image_bytes);
	hipMemcpy(convolution_device_input, mnist_img, image_bytes, hipMemcpyHostToDevice);

	int filter_in_channel = input_image_channel;
	int filter_out_channel = input_image_channel;
	int filter_height = 3;
	int filter_width = 3;
	const float kernel_template[3][3] = {
	{1, 1, 1},
	{1, -8, 1},
	{1, 1, 1}
	};
	float h_kernel[filter_in_channel][filter_out_channel][filter_height][filter_width];
	for (int kernel = 0; kernel < filter_in_channel; ++kernel) {
		for (int channel = 0; channel < filter_out_channel; ++channel) {
		  for (int row = 0; row < filter_height; ++row) {
			for (int column = 0; column < filter_width; ++column) {
			  h_kernel[kernel][channel][row][column] = kernel_template[row][column];
			}
		  }
		}
	}
	float* filter{nullptr};
	hipMalloc(&filter, sizeof(h_kernel));
	hipMemcpy(filter, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);


	convolution_kernel(convolution_device_input, filter, output);
	img_util(output, "test_output_1.png", 0);

	hipFree(filter);
	hipFree(convolution_device_input);

	*/

//===========END of CNN special setting-up phase============

	Convolution_setting_struct *convolution_settings = new Convolution_setting_struct[CNN_total_layer_num];

	//set parameters
	int per_run_img_num=6;
	float total_event_to_infer = 23;
	vector<int> labels;
	vector<long> frame_nums;
	int checked_label_idx = 0;

	if(mode_select==0){
		cout<<"how many million events to run: ";
		cin>>total_event_to_infer;
	}
	else if(mode_select==1){

	}else if(mode_select==2){

	}else if(mode_select==3){

	}

    static int img_load_max  = 10000000;

	int time_per_event = input_int;
	int calculated_total_time = time_per_event*total_event_to_infer*1000000;
	#undef MAX_TIME
	#define MAX_TIME calculated_total_time
	printf("==Training Total Iter: %d==", MAX_TIME);
	int total_neuron_num = 0;
	int total_spiking_num = 0;
	for(int i=0;i<CNN_total_layer_num;i++){
		total_neuron_num += network_config->layer[i].neuron_num;
		if(i!=0) total_spiking_num += network_config->layer[i].neuron_num;
	}
	total_spiking_num += 10;
	total_neuron_num += 10;
	//total_neuron_num = 20000;
	cout<<endl<<"total neuron num: "<<total_neuron_num<<endl;
	cout<<"total spiking neuron num: "<<total_spiking_num<<endl;
	#undef SIZE
	#define SIZE total_neuron_num
	#undef SPIKING_NEURON_NUM
	#define SPIKING_NEURON_NUM total_spiking_num


	float max_frequency = 100; //in Hz default 22
	float min_frequency = 10;

	int input_neuron_num = input_image_w*input_image_l*input_image_channel;
	int input_image_signal_channel_size  = input_image_w*input_image_l;
	int spiking_neuron_num = SPIKING_NEURON_NUM;
	int output_layer_neuron_num = OUTPUT_LAYER_NEURON_NUM;
	int tenpercent_iter = MAX_TIME/10;
	int connection_size = MAX_CONNECTION;
	int syn_timer_max = 25;
	int input_signal_width = 10;	//default 25
	int inhibition_time = 10;	//default 10

	float target_frequency_param = 0.5;
	float target_frequency = 100;


	Event_Camera_Input *events_host = new Event_Camera_Input[img_load_max];
	Event_Camera_Input *events_GPU;
	hipMalloc((void **)&events_GPU,img_load_max*sizeof(Event_Camera_Input));

	int current_input_file_id = 8;
	int input_file_id_max = 20;
	string image_file = "";
	if (current_input_file_id<10) {
		image_file = "/hdd2/extra_home/xshe6/Event_camera/event_based/user0" + to_string(current_input_file_id) + "_event_based.csv";
	}
	else{
		image_file = "/hdd2/extra_home/xshe6/Event_camera/event_based/user" + to_string(current_input_file_id) + "_event_based.csv";
	}
	//string image_file = "/hdd2/extra_home/xshe6/Event_camera/event_based/user01_event_based.csv";//"dvs_gesture_event_based_test.csv";

	cout<<endl<<"Image loading"<<endl;
	clock_t load_start = clock();
    std::vector<std::string> folder_list;


    int input_folder_cnt = 0;
    int current_total_read_event = 0;
    int current_file_total_line = 0;
    int current_file_target_line = 0;
	if(input_image_channel==1 || input_image_channel==2){
		current_file_total_line = IBM_DVS128_event_based_count_line(image_file);
		current_total_read_event = IBM_DVS128_event_based_one_line(image_file, events_host, current_file_total_line, current_file_target_line);
		if (current_total_read_event>=img_load_max) current_total_read_event=img_load_max;
		current_file_target_line++;
	    hipMemcpy(events_GPU,events_host,img_load_max*sizeof(Event_Camera_Input),hipMemcpyHostToDevice);
	}else{
		printf("Input channel error.");
		return;
	}
	//current_total_read_event = 20;
	clock_t load_end = clock();
	cout<<"Line: "<<current_file_target_line<<"/"<<current_file_total_line<<", total loaded:"<< current_total_read_event<<endl;
	cout<<endl<<"Image loading done"<<", time used is " << (load_end - load_start)/1000 << " (ms)"<<endl;
	//CIFAR_read_image_one_channel(mnist_img, input_image_signal_channel_size, input_int_2, 0);
	//MNIST_read_image(image_file, mnist_img, training_set_number);


	//CIFAR_read_label(mnist_label, 0);
	//MNIST_read_label(image_label_file, mnist_label, training_set_number);
	//special_function: learn one category
	printf("=0=\n");
	int learn_one_digit = 0;
	int *num_one_digit_img = new int[1];


	//int synapse_size = SIZE*SIZE;
	//cout<<SIZE<<endl;
    Neuron *NeuronList = new Neuron[spiking_neuron_num];
    Input_neuron *Input_neuronlist = new Input_neuron[input_neuron_num];
	//unsigned char *synapse_timer = new unsigned char[synapse_size];  //this is the array that stores timer used in STPD. e.g Neuron x --->  Neuron y Spike! In the array index [(x-1)*SIZE+(y-1)]  => 1
	//hiprandState_t *states;
	//float *random_number_list = new float[SIZE];
	float *log_v_host = new float[MAX_TIME];
	float *log_spike_host = new float[total_depth_number];

	float *log_total_spike_host = new float[SIZE];
	for(int i=0; i < SIZE; i++){
		log_total_spike_host[i] = 0;
	}
	int *spike_flag = new int[CNN_total_layer_num];
	for(int i=0; i < CNN_total_layer_num; i++){
		spike_flag[i] = 0;
	}
	for(int i=0; i<total_depth_number; i++) log_spike_host[i] = 0;
	//init_log_v(log_v_host);
	//init_data_log(log_v_host,log_spike_host,log_total_spike_host, MAX_TIME);
	neuron_list_init(NeuronList, spiking_neuron_num);
	input_neuron_list_init(Input_neuronlist, input_neuron_num);

	printf("=1=\n");
	//
	if(resume_learning){
		printf("RESUME LEARNING\n");
		read_neuron_list(NeuronList, 1, "3device2_output_network.txt");
		//read_neuron_list(NeuronList, 1, "spike_cnn.txt");
	}else{
		read_neuron_list(NeuronList, 1, "3device2_output_network.txt");
		//read_neuron_list(NeuronList, 1, "device2_output_network.txt");
	}
	bool nomalize_weight = false;
	if(nomalize_weight){
		for(int i=0; i<3; i++){
			float start_depth = network_config->layer[3].first_depth_id - 0.1+i;
			float end_depth = network_config->layer[3].first_depth_id + 0.1+i;

			normalize_weight(NeuronList, start_depth, end_depth, 1, spiking_neuron_num);
		}
	}
    //write_neuron_list(NeuronList, "learning_output_confirm.txt", SIZE);
	//check_neuron(NeuronList, 800, 820);


	//Neuron *old_device_neurons;
	//unsigned char *snapse_timer_device;
	float *log_v;
	float *log_spike;
	float *log_spike_default;
	float *log_total_spike;
	int *spike_flag_device;


    printf("2\n");
	//random number function:

    float rand_list_size_to_total_connection_ratio = 1;
	int rand_numb_size = SPIKING_NEURON_NUM*MAX_CONNECTION;

	int SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
    dim3 dimBlock( ThreadsPerBlock, ThreadsPerBlock );
    dim3 dimGrid( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));
	dim3 print_grid(1);
	dim3 print_block(1);
	dim3 dimBlock_unit( 1, 1 );
	dim3 dimGrid_unit(1, 1);

	int SIZE_PER_SIDE_whole_network = sqrt(spiking_neuron_num)+1;
    dim3 dimBlock_whole_network( ThreadsPerBlock*2, ThreadsPerBlock );
    dim3 dimGrid_whole_network( (SIZE_PER_SIDE_whole_network/dimBlock.x+1), (SIZE_PER_SIDE_whole_network/dimBlock.y+1));
    printf("2.1\n");

	hiprandState_t *states;

//	if (STOCHASTIC_STDP) rand_init<<<dimGrid,dimBlock>>>(time(0), rand_numb_size, states);
//	float *random_number_list = new float[rand_numb_size];
//	float *random_number_list_device;
//	SIZE_PER_SIDE = sqrt(rand_numb_size)+1;
//	dim3 dimBlock_synapse( ThreadsPerBlock, ThreadsPerBlock );
//	dim3 dimGrid_synapse( (SIZE_PER_SIDE/dimBlock.x+1), (SIZE_PER_SIDE/dimBlock.y+1));

//	hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
//	hipMemcpy(random_number_list_device,random_number_list,rand_numb_size*sizeof(float),hipMemcpyHostToDevice);
//	if (STOCHASTIC_STDP) random<<<dimGrid_synapse,dimBlock_synapse>>>(random_number_list_device, rand_numb_size, states);

    hiprandGenerator_t gen_uniform;
    float *random_number_list_device;
    if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING || DEVICE_VARIATION){
    	hipMalloc((void **)&states, rand_numb_size * sizeof(hiprandState_t));
		hipMalloc((void **)&random_number_list_device,rand_numb_size*sizeof(float));
		hiprandCreateGenerator(&gen_uniform, HIPRAND_RNG_PSEUDO_DEFAULT);
		hiprandSetPseudoRandomGeneratorSeed(gen_uniform, time(0));
		hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);
    }


    hiprandGenerator_t gen_normal;
    float *random_number_normal_device;
    float normal_mean = 0;
    float normal_sd = 5.0;
    if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING || DEVICE_VARIATION){
		hipMalloc((void **)&random_number_normal_device,rand_numb_size*sizeof(float));
		hiprandCreateGenerator(&gen_normal, HIPRAND_RNG_PSEUDO_DEFAULT);
		hiprandSetPseudoRandomGeneratorSeed(gen_normal, time(0));
		hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);
    }


    printf("2.11\n");
    //Setting up input instance matrix:
    float **d_input_instance;
    float **d_convolution_result;
    float **h_input_instance;
    float **h_convolution_result;
    float *probe = new float[1000];
	int instance_array_size = CNN_total_layer_num;
	hipMalloc(&d_input_instance, instance_array_size*sizeof(float *));
	int convolution_result_size = CNN_total_layer_num - 1;
	hipMalloc(&d_convolution_result, convolution_result_size*sizeof(float *));
    h_input_instance = (float**)malloc(instance_array_size * sizeof(float*));
    h_convolution_result = (float**)malloc(convolution_result_size * sizeof(float*));
    CNN_util(network_config, d_input_instance, d_convolution_result, h_input_instance, h_convolution_result, 0);
    printf("2.2\n");
//	float **add = &h_convolution_result[0];
//	printf("Address On GPU: %p\n", add);

    //========Setting up device neuron list============

	Neuron *Neuron_list_device;
    Input_neuron *Input_neuronlist_device;
    hipMalloc((void **)&Neuron_list_device, spiking_neuron_num*sizeof(Neuron));
    hipMalloc((void **)&Input_neuronlist_device, input_neuron_num*sizeof(Input_neuron));
    //hipMalloc((void **)&old_device_neurons, SIZE*sizeof(Neuron));

    //hipMalloc((void **)&states, SIZE * sizeof(hiprandState_t));
    hipMalloc((void **)&log_v, MAX_TIME * sizeof(float));
    hipMalloc((void **)&log_spike, total_depth_number * sizeof(float));
    hipMalloc((void **)&log_spike_default, total_depth_number * sizeof(float));
    //hipMalloc((void **)&log_total_spike, SIZE * sizeof(float));
    gpuErrchk( hipMalloc((void **)&log_total_spike, SIZE * sizeof(float)) );
    hipMalloc((void **)&spike_flag_device, instance_array_size*sizeof(int));
    //rand_init<<<dimGrid,dimBlock>>>(time(0), states);
    printf("2.3\n");
    hipMemcpy(Neuron_list_device,NeuronList,spiking_neuron_num*sizeof(Neuron),hipMemcpyHostToDevice);
    hipMemcpy(Input_neuronlist_device,Input_neuronlist,input_neuron_num*sizeof(Input_neuron),hipMemcpyHostToDevice);
    //hipMemcpy(old_device_neurons,NeuronList,SIZE*sizeof(Neuron),hipMemcpyHostToDevice);
    //hipMemcpy(random_number_list_device, random_number_list, SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(log_v,log_v_host,MAX_TIME*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(log_spike_default,log_spike_host,total_depth_number*sizeof(float),hipMemcpyHostToDevice);
    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
    hipMemcpy(spike_flag_device,spike_flag,instance_array_size*sizeof(int),hipMemcpyHostToDevice);
    printf("3\n");
    //cout<<"network size: "<<SIZE<<endl;
    int network_size = SIZE;

    int max_time = MAX_TIME;
    int first_layer_time = 1;
    int second_layer_time = first_layer_time+(max_time-first_layer_time)*2/3;
    int third_layer_time = first_layer_time+(max_time-first_layer_time)*2/3;
    if(CNN_total_layer_num==3) {
    	second_layer_time = max_time + 1;
    	third_layer_time = max_time + 1;
    }
    if(CNN_total_layer_num==5){
    	first_layer_time = 1;
    	second_layer_time = 2;
    	third_layer_time = 3;
    }
    if(CNN_total_layer_num==4){
    	first_layer_time = 1;
    	second_layer_time = 2;
    	third_layer_time = max_time+1;
    }
    if (CNN_total_layer_num==2){
    	float start_depth = network_config->layer[1].first_depth_id - 0.1;
    	float end_depth = network_config->layer[1].last_depth_id + 0.1;
		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
		update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
		update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
		update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -66.2);
    	first_layer_time = max_time + 1;
    	second_layer_time = max_time + 1;
    	third_layer_time = max_time + 1;
    }
    if(resume_learning){
    	first_layer_time = 100;
    	second_layer_time = max_time;
    }

    //hipMemcpy(Neuron_list_device,old_device_neurons,sizeof(Neuron)*SIZE,hipMemcpyDeviceToDevice);
    //first change raw img data into frequency
    int mnist_start_index = 0;
    int mnist_end_index = input_neuron_num;
    //change pixel signal to frequency

    std::srand ( unsigned ( std::time(0) ) );


    //for inference, don't shuffle
	#undef shuffle_image
	#define shuffle_image 0



    hipDeviceSynchronize();

    //data_check(Neuron_list_device,log_total_spike,SIZE,1);
    float *one_mnist_img = new float[input_neuron_num];

    clock_t iter_start, iter_log;
    iter_start = clock();
    int log_interval = MAX_TIME/10;
	bool enable_log_interval = false;
	bool mass_log_spike_out = false;
    //read_filter_GPU_one_layer<<<1, 1>>>(d_network_config, h_filter_array[0], 1);
    //read_filter_GPU<<<1, 1>>>(d_network_config, d_filter_array);

//    int reiter_run = 1;

    int time = 0;
    int training_img_index = 0;
    int training_img_grand_total = 0;

    //============now load all convolution settings===========
	for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
		if (layer_iter==0) {
			convolution_kernel_setup(convolution_settings, network_config, layer_iter);
		}else{
			if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel_setup(convolution_settings, network_config, layer_iter);
		}
	}
	copy_filter_to_cuDNN(Neuron_list_device, d_network_config, d_filter_array, spiking_neuron_num);
	cout<<"Filter copy complete"<<endl;
    hipDeviceSynchronize();
	int event_count = 0;
	while (time<=max_time){

		//if(time==first_layer_time)MNIST_drive(NeuronList, Input_neuronlist, mnist_img, network_size, training_set_number, mnist_start_index, mnist_end_index, max_frequency*2, min_frequency, 1);
    	if(DEVICE_VARIATION){
            hiprandGenerateNormal(gen_normal, random_number_normal_device, rand_numb_size, normal_mean, normal_sd);
    	}
    	if(STOCHASTIC_STDP || STOCHASTIC_ROUNDING){
    	    hiprandGenerateUniform(gen_uniform, random_number_list_device, rand_numb_size);
    	}

    	if(time%log_interval == 0 && enable_log_interval){
    	    hipMemcpy(NeuronList,Neuron_list_device,spiking_neuron_num*sizeof(Neuron),hipMemcpyDeviceToHost);
    		printf("NN data copy complete\n");
    		string interval_file_name = "device2_output_at_iter_" + to_string(time) + ".txt";
    		//string interval_weight_file_name = to_string(time);
    	    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, (to_string(time)+"_"));
    		if (time>0) {
    			write_neuron_list(NeuronList, interval_file_name, spiking_neuron_num);
    		    hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
    			filter_util(network_config, NeuronList, network_size, input_neuron_num, h_filter_array, d_filter_array, to_string(time), 1);	//write filter to file
    		}
    	}

    	if(time==first_layer_time){

        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -2.07);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, 0.02);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -65.2);
    		cout<<"Changing param of long-term neuron, start: "<< start_depth+16<<" end: "<<end_depth<<endl;
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+depth_list[0]/2, end_depth, 5, -1.6);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+depth_list[0]/2, end_depth, 4, 0.4);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+depth_list[0]/2, end_depth, 0, 0.001);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+depth_list[0]/2, end_depth, -64.2);

    		hipDeviceSynchronize();
    	}else if(time==second_layer_time){
        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -64);


        	start_depth = network_config->layer[2].first_depth_id - 0.1;
        	end_depth = network_config->layer[2].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -2.07);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, 0.02);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -65.2);
    		cout<<"Changing param of long-term neuron, start: "<< start_depth+16<<" end: "<<end_depth<<endl;
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+depth_list[1]/2, end_depth, 5, -1.6);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+depth_list[1]/2, end_depth, 4, 0.4);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+depth_list[1]/2, end_depth, 0, 0.001);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+depth_list[1]/2, end_depth, -64.2);

        	start_depth = network_config->layer[3].first_depth_id - 0.1;
        	end_depth = network_config->layer[3].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.0);
			cout<<"Changing param, start: "<< start_depth+32<<" end: "<<end_depth<<endl;
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -1.6);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.4);
			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, 0.001);
    		cout<<"Parameter Changing complete.\n";
    		hipDeviceSynchronize();
    	}else if(time==third_layer_time){
        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -60.2);

        	start_depth = network_config->layer[2].first_depth_id - 0.1;
        	end_depth = network_config->layer[2].last_depth_id + 0.1;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -66.0);

        	start_depth = network_config->layer[3].first_depth_id - 0.1;
        	end_depth = network_config->layer[3].last_depth_id + 0.1;
    		//cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 5, -5.07);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 4, 0.453);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, 0, -0.02);
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.2);
//    		cout<<"Changing param, start: "<< start_depth+32<<" end: "<<end_depth<<endl;
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 5, -1.6);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 4, 0.16);
//			update_param<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth+128, end_depth, 0, -0.001);

        	start_depth = network_config->layer[4].first_depth_id - 0.1;
        	end_depth = network_config->layer[4].last_depth_id + 0.1;
    		cout<<"Changing threshold, start: "<< start_depth<<" end: "<<end_depth<<endl;
    		change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.0);

    		cout<<"Parameter Changing complete.\n";
    		hipDeviceSynchronize();
    	}

//    	if(time%100==0)cout<<time<<" ";

    	if(time%tenpercent_iter == 0){
    		iter_log = clock();
    		cout<<to_string(10*(time/tenpercent_iter))<<"% done, time used is: " << (iter_log - iter_start)/1000 << " (ms)" << endl;
    	}
    	//fault below here:

    	if(time%time_per_event){//at the beginning of each img's training, load into
    		//cout<<"Image Load Iter: "<<time<<endl;
    		//cout<<";;;"<<training_img_index<<"\n";
    		//if(training_img_index>41445) cout<<training_img_index<<", "<<checked_label_idx<<"!";
    		if(mode_select==0 && time!=0)
    		{
    			event_count++;
    			bool log_this_spike = False;
    			int length_normalizer = 1;
	    		while(events_host[event_count].valid==False && event_count<current_total_read_event){
	    			event_count++;
	    		}
	    		if (event_count>=current_total_read_event){
	    			//cout<<endl<<"Image loading"<<endl;

	    			if (current_file_target_line>=current_file_total_line){
	    				current_file_target_line = 0;
		    			current_input_file_id ++;

		    			if(current_input_file_id>input_file_id_max) current_input_file_id = 1;

		    			if (current_input_file_id<10) {
		    				image_file = "/hdd2/extra_home/xshe6/Event_camera/event_based/user0" + to_string(current_input_file_id) + "_event_based.csv";
		    			}
		    			else{
		    				image_file = "/hdd2/extra_home/xshe6/Event_camera/event_based/user" + to_string(current_input_file_id) + "_event_based.csv";
		    			}

		    			cout<<"Target File: "<<image_file<<endl;
		    			current_file_total_line = IBM_DVS128_event_based_count_line(image_file);
	    			}

	    			log_this_spike = True;
	    			length_normalizer = current_total_read_event;
	    		    current_total_read_event = 0;
	    			current_total_read_event = IBM_DVS128_event_based_one_line(image_file, events_host, current_file_total_line, current_file_target_line);
	    			//current_total_read_event = 20;
	    			if (current_total_read_event>=img_load_max) current_total_read_event=img_load_max;
	    			current_file_target_line++;

					cout<<"Line: "<<current_file_target_line<<"/"<<current_file_total_line<<", total loaded:"<< current_total_read_event<<endl;
					gpuErrchk( hipMemcpy(events_GPU,events_host,img_load_max*sizeof(Event_Camera_Input),hipMemcpyHostToDevice) );
	    			event_count=0;
	    		}

				if(log_this_spike){//print current spike numbers and reset neurons
				    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );
				    ofstream myfile;
				    myfile.open((index_prefix+"inf_multirun_output_spike.csv"), std::ios_base::app);
				    if (myfile.is_open()){
				    	//myfile << "This is a new test\n";
				    	//cout<<"Checking number of neuron spike at image index "<<training_img_index<<endl;

				    	//log last layer
				    	int log_start = 0;
				    	for (int layer_i=1; layer_i<CNN_total_layer_num-1; layer_i++) log_start+=network_config->layer[layer_i].neuron_num;
				    	for(int i=(log_start); i < network_config->layer[CNN_total_layer_num-1].neuron_num+log_start ; i++){
				    		//printf("_%f_", log_v_host[i]);
							myfile << log_total_spike_host[i] << ", ";
//							cout<<log_total_spike_host[i]<<" ";

//								if( (i>=network_config->layer[1].neuron_num) && (i<network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
//									myfile << log_total_spike_host[i] << ", ";
//								}
				    	}

				    	//log any layer
//				    	int log_start = 0;
//				    	int log_end = 0;
//				    	int layer_to_log = 3;
//				    	for (int layer_i=1; layer_i<layer_to_log; layer_i++) log_start+=network_config->layer[layer_i].neuron_num;
//				    	for (int layer_i=1; layer_i<layer_to_log+1; layer_i++) log_end+=network_config->layer[layer_i].neuron_num;
//				    	for(int i=(log_start); i < log_end; i++){
//							myfile << log_total_spike_host[i] << ", ";
//				    	}

//				    	for(int i=0; i < spiking_neuron_num; i++){
//				    		//printf("_%f_", log_v_host[i]);
//				    		if(CNN_total_layer_num==3){
//								if(i>=network_config->layer[1].neuron_num){
//									myfile << log_total_spike_host[i] << ", ";
////									cout<<log_total_spike_host[i]<<" ";
//								}
//				    		}else if(CNN_total_layer_num==4){
//								if(i>=(network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
//									myfile << log_total_spike_host[i] << ", ";
//									cout<<log_total_spike_host[i]<<" ";
//								}
////								if( (i>=network_config->layer[1].neuron_num) && (i<network_config->layer[1].neuron_num+network_config->layer[2].neuron_num)){
////									myfile << log_total_spike_host[i] << ", ";
////								}
//				    		}
//				    	}
				    	myfile<<endl;
				    	myfile.close();
				    }
				    if (mass_log_spike_out){
						ofstream myfile_2(("./spike_log/mass_logging/" + to_string(training_img_index) + "inf_out_device2_spike_of_neuron_out.csv"));
						if (myfile_2.is_open()){
							//myfile << "This is a new test\n";
					//    	cout<<"Checking number of neuron spike:\n";
							for(int i=0; i < spiking_neuron_num; i++){
								//printf("_%f_", log_v_host[i]);
								myfile_2 << log_total_spike_host[i] << ", ";
					//    		if(i>=network_config->layer[1].neuron_num) cout<<log_total_spike_host[i]<<" ";

							}
							myfile_2.close();
						}
				    }
//					for(int i=0; i < SIZE; i++){
//						log_total_spike_host[i] = 0;
//					}
					std::fill(log_total_spike_host, log_total_spike_host+SIZE, 0);
				    gpuErrchk( hipMemcpy(log_total_spike,log_total_spike_host,SIZE*sizeof(float),hipMemcpyHostToDevice) );
				    //print spike numbers done
//				    hipMemcpy(Neuron_list_device,NeuronList,spiking_neuron_num*sizeof(Neuron),hipMemcpyHostToDevice);
//		        	float start_depth = network_config->layer[1].first_depth_id - 0.1;
//		        	float end_depth = network_config->layer[1].last_depth_id + 0.1;
//		    		reset_all_state<<<dimBlock_whole_network, dimGrid_whole_network>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);
//		        	start_depth = network_config->layer[2].first_depth_id - 0.1;
//		        	end_depth = network_config->layer[2].last_depth_id + 0.1;
//		    		reset_all_state<<<dimBlock_whole_network, dimGrid_whole_network>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth);

		    		//change_threshold<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, start_depth, end_depth, -63.0);
		    		hipDeviceSynchronize();
				}
    		}
    	}
    	//cout<<"One IMG loaded"<<endl;
    	//enter spiking neuron simulation:
    	int one_layer_neuron_num = 0;
    	if(time<first_layer_time){
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main_event_based(Neuron_list_device, Input_neuronlist_device, events_GPU, event_count, network_config, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.7*input_float, time, true);
				}
			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {

	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition){

	    	}
	    	else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 1, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}
    	}else if(time<second_layer_time){
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main_event_based(Neuron_list_device, Input_neuronlist_device, events_GPU, event_count, network_config, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.5*input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==2){
					spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 2*input_float, time, true);
				}

			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {

	    	}else if(forced_lateral_inhibition_at_last_layer && CNN_total_layer_num==3){//if this is the last layer, use lateral_inhibition
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition && CNN_total_layer_num!=3){

	    	}
	    	else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}

    	}else{
			for(int layer_iter=0;layer_iter<CNN_total_layer_num;layer_iter++){
				one_layer_neuron_num = network_config->layer[layer_iter].neuron_num;
				int convolution_result_index = layer_iter - 1;
				if (layer_iter==0) {//fault at convolution kernel and spiking cnn
					convolution_result_index = 0;
					spiking_cnn_main_event_based(Neuron_list_device, Input_neuronlist_device, events_GPU, event_count, network_config, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, input_float, time, false);
					convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==1){
					//spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.5*input_float, time, false);
					spiking_cnn_main_event_based(Neuron_list_device, Input_neuronlist_device, events_GPU, event_count, network_config, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.5*input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, h_convolution_result, probe);
				}else if(layer_iter==2){
					//spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.5*input_float, time, false);
					spiking_cnn_main_event_based(Neuron_list_device, Input_neuronlist_device, events_GPU, event_count, network_config, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 0.5*input_float, time, false);
					if (layer_iter!=(CNN_total_layer_num-1)) convolution_kernel(convolution_settings[layer_iter], layer_iter, h_input_instance, h_filter_array, \
							h_convolution_result, probe);
				}else if(layer_iter==3){
					//spiking_cnn_main(Neuron_list_device, Input_neuronlist_device, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 2*input_float, time, true);
					spiking_cnn_main_event_based(Neuron_list_device, Input_neuronlist_device, events_GPU, event_count, network_config, d_network_config, random_number_list_device, d_convolution_result, d_input_instance, \
							layer_iter, network_size, input_neuron_num, log_v, log_spike, log_total_spike, spike_flag_device, input_signal_width, 2*input_float, time, true);
				}

			}
			//=================TRY WITH LAYER wise inhibition=====================
	    	if(depth_wise_inhibition) {

	    	}else if(forced_lateral_inhibition_at_last_layer && CNN_total_layer_num==3){//if this is the last layer, use lateral_inhibition
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}else if(through_depth_inhibition){

	    	}else if(apply_local_inhibition && CNN_total_layer_num!=3){

	    	}
	    	else{
	    		lateral_inhibition_mother_thread<<<dimBlock_unit, dimGrid_unit>>>(Neuron_list_device, spiking_neuron_num, 2, inhibition_time, d_network_config, spike_flag_device);
	    	}
			if(HOMEOSTASIS_ENABLE){
				if(time%HOMEOSTASIS_UPDATE_FREQUENCY == 0 && time != 0){
					//spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, log_total_spike, target_frequency, time, log_spike, 0, 1);
				}
			}

    	}
    	time ++;
    }
    //spiking_learning_drive(Neuron_list_device, network_size, inhibition_time, 2);
	//hipDeviceSynchronize();



	filter_util(network_config, Neuron_list_device, spiking_neuron_num, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 2);
    hipMemcpy(NeuronList,Neuron_list_device,spiking_neuron_num*sizeof(Neuron),hipMemcpyDeviceToHost);

    hipMemcpy(log_v_host,log_v,MAX_TIME*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(log_spike_host,log_spike,total_depth_number*sizeof(float),hipMemcpyDeviceToHost);
    gpuErrchk( hipMemcpy(log_total_spike_host,log_total_spike,SIZE*sizeof(float),hipMemcpyDeviceToHost) );


    //print out the synapse conductance data
    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2);

    ofstream myfile ((index_prefix+"inf_out_device2_spike_of_neuron_out.csv"));
    if (myfile.is_open()){
    	//myfile << "This is a new test\n";
    	cout<<"Checking number of neuron spike:\n";
    	for(int i=0; i < spiking_neuron_num; i++){
    		//printf("_%f_", log_v_host[i]);
//    		myfile << log_total_spike_host[i] << ", ";
//    		if(i>=network_config->layer[1].neuron_num) cout<<log_total_spike_host[i]<<" ";

    	}
    	myfile.close();
    }

//    ofstream myfile_p ((index_prefix+"probe.csv"));
//    if (myfile_p.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < 1000; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_p << probe[i] << ", ";
//    	}
//    	myfile_p.close();
//    }

//
//    ofstream myfile_0 ((index_prefix+"device2_out_v.csv"));
//    if (myfile_0.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_0 << log_v_host[i] << ", ";
//    	}
//    	myfile.close();
//    }
//
//    ofstream myfile_2 ((index_prefix+"device2_spike_of_one.csv"));
//    if (myfile_2.is_open()){
//    	//myfile << "This is a new test\n";
//    	for(int i=0; i < MAX_TIME; i++){
//    		//printf("_%f_", log_v_host[i]);
//    		myfile_2 << log_spike_host[i] << ", ";
//    	}
//    	myfile_2.close();
//    }



    hipMemcpy(h_filter_array, d_filter_array, filter_array_size* sizeof(float*), hipMemcpyDeviceToHost);
	//filter_util(network_config, NeuronList, network_size, input_neuron_num, h_filter_array, d_filter_array, index_prefix, 1);	//write filter to file
    write_neuron_list(NeuronList, ("inf_out_device2_output_network.txt"), spiking_neuron_num);
    //data_check(NeuronList,log_total_spike,SIZE, mnist_start_index, mnist_end_index, 2, "");
    //===clean up===
    //delete[] random_number_list;
    delete[] log_v_host;
	delete[] NeuronList;
	delete[] log_spike_host;
	delete[] log_total_spike_host;
	delete[] NeuronList_temp;
	delete[] one_mnist_img;
	delete[] probe;
//	delete[] random_number_list;
	delete[] spike_flag;
	delete[] num_one_digit_img;
	//hipFree(states);
	hipFree(log_v);
	hipFree(log_spike);
	hipFree(log_total_spike);
	hipFree(Neuron_list_device);
	//hipFree(old_device_neurons);
	hipFree(random_number_list_device);
	hipFree(d_network_config);
	hipFree(states);
	hipFree(spike_flag_device);
	hipFree(log_spike_default);



}

