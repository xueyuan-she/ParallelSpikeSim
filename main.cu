#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <vector>
#include <string>
#include "header.h"
#include <stdlib.h>
#include <streambuf>
#include <sstream>
#include <fstream>
#include <math.h>
#include "CImg.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include "cifar10_reader.hpp"
//#include "learning_options.cu"
//#include "mnist/mnist_reader_less.hpp"
#include <boost/filesystem.hpp>
#include <opencv2/opencv.hpp>
using namespace std;


#define tau 10
#define exp_coeff 1.442695
#define SIZE 50000  //for ROI, use 30000
#define MAX_TIME 2500000 //in ms
#define TEST_TIME 1000

int main()
{
	//clock_t t1_reading, t2_reading;
	clock_t t_start, t_end;
	//float time;

	//cout << "================ Welcome to Xueyuan She and Yun Long's ParallelSpikeSim =================" << endl << endl;
	//cout << endl;
	cout<<"Function Select: ";
	int mode_select;
	cin >> mode_select;
	t_start = clock();

	int input_index = 0;
	switch (mode_select){

					case 0:
					{
						hipSetDevice(0);
						run_cnn_multilayer_inference("", 0.8, -1.0, 10000, 5, "spike_cnn.txt");
					}
					break;

					case 1:
					{
						printf("Case 1 selected/n");
						spiking_learning_label("device2_output_network.txt", "device2_output_network_flaged_network_4.csv", 500, 1000, 1, 0);
					}
					break;
					//case 2: run_test(); break; ./rotating_f_mnist/test_2_4/rotating_mnist_val
					case 3: {

					}
					break;
					case 4:
					{
						hipSetDevice(1);
						run_time_sequence("", 1, -1.0, 10, 5, "spike_cnn.txt");
					}
					break;
					case 5:
						cout<<"Running CNN Multilayer"<<endl;
						hipSetDevice(2);
						run_cnn_multilayer("", 1, -1.0, 100, 5, "spike_cnn.txt");
					break;
					case 6:
					{
						hipSetDevice(0);
						run_cnn("", 0.8, -1.0, 500, 5, "spike_cnn.txt");
					}
					break;
					case 7:
					{
						hipSetDevice(1);
						cout<<"Ruuning H-SNN Learning Layer by Layer"<<endl;
						//run HSNN learning
						for (int layer_to_learn=1; layer_to_learn<CNN_total_layer_num; layer_to_learn++){
							cout<<endl<<"==========Learning Layer "<<layer_to_learn<<"=========="<<endl;
							if (layer_to_learn==1) run_event_based_learning_hsnn("1", 1, -1.0, 2, 5, "spike_cnn.txt", 0, layer_to_learn);
							else run_event_based_learning_hsnn(to_string(layer_to_learn), 1, -1.0, 2, 5, "spike_cnn.txt", 1, layer_to_learn);
						}
					}
					break;
					case 8:
					{
						hipSetDevice(2);
						run_event_based_inference_hsnn("", 0.8, -1.0, 2, 5, "spike_cnn.txt");
					}
					break;


	}

	t_end = clock();
	cout << "Information summary: " << endl;

	//cout << "Calling GPU kernel uses: " << elapase_time[0]/1000 << " (ms)" << endl;
	//cout << "Actual GPU kernel elapse time is: " << elapase_time[1] << " (ms)" << endl << endl;
	cout << "Total simulation time is " << (t_end - t_start)/1000 << " (ms)" << endl;

	cout << endl;
	cout << "============ Simulation is done, please check your output ============" << endl << endl;
	cout << "Thanks for using my Simulator" << endl << endl;

	return 0;
}